#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "kernel.h"
#include "plugin.h"
#include "hip/hip_fp16.h"
#include "gatherNMSOutputs.h"
#include <array>

// __half minus with fallback to float for old sm
inline __device__ __half minus_fb(const __half & a, const __half & b) {
#if __CUDA_ARCH__ >= 530
    return a - b;
#else
    return __float2half(__half2float(a) - __half2float(b));
#endif
}

template <typename T_BBOX>
__device__ T_BBOX saturate(T_BBOX v)
{
    return max(min(v, T_BBOX(1)), T_BBOX(0));
}

template <>
__device__ __half saturate(__half v)
{
#if __CUDA_ARCH__ >= 800
    return __hmax(__hmin(v, __half(1)), __half(0));
#elif __CUDA_ARCH__ >= 530
    return __hge(v, __half(1)) ? __half(1) : (__hle(v, __half(0)) ? __half(0) : v);
#else
    return max(min(v, float(1)), float(0));
#endif
}

template <typename T_BBOX, typename T_SCORE, unsigned nthds_per_cta>
__launch_bounds__(nthds_per_cta)
    __global__ void gatherNMSOutputs_kernel(
        const bool shareLocation,
        const int numImages,
        const int numPredsPerClass,
        const int numClasses,
        const int topK,
        const int keepTopK,
        const int* indices,
        const T_SCORE* scores,
        const T_BBOX* bboxData,
        int* numDetections,
        T_BBOX* nmsedBoxes,
        T_BBOX* nmsedScores,
        T_BBOX* nmsedClasses,
        bool clipBoxes,
        const T_SCORE scoreShift
        )
{
    if (keepTopK > topK)
        return;
    for (int i = blockIdx.x * nthds_per_cta + threadIdx.x;
         i < numImages * keepTopK;
         i += gridDim.x * nthds_per_cta)
    {
        const int imgId = i / keepTopK;
        const int detId = i % keepTopK;
        const int offset = imgId * numClasses * topK;
        const int index = indices[offset + detId];
        const T_SCORE score = scores[offset + detId];
        if (index == -1)
        {
            nmsedClasses[i] = -1;
            nmsedScores[i] = 0;
            nmsedBoxes[i * 4] = 0;
            nmsedBoxes[i * 4 + 1] = 0;
            nmsedBoxes[i * 4 + 2] = 0;
            nmsedBoxes[i * 4 + 3] = 0;
        }
        else
        {
            const int bboxOffset = imgId * (shareLocation ? numPredsPerClass : (numClasses * numPredsPerClass));
            const int bboxId = ((shareLocation ? (index % numPredsPerClass)
                        : index % (numClasses * numPredsPerClass)) + bboxOffset) * 4;
            nmsedClasses[i] = (index % (numClasses * numPredsPerClass)) / numPredsPerClass; // label
            nmsedScores[i] = score;                                                        // confidence score
            nmsedScores[i] = minus_fb(nmsedScores[i], scoreShift);
            const T_BBOX xMin = bboxData[bboxId];
            const T_BBOX yMin = bboxData[bboxId + 1];
            const T_BBOX xMax = bboxData[bboxId + 2];
            const T_BBOX yMax = bboxData[bboxId + 3];
            // clipped bbox xmin
            nmsedBoxes[i * 4] = clipBoxes ? saturate(xMin) : xMin;
            // clipped bbox ymin
            nmsedBoxes[i * 4 + 1] = clipBoxes ? saturate(yMin) : yMin;
            // clipped bbox xmax
            nmsedBoxes[i * 4 + 2] = clipBoxes ? saturate(xMax) : xMax;
            // clipped bbox ymax
            nmsedBoxes[i * 4 + 3] = clipBoxes ? saturate(yMax) : yMax;
            atomicAdd(&numDetections[i / keepTopK], 1);
        }
    }
}

template <typename T_BBOX, typename T_SCORE>
pluginStatus_t gatherNMSOutputs_gpu(
    hipStream_t stream,
    const bool shareLocation,
    const int numImages,
    const int numPredsPerClass,
    const int numClasses,
    const int topK,
    const int keepTopK,
    const void* indices,
    const void* scores,
    const void* bboxData,
    void* numDetections,
    void* nmsedBoxes,
    void* nmsedScores,
    void* nmsedClasses,
    bool clipBoxes,
    const float scoreShift
    )
{
    hipMemsetAsync(numDetections, 0, numImages * sizeof(int), stream);
    const int BS = 32;
    const int GS = 32;
    gatherNMSOutputs_kernel<T_BBOX, T_SCORE, BS><<<GS, BS, 0, stream>>>(shareLocation, numImages, numPredsPerClass,
                                                                           numClasses, topK, keepTopK,
                                                                           (int*) indices, (T_SCORE*) scores, (T_BBOX*) bboxData,
                                                                           (int*) numDetections,
                                                                           (T_BBOX*) nmsedBoxes,
                                                                           (T_BBOX*) nmsedScores,
                                                                           (T_BBOX*) nmsedClasses,
                                                                           clipBoxes,
                                                                           T_SCORE(scoreShift)
                                                                            );

    CSC(hipGetLastError(), STATUS_FAILURE);
    return STATUS_SUCCESS;
}

// gatherNMSOutputs LAUNCH CONFIG {{{
typedef pluginStatus_t (*nmsOutFunc)(hipStream_t,
                               const bool,
                               const int,
                               const int,
                               const int,
                               const int,
                               const int,
                               const void*,
                               const void*,
                               const void*,
                               void*,
                               void*,
                               void*,
                               void*,
                               bool,
                               const float);
struct nmsOutLaunchConfig
{
    DataType t_bbox;
    DataType t_score;
    nmsOutFunc function;

    nmsOutLaunchConfig(DataType t_bbox, DataType t_score)
        : t_bbox(t_bbox)
        , t_score(t_score)
    {
    }
    nmsOutLaunchConfig(DataType t_bbox, DataType t_score, nmsOutFunc function)
        : t_bbox(t_bbox)
        , t_score(t_score)
        , function(function)
    {
    }
    bool operator==(const nmsOutLaunchConfig& other)
    {
        return t_bbox == other.t_bbox && t_score == other.t_score;
    }
};

using nvinfer1::DataType;

static std::array<nmsOutLaunchConfig, 2> nmsOutLCOptions = {
  nmsOutLaunchConfig(DataType::kFLOAT, DataType::kFLOAT, gatherNMSOutputs_gpu<float, float>),
  nmsOutLaunchConfig(DataType::kHALF, DataType::kHALF, gatherNMSOutputs_gpu<__half, __half>)
};

pluginStatus_t gatherNMSOutputs(
    hipStream_t stream,
    const bool shareLocation,
    const int numImages,
    const int numPredsPerClass,
    const int numClasses,
    const int topK,
    const int keepTopK,
    const DataType DT_BBOX,
    const DataType DT_SCORE,
    const void* indices,
    const void* scores,
    const void* bboxData,
    void* numDetections,
    void* nmsedBoxes,
    void* nmsedScores,
    void* nmsedClasses,
    bool clipBoxes,
    const float scoreShift
    )
{
    nmsOutLaunchConfig lc = nmsOutLaunchConfig(DT_BBOX, DT_SCORE);
    for (unsigned i = 0; i < nmsOutLCOptions.size(); ++i)
    {
        if (lc == nmsOutLCOptions[i])
        {
            DEBUG_PRINTF("gatherNMSOutputs kernel %d\n", i);
            return nmsOutLCOptions[i].function(stream,
                                          shareLocation,
                                          numImages,
                                          numPredsPerClass,
                                          numClasses,
                                          topK,
                                          keepTopK,
                                          indices,
                                          scores,
                                          bboxData,
                                          numDetections,
                                          nmsedBoxes,
                                          nmsedScores,
                                          nmsedClasses,
                                          clipBoxes,
                                          scoreShift
                                          );
        }
    }
    return STATUS_BAD_PARAM;
}
