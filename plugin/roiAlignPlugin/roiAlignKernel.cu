#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 1993-2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 * 
 * ************************************************************************
 * Modified from Pytorch
 * Copyright (c) 2016-present, Facebook, Inc.
 *
 * See https://github.com/pytorch/pytorch/blob/master/LICENSE for details
 * ************************************************************************
 * Modified from ONNX Runtime
 * Copyright (c) Microsoft Corporation
 * 
 * See https://github.com/microsoft/onnxruntime/blob/master/LICENSE for details
 * ************************************************************************
 */


#include <hip/hip_runtime.h>
#include "hip/hip_fp16.h"
#include "common/common.cuh"
#include "roiAlignKernel.h"

using half = __half;

__device__ half floatMax(half a, half b)
{
    #if __CUDA_ARCH__ >= 800
        return __hmax(a, b);
    #else
        return __float2half(max(__half2float(a), __half2float(b)));
    #endif
}

__device__ float floatMax(float a, float b)
{
    return max(a, b);
}

template <typename T>
__device__ T bilinearInterpolate(T const* bottomData, int32_t const height, int32_t const width, T y, T x,
    int32_t const isModeAvg, int32_t const index /* index for debug only*/)
{
    // deal with cases that inverse elements are out of feature map boundary
    if (y < static_cast<T>(-1.0) || y > static_cast<T>(height) || x < static_cast<T>(-1.0) || x > static_cast<T>(width))
    {
        // empty
        return 0;
    }

    if (y <= static_cast<T>(0))
    {
        y = 0;
    }
    if (x <= static_cast<T>(0))
    {
        x = 0;
    }

    int32_t yLow = static_cast<int32_t>(y);
    int32_t xLow = static_cast<int32_t>(x);
    int32_t yHigh;
    int32_t xHigh;

    if (yLow >= height - 1)
    {
        yHigh = yLow = height - 1;
        y = static_cast<T>(yLow);
    }
    else
    {
        yHigh = yLow + 1;
    }

    if (xLow >= width - 1)
    {
        xHigh = xLow = width - 1;
        x = static_cast<T>(xLow);
    }
    else
    {
        xHigh = xLow + 1;
    }

    T ly = y - static_cast<T>(yLow);
    T lx = x - static_cast<T>(xLow);
    T hy = static_cast<T>(1.) - ly, hx = static_cast<T>(1.) - lx;
    // do bilinear interpolation
    T v1 = bottomData[yLow * width + xLow];
    T v2 = bottomData[yLow * width + xHigh];
    T v3 = bottomData[yHigh * width + xLow];
    T v4 = bottomData[yHigh * width + xHigh];
    T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

    T val;
    if (isModeAvg)
    {
        val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4); // mode Avg
    }
    else
    {
        val = floatMax(floatMax(floatMax(w1 * v1, w2 * v2), w3 * v3), w4 * v4); // mode Max
    }

    return val;
}

template <typename T>
__global__ void RoIAlignForward(int32_t const nthreads, T const* bottomData, T const spatialScale, int32_t const channels,
    int32_t const height, int32_t const width, int32_t const pooledHeight, int32_t const pooledWidth, int32_t const samplingRatio,
    T const* bottomRois, T* topData, int32_t const isModeAvg, int32_t const* batchIndicesPtr,
    int32_t const aligned)
{
    for (size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads; index += blockDim.x * gridDim.x)
    {
        // (n, c, ph, pw) is an element in the pooled output
        int32_t pw = index % pooledWidth;
        int32_t ph = (index / pooledWidth) % pooledHeight;
        int32_t c = (index / pooledWidth / pooledHeight) % channels;
        int32_t n = index / pooledWidth / pooledHeight / channels;

        T const* offsetBottomRois = bottomRois + n * 4;
        auto const roiBatchInd = batchIndicesPtr[n];

        bool continuousCoordinate = aligned;
        // Do not using rounding; this implementation detail is critical
        T roiOffset = static_cast<T>(continuousCoordinate ? 0.5 : 0);
        T roiStartW = offsetBottomRois[0] * spatialScale - roiOffset;
        T roiStartH = offsetBottomRois[1] * spatialScale - roiOffset;
        T roiEndW = offsetBottomRois[2] * spatialScale - roiOffset;
        T roiEndH = offsetBottomRois[3] * spatialScale - roiOffset;

        T roiWidth = roiEndW - roiStartW;
        T roiHeight = roiEndH - roiStartH;
        if (!continuousCoordinate)
        { // backward compatiblity
            // Force malformed ROIs to be 1x1
            roiWidth = floatMax(roiWidth, static_cast<T>(1.));
            roiHeight = floatMax(roiHeight, static_cast<T>(1.));
        }
        T binSizeH = static_cast<T>(roiHeight) / static_cast<T>(pooledHeight);
        T binSizeW = static_cast<T>(roiWidth) / static_cast<T>(pooledWidth);

        T const* offsetBottomData = bottomData + static_cast<int32_t>((roiBatchInd * channels + c) * height * width);

        // We use roiBinGrid to sample the grid and mimic integral
        int32_t roiBinGridH;
        if (samplingRatio > 0)
        {
            roiBinGridH = samplingRatio;
        }
        else
        {
            roiBinGridH = ceilf(roiHeight / static_cast<T>(pooledHeight));
        }

        int32_t roiBinGridW;
        if (samplingRatio > 0)
        {
            roiBinGridW = samplingRatio;
        }
        else
        {
            roiBinGridW = ceilf(roiWidth / static_cast<T>(pooledWidth));
        }
        // We do average (integral) pooling inside a bin
        T const count = roiBinGridH * roiBinGridW; // e.g. = 4

        T const yOff = roiStartH + static_cast<T>(ph) * binSizeH;
        T const yFac = binSizeH / static_cast<T>(roiBinGridH);

        T const xOff = roiStartW + static_cast<T>(pw) * binSizeW;
        T const xFac = binSizeW / static_cast<T>(roiBinGridW);

        T outputVal = 0.;
        bool maxFlag = false;
        for (int32_t iy = 0; iy < roiBinGridH; iy++) // e.g., iy = 0, 1
        {
            T const y = yOff + static_cast<T>(iy + .5F) * yFac; // e.g., 0.5, 1.5
            for (int32_t ix = 0; ix < roiBinGridW; ix++)
            {
                T const x = xOff + static_cast<T>(ix + .5F) * xFac;

                T val = bilinearInterpolate(offsetBottomData, height, width, y, x, isModeAvg, index);

                if (isModeAvg)
                {
                    outputVal += val;
                }
                else
                {
                    if (!maxFlag)
                    {
                        outputVal = val;
                        maxFlag = true;
                    }
                    else
                    {
                        outputVal = floatMax(outputVal, val);
                    }
                }
            }
        }
        if (isModeAvg)
        {
            outputVal = outputVal / count;
        }

        topData[index] = outputVal;
    }
}

template <typename T>
hipError_t RoiAlignImpl(hipStream_t stream, int32_t const maxThreadsPerBlock, T const* bottomData, T const spatialScale,
    int32_t const numRois, int32_t const channels, int32_t const height, int32_t const width, int32_t const pooledHeight,
    int32_t const pooledWidth, int32_t const samplingRatio, T const* bottomRois, T* topData, int32_t const isModeAvg,
    int32_t const* batchIndicesPtr, int32_t const aligned)
{
    PLUGIN_ASSERT(bottomData != nullptr);
    PLUGIN_ASSERT(bottomRois != nullptr);
    PLUGIN_ASSERT(batchIndicesPtr != nullptr);
    PLUGIN_ASSERT(topData != nullptr);

    PLUGIN_ASSERT(numRois >= 0);
    PLUGIN_ASSERT(maxThreadsPerBlock > 0);

    PLUGIN_ASSERT(height > 0);
    PLUGIN_ASSERT(width > 0);
    PLUGIN_ASSERT(pooledHeight > 0);
    PLUGIN_ASSERT(pooledWidth > 0);
    PLUGIN_ASSERT(samplingRatio >= 0);
    PLUGIN_ASSERT(isModeAvg == 0 || isModeAvg == 1);
    PLUGIN_ASSERT(static_cast<float>(spatialScale) > 0.0F);
    PLUGIN_ASSERT(aligned == 0 || aligned == 1);

    int32_t const outputSize = numRois * channels * pooledHeight * pooledWidth;

    int32_t blocksPerGrid = static_cast<int32_t>(ceil(static_cast<float>(outputSize)
        / maxThreadsPerBlock)); 

    RoIAlignForward<T><<<blocksPerGrid, maxThreadsPerBlock, 0, stream>>>(outputSize,// nthreads
        bottomData,                                                                 // bottomData
        spatialScale,                                                               // spatialScale
        channels,                                                                   // channels
        height,                                                                     // height
        width,                                                                      // width
        pooledHeight,                                                               // pooledHeight
        pooledWidth,                                                                // pooledWidth
        samplingRatio,                                                              // samplingRatio
        bottomRois,                                                                 // bottomRois
        topData,                                                                    // topData
        isModeAvg,                                                                  // isModeAvg
        batchIndicesPtr,                                                            // batchIndicesPtr
        aligned);

    return hipGetLastError();
}

#define SPECIALIZED_IMPL(T)                                                                                            \
    template hipError_t RoiAlignImpl<T>(hipStream_t stream, int32_t const maxThreadsPerBlock, T const* bottomData,   \
        T const spatialScale, int32_t const numRois, int32_t const channels, int32_t const height,                     \
        int32_t const width, int32_t const pooledHeight, int32_t const pooledWidth, int32_t const samplingRatio,       \
        T const* bottomRois, T* topData, int32_t const isModeAvg, int32_t const* batchIndicesPtr,                      \
        int32_t const aligned);

SPECIALIZED_IMPL(float)
SPECIALIZED_IMPL(half)
