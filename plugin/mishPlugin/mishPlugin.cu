#include "hip/hip_runtime.h"
#include <cassert>
#include "mishPlugin.h"

namespace nvinfer1
{
	MishPlugin::MishPlugin(const int cudaThread) : m_threadCount(cudaThread)
	{
	}

	MishPlugin::MishPlugin(const void* data, size_t length)
	{
		assert(length == sizeof(m_inputSize));
		m_inputSize = *reinterpret_cast<const int*>(data);
	}

	void MishPlugin::serialize(void* buffer)
	{
		*reinterpret_cast<int*>(buffer) = m_inputSize;
	}

	size_t MishPlugin::getSerializationSize()
	{
		return sizeof(m_inputSize);
	}

	int MishPlugin::initialize()
	{
		return 0;
	}

	Dims MishPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims)
	{
		assert(nbInputDims == 1);
		assert(index == 0);
		m_inputSize = inputs[0].d[0] * inputs[0].d[1] * inputs[0].d[2];

		return DimsCHW(inputs[0].d[0], inputs[0].d[1], inputs[0].d[2]);
	}

	__device__ float tanh_activate_kernel(float x) { return (2 / (1 + expf(-2 * x)) - 1); }

	__device__ float softplus_kernel(float x, float threshold = 20) {
		if (x > threshold) return x;
		else if (x < -threshold) return expf(x);
		return logf(expf(x) + 1);
	}

	__global__ void mish_kernel(const float *input, float *output, int num_elem) {

		int idx = threadIdx.x + blockDim.x * blockIdx.x;
		if (idx >= num_elem) return;

		output[idx] = input[idx] * tanh_activate_kernel(softplus_kernel(input[idx]));
	}

	void MishPlugin::forwardGPU(const float *const * inputs, float* output, hipStream_t stream, int batchSize) {
		int block_size = m_threadCount;
		int grid_size = (m_inputSize * batchSize + block_size - 1) / block_size;
		mish_kernel << <grid_size, block_size, 0, stream >> >(inputs[0], output, m_inputSize * batchSize);
	}


	int MishPlugin::enqueue(int batchSize, const void*const * inputs, void** outputs, void* workspace, hipStream_t stream)
	{
		forwardGPU((const float *const *)inputs, (float*)outputs[0], stream, batchSize);
		return 0;
	}

	// deserialization plugin implementation
	IPlugin* MishPluginFactory::createPlugin(const char* layerName, const void* serialData, size_t serialLength)
	{
		return new MishPlugin(serialData, serialLength);
	}
}

