#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "NvInfer.h"
#include "bertCommon.h"
#include "common.h"
#include "qkvToContextPlugin.h"
#include "serialize.hpp"

#include <cassert>
#include <cstring>
#include <iostream>
#include <vector>

using namespace nvinfer1;

namespace bert
{

template <typename T, int TPB, int VPT>
__global__ void maskedSoftmax(const float rsqrtHeadSize, const T* input, T* output, const int* maskIdx)
{
    using BlockReduce = hipcub::BlockReduce<float, TPB>;
    __shared__ union
    {
        T shm[VPT * TPB];
        typename BlockReduce::TempStorage reduce;
    } tmp;

    // grid: (NxS, B)
    const int b = blockIdx.y;
    const int blockOffset = (b * gridDim.x + blockIdx.x) * TPB;
    __shared__ int lastValid;
    if (threadIdx.x == 0)
    {
        lastValid = min(TPB, maskIdx[b]);
    }
    __syncthreads();
    float local[VPT];

    __shared__ float rZ;

    const int idx = (blockOffset + threadIdx.x) * VPT;
    T* myshm = &tmp.shm[threadIdx.x * VPT];
    copy<sizeof(T) * VPT>(&input[idx], myshm);

    __syncthreads();

#pragma unroll
    for (int it = 0; it < VPT; it++)
    {
        local[it]
            = (threadIdx.x < lastValid) ? myExp<float>((rsqrtHeadSize) * float(tmp.shm[it * TPB + threadIdx.x])) : 0.f;
    }
    __syncthreads();

#pragma unroll
    for (int it = 0; it < VPT; it++)
    {

        const auto Z = BlockReduce(tmp.reduce).Reduce(local[it], hipcub::Sum());

        if (threadIdx.x == 0)
        {
            rZ = (1.f) / Z;
        }
        __syncthreads();
        local[it] *= rZ;
    }

#pragma unroll
    for (int it = 0; it < VPT; it++)
    {
        tmp.shm[it * TPB + threadIdx.x] = local[it];
    }
    __syncthreads();
    copy<sizeof(T) * VPT>(myshm, &output[idx]);
}

template <typename T, int TPB, int VPT>
__global__ void softmax(const float rsqrtHeadSize, const T* input, T* output)
{
    float local[VPT];

    using BlockReduce = hipcub::BlockReduce<float, TPB>;

    __shared__ union
    {
        T shm[VPT * TPB];
        typename BlockReduce::TempStorage reduce;
    } tmp;

    __shared__ float rZ;

    const int idx = (TPB * blockIdx.x + threadIdx.x) * VPT;
    T* myshm = &tmp.shm[threadIdx.x * VPT];
    copy<sizeof(T) * VPT>(&input[idx], myshm);

    __syncthreads();

#pragma unroll
    for (int it = 0; it < VPT; it++)
    {
        local[it] = myExp<float>(rsqrtHeadSize * float(tmp.shm[it * TPB + threadIdx.x]));
    }
    __syncthreads();

#pragma unroll
    for (int it = 0; it < VPT; it++)
    {

        const auto Z = BlockReduce(tmp.reduce).Reduce(local[it], hipcub::Sum());

        if (threadIdx.x == 0)
        {
            rZ = 1.f / Z;
        }
        __syncthreads();
        local[it] *= rZ;
    }

#pragma unroll
    for (int it = 0; it < VPT; it++)
    {
        tmp.shm[it * TPB + threadIdx.x] = local[it];
    }
    __syncthreads();
    copy<sizeof(T) * VPT>(myshm, &output[idx]);
}

template <typename T, unsigned TPB>
__global__ void scaledSoftmaxKernelSmall(const int ld, const float rsqrtHeadSize, const T* input, T* output)
{
    scaledSoftmaxSmall<T, TPB>(ld, ld, rsqrtHeadSize, input, output);
}

template <typename T, unsigned TPB>
__global__ void scaledSoftmaxKernel(const int ld, const float rsqrtHeadSize, const T* input, T* output)
{
    scaledSoftmax<T, TPB>(ld, ld, rsqrtHeadSize, input, output);
}

template <typename T>
int computeScaledSoftmax(
    hipStream_t stream, const int ld, const int B, const int N, const float rsqrtHeadSize, const T* input, T* output)
{

    constexpr int VPT = 16 / sizeof(T);

    const dim3 grid(ld * N, B, 1);

    if (ld <= 32)
    {
        const int blockSize = 32;
        scaledSoftmaxKernelSmall<T, blockSize><<<grid, blockSize, 0, stream>>>(ld, rsqrtHeadSize, input, output);
    }
    else if (ld < 128)
    {
        const int blockSize = 128;
        scaledSoftmaxKernelSmall<T, blockSize><<<grid, blockSize, 0, stream>>>(ld, rsqrtHeadSize, input, output);
    }
    else if (ld == 128)
    {
        const int grid = B * N * ld / (VPT);
        softmax<T, 128, VPT><<<grid, 128, 0, stream>>>(rsqrtHeadSize, input, output);
    }

    else if (ld == 384)
    {

        const int grid = B * N * ld / (VPT);
        softmax<T, 384, VPT><<<grid, 384, 0, stream>>>(rsqrtHeadSize, input, output);
    }
    else
    {
        const int blockSize = 256;

        scaledSoftmaxKernel<T, blockSize><<<grid, blockSize, 0, stream>>>(ld, rsqrtHeadSize, input, output);
    }

    CHECK(hipPeekAtLastError());
    return 0;
}

template <typename T, unsigned TPB>
__global__ void maskedScaledSoftmaxKernelSmall(
    const int ld, const float rsqrtHeadSize, const int* maskIdx, const T* input, T* output)
{
    __shared__ int lastValid;

    if (threadIdx.x == 0)
    {
        lastValid = min(ld, maskIdx[blockIdx.y]);
    }
    __syncthreads();

    scaledSoftmaxSmall<T, TPB>(ld, lastValid, rsqrtHeadSize, input, output);
}

template <typename T, unsigned TPB>
__global__ void maskedScaledSoftmaxKernel(
    const int ld, const float rsqrtHeadSize, const int* maskIdx, const T* input, T* output)
{

    __shared__ int lastValid;

    if (threadIdx.x == 0)
    {
        lastValid = min(ld, maskIdx[blockIdx.y]);
    }
    __syncthreads();
    scaledSoftmax<T, TPB>(ld, lastValid, rsqrtHeadSize, input, output);
}

template <typename T>
int computeMaskedScaledSoftmax(hipStream_t stream, const int ld, const int B, const int N, const float rsqrtHeadSize,
    const int* maskIdx, const T* input, T* output)
{
    // Mask idx is of length B and assumes the valid region is contiguous starting
    // from the beginning of the sequence

    const dim3 grid(ld * N, B, 1);
    // for smaller problems, e.g. BERT base B=1, this is not optimal
    if (ld <= 32)
    {
        constexpr int blockSize = 32;
        maskedScaledSoftmaxKernelSmall<T, blockSize>
            <<<grid, blockSize, 0, stream>>>(ld, rsqrtHeadSize, maskIdx, input, output);
    }
    else if (ld < 128)
    {
        constexpr int blockSize = 128;
        maskedScaledSoftmaxKernelSmall<T, blockSize>
            <<<grid, blockSize, 0, stream>>>(ld, rsqrtHeadSize, maskIdx, input, output);
    }
    else if (ld == 128)
    {
        if (B == 1)
        {
            constexpr int VPT = 4 / sizeof(T);
            constexpr int blockSize = 128;
            const dim3 grid(ld * N / VPT, B, 1);
            maskedSoftmax<T, blockSize, VPT><<<grid, blockSize, 0, stream>>>(rsqrtHeadSize, input, output, maskIdx);
        }
        else
        {
            constexpr int VPT = 16 / sizeof(T);
            constexpr int blockSize = 128;
            const dim3 grid(ld * N / VPT, B, 1);
            maskedSoftmax<T, blockSize, VPT><<<grid, blockSize, 0, stream>>>(rsqrtHeadSize, input, output, maskIdx);
        }
    }
    else if (ld == 384)
    {
        if (B == 1)
        {
            constexpr int VPT = 4 / sizeof(T);
            constexpr int blockSize = 384;
            const dim3 grid(ld * N / VPT, B, 1);
            maskedSoftmax<T, blockSize, VPT><<<grid, blockSize, 0, stream>>>(rsqrtHeadSize, input, output, maskIdx);
        }
        else
        {
            constexpr int VPT = 16 / sizeof(T);
            constexpr int blockSize = 384;
            const dim3 grid(ld * N / VPT, B, 1);
            maskedSoftmax<T, blockSize, VPT><<<grid, blockSize, 0, stream>>>(rsqrtHeadSize, input, output, maskIdx);
        }
    }
    else
    {
        constexpr int blockSize = 256;
        maskedScaledSoftmaxKernel<T, blockSize>
            <<<grid, blockSize, 0, stream>>>(ld, rsqrtHeadSize, maskIdx, input, output);
    }

    CHECK(hipPeekAtLastError());
    return 0;
}

std::pair<int, int> tuneBatchedGemm(const int B, const int S, const int numHeads, const int headSize)
{
    const int nruns = 500;
    hipblasHandle_t cublas;
    hipblasCreate(&cublas);
    hipStream_t stream;
    hipStreamCreate(&stream);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipblasSetStream(cublas, stream);
    hipblasSetMathMode(cublas, HIPBLAS_TENSOR_OP_MATH);

    using T = half;
    const int omatSize = S * S;
    const int numMats = B * numHeads;
    const int ldQKV = 3 * B * numHeads * headSize;
    const int strideQKV = 3 * headSize;
    const int ldOut = B * numHeads * headSize;
    const int strideOut = headSize;

    const size_t inBytes = S * B * 3 * numHeads * headSize * sizeof(T);
    const size_t qkBytes = S * S * B * numHeads * sizeof(T);
    const size_t outBytes = S * B * numHeads * headSize * sizeof(T);

    T* input = nullptr;
    T* qkptr = nullptr;
    T* output = nullptr;
    hipMalloc(&input, inBytes);
    hipMalloc(&qkptr, qkBytes);
    hipMalloc(&output, outBytes);
    hipMemset(input, 1, inBytes);
    hipMemset(qkptr, 1, qkBytes);

    // input: SxBx3xNxH
    const T* qptr = input;
    const T* kptr = qptr + headSize;
    const T* vptr = kptr + headSize;

    const int startAlgo = (int) CUBLAS_GEMM_DEFAULT_TENSOR_OP;
    const int endAlgo = (int) CUBLAS_GEMM_ALGO15_TENSOR_OP;
    int best1 = startAlgo;
    int best2 = startAlgo;
    float ms1 = 1000000;
    float ms2 = 1000000;
    for (int a = startAlgo; a <= endAlgo; a++)
    {
        hipblasGemmAlgo_t algo = static_cast<hipblasGemmAlgo_t>(a);
        float ms1_, ms2_;
        // qkptr: BxNxSxS
        hipEventRecord(start, stream);
        for (int r = 0; r < nruns; r++)
        {
            CHECK(hipblasGemmStridedBatchedEx<T>(cublas, HIPBLAS_OP_T, HIPBLAS_OP_N, S, S, headSize, T(1.f), kptr, ldQKV,
                strideQKV, qptr, ldQKV, strideQKV, T(0.f), qkptr, S, omatSize, numMats, algo));
        }

        hipEventRecord(stop, stream);
        hipStreamSynchronize(stream);
        hipEventElapsedTime(&ms1_, start, stop);
        if (ms1_ < ms1)
        {
            best1 = algo;
            ms1 = ms1_;
        }

        // pptr: BxNxSxS
        // output: SxBxNxH
        hipEventRecord(start, stream);
        for (int r = 0; r < nruns; r++)
        {
            CHECK(hipblasGemmStridedBatchedEx<T>(cublas, HIPBLAS_OP_N, HIPBLAS_OP_N, headSize, S, S, 1.f, vptr, ldQKV,
                strideQKV, qkptr, S, omatSize, 0.f, output, ldOut, strideOut, numMats, algo));
        }

        hipEventRecord(stop, stream);
        hipStreamSynchronize(stream);
        hipEventElapsedTime(&ms2_, start, stop);

        if (ms2_ < ms2)
        {
            best2 = algo;
            ms2 = ms2_;
        }
    }

    hipFree(input);
    hipFree(qkptr);
    hipFree(output);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipStreamDestroy(stream);
    hipblasDestroy(cublas);
    return std::make_pair(best1, best2);
}

template <typename T>
int QKVToContextPluginDynamic::qkvToCtx(hipblasHandle_t& cublas, const int B, const int S, const int numHeads,
    const int headSize, const float rsqrtHeadSize, const T* input, T* output, T* qkptr, T* pptr, hipStream_t stream,
    const int* maskIdx)
{

    const int omatSize = S * S;
    const int numMats = B * numHeads;
    const T* qptr = input;
    const T* kptr = qptr + headSize;
    const T* vptr = kptr + headSize;

    hipblasSetStream(cublas, stream);
    CublasConfigHelper helper(cublas);

    // Q, K, V: BxNxSxH (inputs)
    // Q * K': BxNxSxS (-> scratch1)
    // P: BxNxSxS (-> scratch2)
    // P * V: BxNxSxH (output)

    const int ldQKV = 3 * B * numHeads * headSize;
    const int strideQKV = 3 * headSize;
    
    if (mType == DataType::kHALF)
    {
        CHECK(hipblasGemmStridedBatchedEx<T>(cublas, HIPBLAS_OP_T, HIPBLAS_OP_N, S, S, headSize, 1.f, kptr, ldQKV,
            strideQKV, qptr, ldQKV, strideQKV, 0.f, qkptr, S, omatSize, numMats,
            static_cast<hipblasGemmAlgo_t>(mAlgoBatchedEx1)));
    }
    else 
    {

        CHECK(cublasGemmStridedBatched<T>(cublas, HIPBLAS_OP_T, HIPBLAS_OP_N, S, S, headSize, 1.f, kptr, ldQKV, strideQKV,
            qptr, ldQKV, strideQKV, 0.f, qkptr, S, omatSize, numMats));
    }

    // apply softmax
    if (maskIdx)
    { // if we have a mask
        computeMaskedScaledSoftmax<T>(stream, S, B, numHeads, rsqrtHeadSize, maskIdx, qkptr, pptr);
    }
    else
    { // if we don't have a mask
        computeScaledSoftmax<T>(stream, S, B, numHeads, rsqrtHeadSize, qkptr, pptr);
    }

    // compute P*V (as V*P)

    const int ldOut = B * numHeads * headSize;
    const int strideOut = headSize;
    if (mType == DataType::kHALF)
    {

        CHECK(hipblasGemmStridedBatchedEx<T>(cublas, HIPBLAS_OP_N, HIPBLAS_OP_N, headSize, S, S, 1.f, vptr, ldQKV,
            strideQKV, pptr, S, omatSize, 0.f, output, ldOut, strideOut, numMats,
            static_cast<hipblasGemmAlgo_t>(mAlgoBatchedEx2)));
    }
    else 
    {

        CHECK(cublasGemmStridedBatched<T>(cublas, HIPBLAS_OP_N, HIPBLAS_OP_N, headSize, S, S, 1.f, vptr, ldQKV, strideQKV,
            pptr, S, omatSize, 0.f, output, ldOut, strideOut, numMats));
    }
    return 0;
}

namespace
{
static const char* QKV_TO_CONTEXT_PLUGIN_VERSION{"1"};
static const char* QKV_TO_CONTEXT_PLUGIN_NAME{"CustomQKVToContextPluginDynamic"};
} // namespace

// Static class fields initialization
PluginFieldCollection QKVToContextPluginDynamicCreator::mFC{};
std::vector<PluginField> QKVToContextPluginDynamicCreator::mPluginAttributes;

REGISTER_TENSORRT_PLUGIN(QKVToContextPluginDynamicCreator);

constexpr size_t kAlignment = 256;
constexpr uint32_t IIDX = 0; // index of the input tensor
constexpr uint32_t MIDX = 1; // index of the mask

QKVToContextPluginDynamic::QKVToContextPluginDynamic(
    const std::string name, const DataType type, const int hiddenSize, const int numHeads, bool hasImask)
    : mLayerName(name)
    , mHiddenSize(hiddenSize)
    , mNumHeads(numHeads)
    , mHasImask(hasImask)
    , mType(type)
      , mAlgoBatchedEx1(CUBLAS_GEMM_DEFAULT_TENSOR_OP)
      , mAlgoBatchedEx2(CUBLAS_GEMM_DEFAULT_TENSOR_OP)
{
    assert(hiddenSize % numHeads == 0);
    mHeadSize = hiddenSize / numHeads;
    mRsqrtHeadSize = 1.f / sqrt(float(mHeadSize));
}

QKVToContextPluginDynamic::QKVToContextPluginDynamic(const std::string name, const void* data, size_t length)
    : mLayerName(name)
{
    gLogVerbose << "QKV Deser Start" << std::endl;
    deserialize_value(&data, &length, &mType);
    deserialize_value(&data, &length, &mNumHeads);
    deserialize_value(&data, &length, &mHeadSize);
    deserialize_value(&data, &length, &mRsqrtHeadSize);
    deserialize_value(&data, &length, &mHasImask);
    deserialize_value(&data, &length, &mHiddenSize);
    deserialize_value(&data, &length, &mAlgoBatchedEx1);
    deserialize_value(&data, &length, &mAlgoBatchedEx2);
    gLogVerbose << "QKV Deser done" << std::endl;
}

// IPluginV2DynamicExt Methods
nvinfer1::IPluginV2DynamicExt* QKVToContextPluginDynamic::clone() const
{
    gLogVerbose << "QKV Clone" << std::endl;
    auto ret = new QKVToContextPluginDynamic(mLayerName, mType, mHiddenSize, mNumHeads, mHasImask);
    ret->initialize();
    gLogVerbose << "QKV Clone done" << std::endl;
    return ret;
}

DimsExprs QKVToContextPluginDynamic::getOutputDimensions(
    int outputIndex, const DimsExprs* inputs, int nbInputs, IExprBuilder& exprBuilder)
{
    // Input is BxSx3*N*H, output should be BxSxN*H
    assert(outputIndex == 0);
    // Copy over everything
    DimsExprs output(inputs[IIDX]);
    // Divide last dim by three
    auto three = exprBuilder.constant(3);
    output.d[HDIM] = exprBuilder.operation(DimensionOperation::kFLOOR_DIV, *inputs[IIDX].d[HDIM], *three);
    return output;
}
bool QKVToContextPluginDynamic::supportsFormatCombination(
    int pos, const PluginTensorDesc* inOut, int nbInputs, int nbOutputs)
{
    assert(pos >= 0);
    assert(pos < 2 + mHasImask);
    assert(nbInputs == 1 + mHasImask);
    const auto* in = inOut;
    const auto* out = inOut + nbInputs;
    if (pos == 0)
    {
        // must not check descriptions > pos
        return (in->type == mType) &&                // precision
            (in->format == TensorFormat::kLINEAR) && // format
            (in->dims.nbDims == 5) &&                // num dims
            ((in->dims.d[HDIM] % 3) == 0) &&         // see getOutputDimensions
            ((in->dims.d[3]) == 1) &&                // for fc
            ((in->dims.d[4]) == 1)                   // for fc
            ;
    }
    else
    { // pos==1
        if ((mHasImask && pos == 1))
        {
            const auto* inMask = &inOut[1];
            return (inMask->type == DataType::kINT32) &&     // precision
                (inMask->format == TensorFormat::kLINEAR) && // format
                (inMask->dims.nbDims == 1) &&                // num dims
                ((inMask->dims.d[0]) == in->dims.d[BDIM])    // check B
                ;
        }
        if (!mHasImask || (pos == 2))
        {
            return (in->type == out->type) &&                      // precision
                (out->format == TensorFormat::kLINEAR) &&          // format
                (out->dims.nbDims == 5) &&                         // num dims
                ((in->dims.d[HDIM] / 3) == (out->dims.d[HDIM])) && // div 3
                ((out->dims.d[3]) == 1) &&                         // for fc
                ((out->dims.d[4]) == 1) &&                         // for fc
                ((out->dims.d[BDIM]) == in->dims.d[BDIM]) &&       // check B
                ((out->dims.d[SDIM]) == in->dims.d[SDIM])          // check S
                ;
        }
    }
    return false;
}
void QKVToContextPluginDynamic::configurePlugin(
    const DynamicPluginTensorDesc* in, int nbInputs, const DynamicPluginTensorDesc* out, int nbOutputs)
{
    assert(nbInputs == 1 + mHasImask);
    assert(nbOutputs == 1);
    const PluginTensorDesc& inDesc = in[IIDX].desc;
    TRT_UNUSED inDesc;
    const PluginTensorDesc& outDesc = out->desc;
    TRT_UNUSED outDesc;
    assert(mType == inDesc.type);
    assert(mType == outDesc.type);
    assert(inDesc.dims.d[BDIM] == outDesc.dims.d[BDIM]);
    assert(inDesc.dims.d[SDIM] == outDesc.dims.d[SDIM]);
    assert(inDesc.dims.d[HDIM] == 3 * outDesc.dims.d[HDIM]);
    if (mHasImask)
    {
        const PluginTensorDesc& maskDesc = in[MIDX].desc;
        TRT_UNUSED maskDesc;
        assert(maskDesc.type == DataType::kINT32);
        assert(maskDesc.dims.d[0] == inDesc.dims.d[BDIM]);
    }

    const int S = in->max.d[SDIM];
    const int B = in->max.d[BDIM];
    std::tie(mAlgoBatchedEx1, mAlgoBatchedEx2) = tuneBatchedGemm(B, S, mNumHeads, mHeadSize);
    gLogVerbose << "QKV Plugin - Selected Algos for batch gemms: " << mAlgoBatchedEx1 << ", " << mAlgoBatchedEx2 << "\n";
}

size_t QKVToContextPluginDynamic::scratchSize(const int B, const int S) const
{
    const size_t wordSize = samplesCommon::getElementSize(mType);
    const size_t len = B * mNumHeads * S * S;
    const size_t bytes = len * wordSize;

    return bytes;
}

size_t QKVToContextPluginDynamic::getWorkspaceSize(
    const PluginTensorDesc* inputs, int nbInputs, const PluginTensorDesc* outputs, int nbOutputs) const
{
    const int B = inputs->dims.d[BDIM];
    const int S = inputs->dims.d[SDIM];

    const size_t bytesAligned = alignTo<size_t>(scratchSize(B, S), kAlignment);
    const size_t ws = 2UL * bytesAligned;

    return ws;
}

// IPluginV2Ext Methods
DataType QKVToContextPluginDynamic::getOutputDataType(
    int index, const nvinfer1::DataType* inputTypes, int nbInputs) const
{
    assert(index == 0);
    assert(inputTypes[0] == DataType::kFLOAT || inputTypes[0] == DataType::kHALF);
    return inputTypes[0];
}

// IPluginV2 Methods
const char* QKVToContextPluginDynamic::getPluginType() const
{
    return QKV_TO_CONTEXT_PLUGIN_NAME;
}

const char* QKVToContextPluginDynamic::getPluginVersion() const
{
    return QKV_TO_CONTEXT_PLUGIN_VERSION;
}

int QKVToContextPluginDynamic::getNbOutputs() const
{
    return 1;
}

int QKVToContextPluginDynamic::initialize()
{
    hipblasCreate(&cublas);
    return 0;
}

void QKVToContextPluginDynamic::terminate()
{
    CHECK(hipblasDestroy(cublas));
}

size_t QKVToContextPluginDynamic::getSerializationSize() const
{
    return sizeof(mNumHeads) + sizeof(mHeadSize) + sizeof(DataType) + sizeof(mRsqrtHeadSize) + sizeof(mHasImask)
        + sizeof(mHiddenSize) + sizeof(mAlgoBatchedEx1) + sizeof(mAlgoBatchedEx2);
}

void QKVToContextPluginDynamic::serialize(void* buffer) const
{
    serialize_value(&buffer, mType);
    serialize_value(&buffer, mNumHeads);
    serialize_value(&buffer, mHeadSize);
    serialize_value(&buffer, mRsqrtHeadSize);
    serialize_value(&buffer, mHasImask);
    serialize_value(&buffer, mHiddenSize);
    serialize_value(&buffer, mAlgoBatchedEx1);
    serialize_value(&buffer, mAlgoBatchedEx2);
}

void QKVToContextPluginDynamic::destroy()
{
    delete this;
}

void QKVToContextPluginDynamic::setPluginNamespace(const char* libNamespace)
{
    mNamespace = libNamespace;
}

const char* QKVToContextPluginDynamic::getPluginNamespace() const
{
    return mNamespace.c_str();
}

int QKVToContextPluginDynamic::enqueue(const PluginTensorDesc* inputDesc, const PluginTensorDesc* outputDesc,
    const void* const* inputs, void* const* outputs, void* workspace, hipStream_t stream)
{

    const int batchSize = inputDesc->dims.d[BDIM];
    const int S = inputDesc->dims.d[SDIM];

    const size_t bytesAligned = alignTo<size_t>(scratchSize(batchSize, S), kAlignment);
    char* scratch1 = static_cast<char*>(workspace);
    char* scratch2 = scratch1 + bytesAligned;

    const int* maskIdx = mHasImask ? static_cast<const int*>(inputs[1]) : nullptr;

    int status = -1;
    if (mType == DataType::kFLOAT)
    {
        const float* input = static_cast<const float*>(inputs[0]);
        float* output = static_cast<float*>(outputs[0]);
        float* scr1 = reinterpret_cast<float*>(scratch1);
        float* scr2 = reinterpret_cast<float*>(scratch2);

        status = qkvToCtx(
            cublas, batchSize, S, mNumHeads, mHeadSize, mRsqrtHeadSize, input, output, scr1, scr2, stream, maskIdx);
    }
    else if (mType == DataType::kHALF)
    {
        const half* input = static_cast<const half*>(inputs[0]);
        half* output = static_cast<half*>(outputs[0]);
        half* scr1 = reinterpret_cast<half*>(scratch1);
        half* scr2 = reinterpret_cast<half*>(scratch2);

        status = qkvToCtx(
            cublas, batchSize, S, mNumHeads, mHeadSize, mRsqrtHeadSize, input, output, scr1, scr2, stream, maskIdx);
    }
    else
    {
        assert(false);
    }

    return status;
}

QKVToContextPluginDynamicCreator::QKVToContextPluginDynamicCreator()
{
    mFC.nbFields = mPluginAttributes.size();
    mFC.fields = mPluginAttributes.data();
}

const char* QKVToContextPluginDynamicCreator::getPluginName() const
{
    return QKV_TO_CONTEXT_PLUGIN_NAME;
}

const char* QKVToContextPluginDynamicCreator::getPluginVersion() const
{
    return QKV_TO_CONTEXT_PLUGIN_VERSION;
}

const PluginFieldCollection* QKVToContextPluginDynamicCreator::getFieldNames()
{
    return &mFC;
}

IPluginV2* QKVToContextPluginDynamicCreator::createPlugin(const char* name, const PluginFieldCollection* fc)
{
    gLogVerbose << "Creating QKV2ContextPlugin...\n";

    int hiddenSize = 0;
    int numHeads = 0;
    bool hasMask = false;
    int typeId = -1;

    for (int i = 0; i < fc->nbFields; i++)
    {
        std::string field_name(fc->fields[i].name);

        if (field_name.compare("type_id") == 0)
        {
            typeId = *static_cast<const int*>(fc->fields[i].data);
            gLogVerbose << "Building typeId: " << typeId << std::endl;
        }
        if (field_name.compare("hidden_size") == 0)
        {
            hiddenSize = *static_cast<const int*>(fc->fields[i].data);
            gLogVerbose << "Building hiddenSize: " << hiddenSize << std::endl;
        }
        if (field_name.compare("num_heads") == 0)
        {
            numHeads = *static_cast<const int*>(fc->fields[i].data);
            gLogVerbose << "Building numHeads: " << numHeads << std::endl;
        }
        if (field_name.compare("has_mask") == 0)
        {
            hasMask = *static_cast<const bool*>(fc->fields[i].data);
            gLogVerbose << "Building hasMask: " << hasMask << std::endl;
        }
    }
    if (typeId < 0 || typeId > 3)
    {
        gLogError << "QKV: Invalid TypeId " << typeId << std::endl;
    }

    if (hiddenSize <= 0)
    {
        gLogError << "QKV: Invalid hiddenSize " << hiddenSize << std::endl;
    }

    if (numHeads <= 0)
    {
        gLogError << "QKV: Invalid numHeads " << numHeads << std::endl;
    }

    gLogVerbose << "Building the Plugin...\n";
    DataType type = static_cast<DataType>(typeId);
    QKVToContextPluginDynamic* p = new QKVToContextPluginDynamic(name, type, hiddenSize, numHeads, hasMask);
    return p;
}

IPluginV2* QKVToContextPluginDynamicCreator::deserializePlugin(
    const char* name, const void* serialData, size_t serialLength)
{
    // This object will be deleted when the network is destroyed, which will
    // call QKVToContextPluginDynamic::destroy()
    return new QKVToContextPluginDynamic(name, serialData, serialLength);
}

void QKVToContextPluginDynamicCreator::setPluginNamespace(const char* libNamespace)
{
    mNamespace = libNamespace;
}

const char* QKVToContextPluginDynamicCreator::getPluginNamespace() const
{
    return mNamespace.c_str();
}
} // namespace bert
