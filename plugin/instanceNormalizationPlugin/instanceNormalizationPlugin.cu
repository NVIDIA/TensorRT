#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 1993-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "common/checkMacrosPlugin.h"
#include "instanceNormalizationPlugin.h"
#include <algorithm>
#include <hip/hip_fp16.h>
#include <stdexcept>

using namespace nvinfer1;
using nvinfer1::plugin::InstanceNormalizationPlugin;
using nvinfer1::plugin::InstanceNormalizationPluginV2;
using nvinfer1::plugin::InstanceNormalizationPluginCreator;
using nvinfer1::plugin::InstanceNormalizationPluginCreatorV2;

template <typename T, int32_t THREADS_PER_CTA>
__global__ __launch_bounds__(THREADS_PER_CTA) void in3dReluActivation(
    T* __restrict dst, T const* __restrict src, float alpha, int32_t count)
{
    int32_t idx = blockIdx.x * THREADS_PER_CTA + threadIdx.x;
    if (idx >= count)
    {
        return;
    }

    float val = src[idx];
    dst[idx] = (val < 0.f) ? val * alpha : val;
}

hipdnnStatus_t convertTrt2cudnnDtype(nvinfer1::DataType trt_dtype, hipdnnDataType_t* cudnn_dtype)
{
    switch (trt_dtype)
    {
    case nvinfer1::DataType::kFLOAT: *cudnn_dtype = HIPDNN_DATA_FLOAT; break;
    case nvinfer1::DataType::kHALF: *cudnn_dtype = HIPDNN_DATA_HALF; break;
    default: return HIPDNN_STATUS_BAD_PARAM;
    }
    return HIPDNN_STATUS_SUCCESS;
}

namespace
{
constexpr char const* INSTANCE_PLUGIN_VERSION{"1"};
constexpr char const* INSTANCE_PLUGIN_VERSION_V2{"2"};
constexpr char const* INSTANCE_PLUGIN_NAME{"InstanceNormalization_TRT"};
} // namespace

PluginFieldCollection InstanceNormalizationPluginCreator::mFC{};
std::vector<PluginField> InstanceNormalizationPluginCreator::mPluginAttributes;

InstanceNormalizationPlugin::InstanceNormalizationPlugin(
    float epsilon, std::vector<float> const& scale, std::vector<float> const& bias, int32_t relu, float alpha)
    : mEpsilon(epsilon)
    , mAlpha(alpha)
    , mRelu(relu)
    , mNchan(scale.size())
    , mHostScale(scale)
    , mHostBias(bias)
{
    PLUGIN_VALIDATE(scale.size() == bias.size());
}

InstanceNormalizationPlugin::InstanceNormalizationPlugin(
    float epsilon, nvinfer1::Weights const& scale, nvinfer1::Weights const& bias, int32_t relu, float alpha)
    : mEpsilon(epsilon)
    , mAlpha(alpha)
    , mRelu(relu)
    , mNchan(scale.count)
{
    PLUGIN_VALIDATE(scale.count == bias.count);
    auto const copyWeights = [](nvinfer1::Weights const& input, std::vector<float>& output)
    {
        output.reserve(input.count);
        if (input.type == nvinfer1::DataType::kFLOAT)
        {
            output.assign(
                static_cast<float const*>(input.values), static_cast<float const*>(input.values) + input.count);
        }
        else if (input.type == nvinfer1::DataType::kHALF)
        {
            for (int32_t c = 0; c < input.count; ++c)
            {
                auto const value = static_cast<unsigned short const*>(input.values);
                output.push_back(__internal_half2float(value[c]));
            }
        }
        else
        {
            throw std::runtime_error("Unsupported scale/bias dtype");
        }
    };

    copyWeights(scale, mHostScale);
    copyWeights(bias, mHostBias);
}

InstanceNormalizationPlugin::InstanceNormalizationPlugin(void const* serialData, size_t serialLength)
{
    deserialize_value(&serialData, &serialLength, &mEpsilon);
    deserialize_value(&serialData, &serialLength, &mNchan);
    deserialize_value(&serialData, &serialLength, &mHostScale);
    deserialize_value(&serialData, &serialLength, &mHostBias);
    deserialize_value(&serialData, &serialLength, &mRelu);
    deserialize_value(&serialData, &serialLength, &mAlpha);
}

InstanceNormalizationPlugin::~InstanceNormalizationPlugin()
{
    terminate();
}

// InstanceNormalizationPlugin returns one output.
int32_t InstanceNormalizationPlugin::getNbOutputs() const noexcept
{
    return 1;
}

DimsExprs InstanceNormalizationPlugin::getOutputDimensions(int32_t outputIndex, nvinfer1::DimsExprs const* inputs,
    int32_t nbInputs, nvinfer1::IExprBuilder& exprBuilder) noexcept
{
    nvinfer1::DimsExprs output(inputs[0]);
    return output;
}

int32_t InstanceNormalizationPlugin::initialize() noexcept
{
    if (!mInitialized)
    {
        PLUGIN_CHECK_CUDNN(hipdnnCreate(&mCudnnHandle));

        PLUGIN_CHECK_CUDNN(hipdnnCreateTensorDescriptor(&mBDescriptor));
        PLUGIN_CHECK_CUDNN(hipdnnCreateTensorDescriptor(&mXDescriptor));
        PLUGIN_CHECK_CUDNN(hipdnnCreateTensorDescriptor(&mYDescriptor));

        // NDHWC path
        // Device info.
        int32_t device;
        PLUGIN_CHECK_CUDA(hipGetDevice(&device));
        hipDeviceProp_t props;
        PLUGIN_CHECK_CUDA(hipGetDeviceProperties(&props, device));

        mContext.sm_count = props.multiProcessorCount;
        mContext.sm_shared_size = props.sharedMemPerMultiprocessor;
        mContext.sm_version = props.major * 100 + props.minor * 10;

        PLUGIN_CHECK_CUDA(hipMalloc(&mDeviceScale, mNchan * sizeof(float)));
        PLUGIN_CHECK_CUDA(hipMalloc(&mDeviceBias, mNchan * sizeof(float)));
        PLUGIN_CHECK_CUDA(hipMemcpy(mDeviceScale, &mHostScale[0], mNchan * sizeof(float), hipMemcpyHostToDevice));
        PLUGIN_CHECK_CUDA(hipMemcpy(mDeviceBias, &mHostBias[0], mNchan * sizeof(float), hipMemcpyHostToDevice));
    }
    mInitialized = true;

    return 0;
}

void InstanceNormalizationPlugin::terminate() noexcept
{
    if (mInitialized)
    {
        PLUGIN_CUDNNASSERT(hipdnnDestroyTensorDescriptor(mYDescriptor));
        PLUGIN_CUDNNASSERT(hipdnnDestroyTensorDescriptor(mXDescriptor));
        PLUGIN_CUDNNASSERT(hipdnnDestroyTensorDescriptor(mBDescriptor));

        PLUGIN_CUDNNASSERT(hipdnnDestroy(mCudnnHandle));

        PLUGIN_CUASSERT(hipFree(mDeviceBias));
        PLUGIN_CUASSERT(hipFree(mDeviceScale));
    }
    mInitialized = false;
}

size_t InstanceNormalizationPlugin::getWorkspaceSize(nvinfer1::PluginTensorDesc const* inputs, int32_t nbInputs,
    nvinfer1::PluginTensorDesc const* outputs, int32_t nbOutputs) const noexcept
{
    nvinfer1::Dims input_dims = inputs[0].dims;
    PLUGIN_ASSERT(input_dims.nbDims == 4 || input_dims.nbDims == 5);

    if (inputs[0].format == nvinfer1::PluginFormat::kLINEAR)
    {
        nvinfer1::Dims input_dims = inputs[0].dims;

        int32_t n = input_dims.d[0];
        int32_t c = input_dims.d[1];

        size_t nchan_bytes = c * sizeof(float);
        size_t scale_size = n * nchan_bytes;
        size_t bias_size = n * nchan_bytes;

        size_t total_wss = scale_size + bias_size;

        return total_wss;
    }
    else if (inputs[0].format == nvinfer1::PluginFormat::kDHWC8 || inputs[0].format == nvinfer1::PluginFormat::kCDHW32)
    {
        PLUGIN_ASSERT(input_dims.nbDims == 5);
        int32_t input_data_type = (inputs[0].type == nvinfer1::DataType::kHALF) ? 1 : 2;
        int32_t output_data_type = (outputs[0].type == nvinfer1::DataType::kHALF) ? 1 : 2;
        nvinfer1::Dims input_dims = inputs[0].dims;

        int32_t n = input_dims.d[0];
        int32_t c = input_dims.d[1];
        int32_t d = input_dims.d[2];
        int32_t h = input_dims.d[3];
        int32_t w = input_dims.d[4];

        InstanceNormFwdParams params;
        // only these parameters are required for workspace computation
        params.nhw = d * h * w;
        params.c = c;
        params.n = n;
        // Reserve memory for the workspaces.
        size_t size_sums, size_counts, size_retired_ctas;
        instanceNormBufferSizesDispatch(
            mContext, params, size_sums, size_counts, size_retired_ctas, input_data_type, output_data_type);
        size_t size_nc = n * c * sizeof(float);
        size_nc = ((size_nc + 256 - 1) / 256) * 256;
        return size_sums + size_counts + size_retired_ctas + 4 * size_nc;
    }
    else
    {
        PLUGIN_ASSERT(0);
    }
    return 0;
}

int32_t InstanceNormalizationPlugin::enqueue(nvinfer1::PluginTensorDesc const* inputDesc,
    nvinfer1::PluginTensorDesc const* outputDesc, void const* const* inputs, void* const* outputs, void* workspace,
    hipStream_t stream) noexcept
{
    nvinfer1::Dims input_dims = inputDesc[0].dims;
    // early return for empty tensor
    if (std::any_of(input_dims.d, input_dims.d + input_dims.nbDims, [](int32_t d) { return d == 0; }))
    {
        return 0;
    }

    auto const callRelu = [this, &stream](void* inOut, int32_t count, nvinfer1::DataType type) {
        if (mRelu > 0)
        {
            int32_t constexpr kBLOCK_SZ = 256;
            switch (type)
            {
            case nvinfer1::DataType::kFLOAT:
                in3dReluActivation<float, kBLOCK_SZ><<<(count + kBLOCK_SZ - 1) / kBLOCK_SZ, kBLOCK_SZ, 0, stream>>>(
                    static_cast<float*>(inOut), static_cast<float*>(inOut), mAlpha, count);
                break;
            case nvinfer1::DataType::kHALF:
                in3dReluActivation<__half, kBLOCK_SZ><<<(count + kBLOCK_SZ - 1) / kBLOCK_SZ, kBLOCK_SZ, 0, stream>>>(
                    static_cast<__half*>(inOut), static_cast<__half*>(inOut), mAlpha, count);
                break;
            default: PLUGIN_ASSERT(0);
            }
        }
    };

    if (input_dims.nbDims <= 4)
    {
        nvinfer1::Dims input_dims = inputDesc[0].dims;
        int32_t n = input_dims.d[0];
        int32_t c = input_dims.d[1];
        int32_t h = input_dims.d[2];
        int32_t w = input_dims.nbDims > 3 ? input_dims.d[3] : 1;
        size_t nchan_bytes = c * sizeof(float);

        float* _d_array = static_cast<float*>(workspace);
        float* d_scale = &_d_array[0];
        float* d_bias = &_d_array[n * c];
        for (int32_t i = 0; i < n; ++i)
        {
            PLUGIN_CUASSERT(
                hipMemcpyAsync(d_scale + i * c, mDeviceScale, nchan_bytes, hipMemcpyDeviceToDevice, stream));
            PLUGIN_CUASSERT(
                hipMemcpyAsync(d_bias + i * c, mDeviceBias, nchan_bytes, hipMemcpyDeviceToDevice, stream));
        }

        PLUGIN_CUDNNASSERT(
            hipdnnSetTensor4dDescriptor(mBDescriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, n * c, 1, 1));
        hipdnnDataType_t cudnn_dtype{};
        PLUGIN_CUDNNASSERT(convertTrt2cudnnDtype(inputDesc[0].type, &cudnn_dtype));
        PLUGIN_CUDNNASSERT(hipdnnSetTensor4dDescriptor(mXDescriptor, HIPDNN_TENSOR_NCHW, cudnn_dtype, 1, n * c, h, w));
        PLUGIN_CUDNNASSERT(hipdnnSetTensor4dDescriptor(mYDescriptor, HIPDNN_TENSOR_NCHW, cudnn_dtype, 1, n * c, h, w));
        float alpha = 1;
        float beta = 0;
        void const* x_ptr = inputs[0];
        void* y_ptr = outputs[0];
        PLUGIN_CUDNNASSERT(hipdnnSetStream(mCudnnHandle, stream));
        // Note: Use of HIPDNN_BATCHNORM_SPATIAL_PERSISTENT can cause numerical
        //       overflows (NaNs) for fp32 data in some circumstances. The lower-
        //       performance HIPDNN_BATCHNORM_SPATIAL should be used if this is not
        //       acceptable.
        PLUGIN_CUDNNASSERT(hipdnnBatchNormalizationForwardTraining(mCudnnHandle, HIPDNN_BATCHNORM_SPATIAL_PERSISTENT,
            &alpha, &beta, mXDescriptor, x_ptr, mYDescriptor, y_ptr, mBDescriptor, d_scale, d_bias, 1., nullptr,
            nullptr, mEpsilon, nullptr, nullptr));

        callRelu(y_ptr, n * c * h * w, inputDesc[0].type);
    }
    else
    {
        if (inputDesc[0].format == nvinfer1::PluginFormat::kLINEAR)
        {
            PLUGIN_CHECK_CUDNN(hipdnnSetStream(mCudnnHandle, stream));
            nvinfer1::Dims input_dims = inputDesc[0].dims;
            int32_t n = input_dims.d[0];
            int32_t c = input_dims.d[1];
            int32_t d = input_dims.d[2];
            int32_t h = input_dims.d[3];
            int32_t w = input_dims.d[4];
            size_t nchan_bytes = c * sizeof(float);

            // Note: We repeat the data for each batch entry so that we can do the full
            //       computation in a single CUDNN call in enqueue().
            float* _d_array = (float*) workspace;
            float* d_scale = &_d_array[0];
            float* d_bias = &_d_array[n * c];
            for (int32_t i = 0; i < n; ++i)
            {
                PLUGIN_CHECK_CUDA(
                    hipMemcpyAsync(d_scale + i * c, mDeviceScale, nchan_bytes, hipMemcpyDeviceToDevice, stream));
                PLUGIN_CHECK_CUDA(
                    hipMemcpyAsync(d_bias + i * c, mDeviceBias, nchan_bytes, hipMemcpyDeviceToDevice, stream));
            }

            int32_t nc_dimA[] = {1, n * c, 1, 1, 1};
            int32_t nc_strideA[] = {nc_dimA[1] * nc_dimA[2] * nc_dimA[3] * nc_dimA[4],
                nc_dimA[2] * nc_dimA[3] * nc_dimA[4], nc_dimA[3] * nc_dimA[4], nc_dimA[4], 1};
            int32_t img_dimA[] = {1, n * c, d, h, w};
            int32_t img_strideA[] = {img_dimA[1] * img_dimA[2] * img_dimA[3] * img_dimA[4],
                img_dimA[2] * img_dimA[3] * img_dimA[4], img_dimA[3] * img_dimA[4], img_dimA[4], 1};

            PLUGIN_CHECK_CUDNN(hipdnnSetTensorNdDescriptor(mBDescriptor, HIPDNN_DATA_FLOAT, 5, nc_dimA, nc_strideA));
            hipdnnDataType_t cudnn_dtype;
            PLUGIN_CHECK_CUDNN(convertTrt2cudnnDtype(inputDesc[0].type, &cudnn_dtype));
            PLUGIN_CHECK_CUDNN(hipdnnSetTensorNdDescriptor(mXDescriptor, cudnn_dtype, 5, img_dimA, img_strideA));
            PLUGIN_CHECK_CUDNN(hipdnnSetTensorNdDescriptor(mYDescriptor, cudnn_dtype, 5, img_dimA, img_strideA));
            float alpha = 1;
            float beta = 0;

            void const* x_ptr = inputs[0];
            void* y_ptr = outputs[0];
            // Note: Use of HIPDNN_BATCHNORM_SPATIAL_PERSISTENT can cause numerical
            //       overflows (NaNs) for fp32 data in some circumstances. The lower-
            //       performance HIPDNN_BATCHNORM_SPATIAL should be used if this is not
            //       acceptable.
            PLUGIN_CHECK_CUDNN(hipdnnBatchNormalizationForwardTraining(mCudnnHandle, HIPDNN_BATCHNORM_SPATIAL_PERSISTENT,
                &alpha, &beta, mXDescriptor, x_ptr, mYDescriptor, y_ptr, mBDescriptor, d_scale, d_bias, 1., nullptr,
                nullptr, mEpsilon, nullptr, nullptr));

            callRelu(y_ptr, n * c * d * h * w, inputDesc[0].type);
        }
        else if (inputDesc[0].format == nvinfer1::PluginFormat::kDHWC8
            || inputDesc[0].format == nvinfer1::PluginFormat::kCDHW32)
        {
            int32_t input_data_type = (inputDesc[0].type == nvinfer1::DataType::kHALF) ? 1 : 2;
            int32_t output_data_type = (outputDesc[0].type == nvinfer1::DataType::kHALF) ? 1 : 2;

            nvinfer1::Dims input_dims = inputDesc[0].dims;
            int32_t n = input_dims.d[0];
            int32_t c = input_dims.d[1];
            int32_t d = input_dims.d[2];
            int32_t h = input_dims.d[3];
            int32_t w = input_dims.d[4];

            InstanceNormFwdParams params;
            params.nhw = d * h * w;
            params.c = c;
            params.n = n;

            size_t size_sums, size_counts, size_retired_ctas;
            instanceNormBufferSizesDispatch(
                mContext, params, size_sums, size_counts, size_retired_ctas, input_data_type, output_data_type);

            size_t size_nc = n * c * sizeof(float);
            size_nc = ((size_nc + 256 - 1) / 256) * 256;

            char* d_buf = static_cast<char*>(workspace);

            params.gmem_sums = reinterpret_cast<GMEM_SUMS_TYPE*>(d_buf);
            d_buf += size_sums;
            params.gmem_counts = reinterpret_cast<int32_t*>(d_buf);
            d_buf += size_counts;
            params.gmem_retired_ctas = reinterpret_cast<int32_t*>(d_buf);
            d_buf += size_retired_ctas;
            params.gmem_running_mean = reinterpret_cast<float*>(d_buf);
            d_buf += size_nc;
            params.gmem_running_var = reinterpret_cast<float*>(d_buf);
            d_buf += size_nc;
            params.gmem_saved_mean = reinterpret_cast<float*>(d_buf);
            d_buf += size_nc;
            params.gmem_saved_var = reinterpret_cast<float*>(d_buf);
            d_buf += size_nc;

            params.gmem_src = inputs[0];
            params.gmem_dst = outputs[0];
            params.gmem_bias = mDeviceBias;
            params.gmem_scale = mDeviceScale;

            params.var_eps = mEpsilon;
            params.exp_avg_factor = 1.F; //(float)exp_avg_factor;
            params.use_relu = mRelu;     // use_relu;
            params.relu_alpha = mAlpha;  // relu_alpha;

            params.in_scale = inputDesc[0].scale;
            PLUGIN_ASSERT(outputDesc[0].scale != 0.F);
            params.out_scale = 1.F / outputDesc[0].scale;

            instanceNormFwdDispatch(mContext, params, stream, input_data_type, output_data_type);
        }
        else
        {
            PLUGIN_ASSERT(false && "Unexpected input format");
        }
    }
    return 0;
}

size_t InstanceNormalizationPlugin::getSerializationSize() const noexcept
{
    return (serialized_size(mEpsilon) + serialized_size(mNchan) + serialized_size(mHostScale)
        + serialized_size(mHostBias) + serialized_size(mRelu) + serialized_size(mAlpha));
}

void InstanceNormalizationPlugin::serialize(void* buffer) const noexcept
{
    serialize_value(&buffer, mEpsilon);
    serialize_value(&buffer, mNchan);
    serialize_value(&buffer, mHostScale);
    serialize_value(&buffer, mHostBias);
    serialize_value(&buffer, mRelu);
    serialize_value(&buffer, mAlpha);
}

bool InstanceNormalizationPlugin::supportsFormatCombination(
    int32_t pos, nvinfer1::PluginTensorDesc const* inOut, int32_t nbInputs, int32_t nbOutputs) noexcept
{
    PLUGIN_ASSERT(inOut && pos < (nbInputs + nbOutputs));
    PLUGIN_ASSERT(pos == 0 || pos == 1);

    // For 4-D or 3-D tensor (nbSpatialDims == 1 or 2), only FP32_Linear and FP16_Linear are supported.
    // For 5-D tensor (nbSpatialDims == 3), FP32_Linear, FP16_Linear, FP16_DHWC8, and INT8_CDHW32 are supported.
    // This is because we have special InstanceNorm3D kernels for vectorized formats from MLPerf-Inference.

    int32_t const nbDims = inOut[pos].dims.nbDims;
    PLUGIN_ASSERT(nbDims >= 3);
    PLUGIN_ASSERT(nbDims <= 5);
    bool const is3DInstanceNorm = (nbDims == 5);

    bool const isFP32Linear
        = (inOut[pos].type == nvinfer1::DataType::kFLOAT && inOut[pos].format == nvinfer1::PluginFormat::kLINEAR
            && inOut[pos].type == inOut[0].type && inOut[pos].format == inOut[0].format);

    bool const isFP16Linear
        = (inOut[pos].type == nvinfer1::DataType::kHALF && inOut[pos].format == nvinfer1::PluginFormat::kLINEAR
            && inOut[pos].type == inOut[0].type && inOut[pos].format == inOut[0].format);

    bool const isFP16DHWC8
        = (inOut[pos].type == nvinfer1::DataType::kHALF && inOut[pos].format == nvinfer1::PluginFormat::kDHWC8
            && inOut[pos].type == inOut[0].type && inOut[pos].format == inOut[0].format);

    bool const isINT8CDHW32
        = (inOut[pos].type == nvinfer1::DataType::kINT8 && inOut[pos].format == nvinfer1::PluginFormat::kCDHW32
            && inOut[pos].type == inOut[0].type && inOut[pos].format == inOut[0].format);

    bool const isFormatOK = isFP32Linear || isFP16Linear || (is3DInstanceNorm && (isFP16DHWC8 || isINT8CDHW32));

    // Kernels for vectorized formats only support the case of C % spv == 0.
    int32_t spv{1};
    switch (inOut[pos].format)
    {
    case nvinfer1::PluginFormat::kDHWC8: spv = 8; break;
    case nvinfer1::PluginFormat::kCDHW32: spv = 32; break;
    default: break;
    }
    int32_t const isAlignmentOK = (inOut[pos].dims.d[1] % spv == 0);

    return isFormatOK && isAlignmentOK;
}

char const* InstanceNormalizationPlugin::getPluginType() const noexcept
{
    return INSTANCE_PLUGIN_NAME;
}

char const* InstanceNormalizationPlugin::getPluginVersion() const noexcept
{
    return INSTANCE_PLUGIN_VERSION;
}

char const* InstanceNormalizationPluginV2::getPluginVersion() const noexcept
{
    return INSTANCE_PLUGIN_VERSION_V2;
}

void InstanceNormalizationPlugin::destroy() noexcept
{
    delete this;
}

template <class PluginType>
IPluginV2DynamicExt* InstanceNormalizationPlugin::cloneBase() const noexcept
{
    try
    {
        auto* plugin = new PluginType{mEpsilon, mHostScale, mHostBias, mRelu, mAlpha};
        plugin->setPluginNamespace(mPluginNamespace.c_str());
        plugin->initialize();
        return plugin;
    }
    catch (std::exception const& e)
    {
        caughtError(e);
    }
    return nullptr;
}

IPluginV2DynamicExt* InstanceNormalizationPlugin::clone() const noexcept
{
    return cloneBase<InstanceNormalizationPlugin>();
}

IPluginV2DynamicExt* InstanceNormalizationPluginV2::clone() const noexcept
{
    return cloneBase<InstanceNormalizationPluginV2>();
}

// Set plugin namespace
void InstanceNormalizationPlugin::setPluginNamespace(char const* pluginNamespace) noexcept
{
    mPluginNamespace = pluginNamespace;
}

char const* InstanceNormalizationPlugin::getPluginNamespace() const noexcept
{
    return mPluginNamespace.c_str();
}

nvinfer1::DataType InstanceNormalizationPlugin::getOutputDataType(
    int32_t index, nvinfer1::DataType const* inputTypes, int32_t nbInputs) const noexcept
{
    PLUGIN_ASSERT(inputTypes && nbInputs > 0 && index == 0);
    return inputTypes[0];
}

// Attach the plugin object to an execution context and grant the plugin the access to some context resource.
void InstanceNormalizationPlugin::attachToContext(
    cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator) noexcept
{
}

// Detach the plugin object from its execution context.
void InstanceNormalizationPlugin::detachFromContext() noexcept {}

void InstanceNormalizationPlugin::configurePlugin(nvinfer1::DynamicPluginTensorDesc const* in, int32_t nbInputs,
    nvinfer1::DynamicPluginTensorDesc const* out, int32_t nbOutputs) noexcept
{
    // Not support dynamic shape in C dimension
    PLUGIN_ASSERT(nbInputs == 1 && in[0].desc.dims.d[1] != -1);
}

// InstanceNormalizationPluginCreator methods
InstanceNormalizationPluginCreator::InstanceNormalizationPluginCreator()
{
    mPluginAttributes.clear();
    mPluginAttributes.emplace_back(PluginField("epsilon", nullptr, PluginFieldType::kFLOAT32, 1));
    mPluginAttributes.emplace_back(PluginField("scales", nullptr, PluginFieldType::kFLOAT32, 1));
    mPluginAttributes.emplace_back(PluginField("bias", nullptr, PluginFieldType::kFLOAT32, 1));
    mPluginAttributes.emplace_back(PluginField("relu", nullptr, PluginFieldType::kINT32, 1));
    mPluginAttributes.emplace_back(PluginField("alpha", nullptr, PluginFieldType::kFLOAT32, 1));

    mFC.nbFields = mPluginAttributes.size();
    mFC.fields = mPluginAttributes.data();
}

char const* InstanceNormalizationPluginCreator::getPluginName() const noexcept
{
    return INSTANCE_PLUGIN_NAME;
}

char const* InstanceNormalizationPluginCreator::getPluginVersion() const noexcept
{
    return INSTANCE_PLUGIN_VERSION;
}

char const* InstanceNormalizationPluginCreatorV2::getPluginVersion() const noexcept
{
    return INSTANCE_PLUGIN_VERSION_V2;
}

PluginFieldCollection const* InstanceNormalizationPluginCreator::getFieldNames() noexcept
{
    return &mFC;
}

template <class PluginType>
IPluginV2DynamicExt* InstanceNormalizationPluginCreator::createPluginBase(
    char const* name, nvinfer1::PluginFieldCollection const* fc) noexcept
{
    try
    {
        std::vector<float> scaleValues;
        std::vector<float> biasValues;
        float epsilon{};
        int32_t relu{};
        float alpha{};
        PluginField const* fields = fc->fields;
        for (int32_t i = 0; i < fc->nbFields; ++i)
        {
            char const* attrName = fields[i].name;
            if (!strcmp(attrName, "epsilon"))
            {
                PLUGIN_VALIDATE(fields[i].type == PluginFieldType::kFLOAT32);
                epsilon = *(static_cast<float const*>(fields[i].data));
            }
            else if (!strcmp(attrName, "scales"))
            {
                PLUGIN_VALIDATE(fields[i].type == PluginFieldType::kFLOAT32);
                int32_t size = fields[i].length;
                scaleValues.reserve(size);
                auto const* w = static_cast<float const*>(fields[i].data);
                for (int32_t j = 0; j < size; j++)
                {
                    scaleValues.push_back(*w);
                    w++;
                }
            }
            else if (!strcmp(attrName, "bias"))
            {
                PLUGIN_VALIDATE(fields[i].type == PluginFieldType::kFLOAT32);
                int32_t size = fields[i].length;
                biasValues.reserve(size);
                auto const* w = static_cast<float const*>(fields[i].data);
                for (int32_t j = 0; j < size; j++)
                {
                    biasValues.push_back(*w);
                    w++;
                }
            }
            else if (!strcmp(attrName, "relu"))
            {
                PLUGIN_VALIDATE(fields[i].type == PluginFieldType::kINT32);
                relu = *(static_cast<int32_t const*>(fields[i].data));
            }
            else if (!strcmp(attrName, "alpha"))
            {
                PLUGIN_VALIDATE(fields[i].type == PluginFieldType::kFLOAT32);
                alpha = *(static_cast<float const*>(fields[i].data));
            }
        }

        Weights scaleWeights{DataType::kFLOAT, scaleValues.data(), (int64_t) scaleValues.size()};
        Weights biasWeights{DataType::kFLOAT, biasValues.data(), (int64_t) biasValues.size()};

        auto* obj = new PluginType(epsilon, scaleWeights, biasWeights, relu, alpha);
        obj->setPluginNamespace(mNamespace.c_str());
        obj->initialize();
        return obj;
    }
    catch (std::exception const& e)
    {
        caughtError(e);
    }
    return nullptr;
}

IPluginV2DynamicExt* InstanceNormalizationPluginCreator::createPlugin(
    char const* name, nvinfer1::PluginFieldCollection const* fc) noexcept
{
    return createPluginBase<InstanceNormalizationPlugin>(name, fc);
}

IPluginV2DynamicExt* InstanceNormalizationPluginCreatorV2::createPlugin(
    char const* name, nvinfer1::PluginFieldCollection const* fc) noexcept
{
    return createPluginBase<InstanceNormalizationPluginV2>(name, fc);
}

template <class PluginType>
IPluginV2DynamicExt* InstanceNormalizationPluginCreator::deserializePluginBase(
    char const* name, void const* serialData, size_t serialLength) noexcept
{
    try
    {
        auto* obj = new PluginType{serialData, serialLength};
        obj->setPluginNamespace(mNamespace.c_str());
        obj->initialize();
        return obj;
    }
    catch (std::exception const& e)
    {
        caughtError(e);
    }
    return nullptr;
}

IPluginV2DynamicExt* InstanceNormalizationPluginCreator::deserializePlugin(
    char const* name, void const* serialData, size_t serialLength) noexcept
{
    return deserializePluginBase<InstanceNormalizationPlugin>(name, serialData, serialLength);
}

IPluginV2DynamicExt* InstanceNormalizationPluginCreatorV2::deserializePlugin(
    char const* name, void const* serialData, size_t serialLength) noexcept
{
    return deserializePluginBase<InstanceNormalizationPluginV2>(name, serialData, serialLength);
}
