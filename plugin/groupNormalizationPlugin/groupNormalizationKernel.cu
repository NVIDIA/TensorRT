#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "groupNormalizationPlugin.h"

namespace nvinfer1
{
namespace plugin
{

template <typename T, unsigned TPB>
__global__ void scaleShiftChannelsInplaceKernel(T* inOut, const int ld, const float* beta, const float* gamma)
{
    // grid is blocks x C x B
    // ld should be H*W
    // blockIdx.z = batch
    // blockIdx.y = channel
    // blockIdx.x = block per col
    const T b = beta[blockIdx.y];
    const T g = gamma[blockIdx.y];

    const int offset = (blockIdx.z * gridDim.y + blockIdx.y) * ld;

    const int tx = blockIdx.x * TPB + threadIdx.x;

    if (tx < ld)
    {
        inOut[offset + tx] = g * inOut[offset + tx] + b;
    }
}

template <typename T>
void scaleShiftChannelsInplace(T* inOut, const int B, const int C, const int channelVolume, const float* beta,
    const float* gamma, hipStream_t stream)
{

    constexpr int TPB = 256;
    const int colBlocks = (channelVolume + TPB - 1) / TPB;
    const dim3 grid(colBlocks, C, B);

    scaleShiftChannelsInplaceKernel<T, TPB><<<grid, TPB, 0, stream>>>(inOut, channelVolume, beta, gamma);

    CUASSERT(hipPeekAtLastError());
}

template void scaleShiftChannelsInplace<float>(float* inOut, const int B, const int C, const int channelVolume, const float* beta,
    const float* gamma, hipStream_t stream);
} /* plugin */
} /* nvinfer1 */
