#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 1993-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "common/kernel.h"

#define CUBLAS_CHECK(condition)                                                                                        \
    do                                                                                                                 \
    {                                                                                                                  \
        hipblasStatus_t status = condition;                                                                             \
        if (status != HIPBLAS_STATUS_SUCCESS)                                                                           \
        {                                                                                                              \
            printf("%s %d CUBLAS FAIL %s\n", __FILE__, __LINE__, cublasGetErrorString(status));                        \
        }                                                                                                              \
    } while (0)

// this scatter kernel works on a 2d table writing rows
// index is 1-D array
// updates is 2-D array
// output is 2-D array
// output[index[i]] = updates[i]
__global__ void scatterKernel(
    char* output,
    const char* updates,
    const int* indices,
    int pitch,
    int rowSize)
{
    int idx = indices[blockIdx.x];
    char* pDst = (char*)output + idx * pitch;
    const char* pSrc = updates + blockIdx.x * rowSize;
    memcpy(pDst, pSrc, rowSize);
}

// Transform nd index to 1 - d index
__global__ void transformIdxKernel(
    int* output,
    const int* transformCoeff, // these are actually the output pitches of the respective dimensions
    const int* indices,
    int sliceRank)
{
    const int* idx = indices + sliceRank * blockIdx.x;
    int transformedIdx = 0;
    for (int i = 0; i < sliceRank; i++)
    {
        transformedIdx += idx[i] * transformCoeff[i];
    }
    output[blockIdx.x] = transformedIdx;
}


pluginStatus_t scatterNDInference(
    hipStream_t stream,
    int* transformCoeff,
    int nOutputDims,
    int sliceRank,
    int nRows,
    int rowSize,
    int copySize,
    int sizeOfElementInBytes,
    const void* index,
    const void* updates,
    const void* data,
    void* output,
    void* workspace)
{
    const int* _index = (const int*)(index);
    const char* _updates = (const char*)(updates);
    char* _output = (char*)(output);
    int* wo = (int*)(workspace);
    int* transformedIdx = wo + sizeof(int)*nOutputDims;
    int* deviceTransformCoeff = wo;
    CSC(hipMemcpy(workspace, transformCoeff, sizeof(int) * nOutputDims, hipMemcpyHostToDevice), STATUS_FAILURE);
    transformIdxKernel<<<nRows, 1, 0, stream>>>(transformedIdx, deviceTransformCoeff, _index, sliceRank);
    CSC(hipMemcpy(output, data, copySize, hipMemcpyDeviceToDevice), STATUS_FAILURE);
    // assuming output pitch = rowSize i.e no padding
    scatterKernel<<<nRows, 1, 0, stream>>>(_output, _updates, transformedIdx, rowSize * 4, rowSize * 4);
    return STATUS_SUCCESS;
}