#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 1993-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "clipPlugin/clip.h"
#include "common/checkMacrosPlugin.h"
#include <hip/hip_fp16.h>

// Integer division rounding up
inline __host__ __device__ constexpr int divUp(int x, int n)
{
    return (x + n - 1) / n;
}

template <typename T1, typename T2, unsigned nthdsPerCTA>
__launch_bounds__(nthdsPerCTA)
    __global__ void clipKernel(
        int n,
        const T1 clipMin,
        const T1 clipMax,
        const T2* input,
        T2* output)
{
    // each global thread handles one element
    int i = blockIdx.x * nthdsPerCTA + threadIdx.x;
    if (i < n)
    {
        T1 inputElement = static_cast<T1>(input[i]);
        T1 tmp = inputElement > clipMin ? inputElement : clipMin;
        output[i] = static_cast<T2>(tmp < clipMax ? tmp : clipMax);
    }
}

int clipInference(
    hipStream_t stream,
    int n,
    float clipMin,
    float clipMax,
    const void* input,
    void* output,
    nvinfer1::DataType type)
{
    const int BS = 512;
    const int GS = divUp(n, BS);

    switch (type)
    {
    case nvinfer1::DataType::kFLOAT:
    {
        clipKernel<float, float, BS><<<GS, BS, 0, stream>>>(n, clipMin, clipMax,
                                                            static_cast<const float*>(input),
                                                            static_cast<float*>(output));
        break;
    }
    case nvinfer1::DataType::kHALF:
    {
        /* Implementing kHALF operation using float operands. function __float2half
         * is not supported for CUDA versions <= 9.1 causing compilation failures
         * Moreover operand > for __half operand is only supported if __CUDA_ARCH__
         * >= 530
         */
        clipKernel<float, half, BS><<<GS, BS, 0, stream>>>(n,
                                                           clipMin, clipMax,
                                                           static_cast<const half*>(input),
                                                           static_cast<half*>(output));
        break;
    }
    case nvinfer1::DataType::kINT32:
    {
        clipKernel<int32_t, int32_t, BS><<<GS, BS, 0, stream>>>(n,
                                                                static_cast<int32_t>(clipMin), static_cast<int32_t>(clipMax),
                                                                static_cast<const int32_t*>(input),
                                                                static_cast<int32_t*>(output));
        break;
    }
    case nvinfer1::DataType::kINT8:
    {
        clipKernel<int8_t, int8_t, BS><<<GS, BS, 0, stream>>>(n,
                                                              static_cast<int8_t>(clipMin), static_cast<int8_t>(clipMax),
                                                              static_cast<const int8_t*>(input),
                                                              static_cast<int8_t*>(output));
        break;
    }
    case nvinfer1::DataType::kUINT8:
    {
        PLUGIN_FAIL("unsupported datatype");
        break;
    }
    case nvinfer1::DataType::kBOOL:
    {
        PLUGIN_FAIL("unsupported datatype");
        break;
    }
    }

    return 0;
}
