#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 1993-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "coordConvACPlugin.h"
#include <hip/hip_fp16.h>

template <typename T_DATA>
__global__ void kernelCopy(int N, T_DATA* inputs, T_DATA* outputs)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
    {
        outputs[index] = inputs[index];
    }
    __syncthreads();
}

template <typename T_DATA>
__global__ void kernelAC(int N, int iH, int iW, float stepACh, float stepACw, T_DATA* outputs)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int channelLength = N / 2;

    if (index < channelLength)
    {
        outputs[index] = -1.0 + (float) (index / iW) * stepACw;
        outputs[index + channelLength] = -1.0 + (float) ((index + channelLength) % iH) * stepACh;
    }
    __syncthreads();
}

template <typename T>
int inferenceAC(
    int batchSize, int iC, int iH, int iW, int oC, int oH, int oW, T* inputs, T* outputs, hipStream_t stream)
{
    // NCHW
    const float coordsRange = 2.0;
    const int nThreads = 512;
    int lenCopy = iC * iH * iW;
    int lenAC = (oC * oH * oW) - lenCopy;

    int nBlocksCopy = (int) ((float) lenCopy / nThreads) + 1;
    int nBlocksAC = (int) ((float) lenAC / nThreads) + 1;

    float stepACh = coordsRange / (float) (iH - 1);
    float stepACw = coordsRange / (float) (iW - 1);

    for (int i = 0; i < batchSize; ++i)
    {
        // NOTE: kernelCopy kernel can be replaced with hipMemcpy function
        kernelCopy<<<nBlocksCopy, nThreads, 0, stream>>>(lenCopy, inputs, outputs);
        outputs += lenCopy;

        kernelAC<<<nBlocksAC, nThreads, 0, stream>>>(lenAC, iH, iW, stepACh, stepACw, outputs);
        outputs += lenAC;
        inputs += lenCopy;
    }

    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n", __FILE__, __LINE__, hipGetErrorString(err));
        return 1;
    }
    return 0;
}

int CoordConvACPlugin::enqueue(
    int batchSize, const void* const* inputs, void* const* outputs, void* workspace, hipStream_t stream) noexcept
{
    switch (iType)
    {
    case DataType::kFLOAT:
        return inferenceAC(batchSize, iC, iH, iW, oC, oH, oW, (float*) inputs[0], (float*) outputs[0], stream);
    case DataType::kHALF:
        return inferenceAC(batchSize, iC, iH, iW, oC, oH, oW, (__half*) inputs[0], (__half*) outputs[0], stream);
    case DataType::kINT8:
    case DataType::kINT32:
    case DataType::kBOOL:
        break;
    }
    return 1;
}
