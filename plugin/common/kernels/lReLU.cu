#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "kernel.h"

template <unsigned nthdsPerCTA>
__launch_bounds__(nthdsPerCTA)
    __global__ void pReLUKernel(
        const int n,
        const float negativeSlope,
        const float* input,
        float* output)
{
    for (int i = blockIdx.x * nthdsPerCTA + threadIdx.x; i < n; i += gridDim.x * nthdsPerCTA)
    {
        output[i] = input[i] > 0 ? input[i] : input[i] * negativeSlope;
    }
}

pluginStatus_t lReLUGPU(
    hipStream_t stream,
    const int n,
    const float negativeSlope,
    const void* input,
    void* output)
{
    const int BS = 512;
    const int GS = (n + BS - 1) / BS;
    pReLUKernel<BS><<<GS, BS, 0, stream>>>(n, negativeSlope,
                                           (const float*) input,
                                           (float*) output);
    return STATUS_SUCCESS;
}

pluginStatus_t lReLUInference(
    hipStream_t stream,
    const int n,
    const float negativeSlope,
    const void* input,
    void* output)
{
    return lReLUGPU(stream, n, negativeSlope, (const float*) input, (float*) output);
}
