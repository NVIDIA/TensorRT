#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "kernel.h"
#include "bboxUtils.h"
#include "hip/hip_fp16.h"
#include <array>

template <typename T_BBOX>
__device__ float bboxSize(
    const Bbox<T_BBOX>& bbox,
    const bool normalized)
{
    if (float(bbox.xmax) < float(bbox.xmin) || float(bbox.ymax) < float(bbox.ymin))
    {
        // If bbox is invalid (e.g. xmax < xmin or ymax < ymin), return 0.
        return 0;
    }
    else
    {
        float width = float(bbox.xmax) - float(bbox.xmin);
        float height = float(bbox.ymax) - float(bbox.ymin);
        if (normalized)
        {
            return width * height;
        }
        else
        {
            // If bbox is not within range [0, 1].
            return (width + 1.f) * (height + 1.f);
        }
    }
}

template <typename T_BBOX>
__device__ void intersectBbox(
    const Bbox<T_BBOX>& bbox1,
    const Bbox<T_BBOX>& bbox2,
    Bbox<T_BBOX>* intersect_bbox)
{
    if (bbox2.xmin > bbox1.xmax || bbox2.xmax < bbox1.xmin || bbox2.ymin > bbox1.ymax || bbox2.ymax < bbox1.ymin)
    {
        // Return [0, 0, 0, 0] if there is no intersection.
        intersect_bbox->xmin = T_BBOX(0);
        intersect_bbox->ymin = T_BBOX(0);
        intersect_bbox->xmax = T_BBOX(0);
        intersect_bbox->ymax = T_BBOX(0);
    }
    else
    {
        intersect_bbox->xmin = max(bbox1.xmin, bbox2.xmin);
        intersect_bbox->ymin = max(bbox1.ymin, bbox2.ymin);
        intersect_bbox->xmax = min(bbox1.xmax, bbox2.xmax);
        intersect_bbox->ymax = min(bbox1.ymax, bbox2.ymax);
    }
}


template <>
__device__ void intersectBbox<__half>(
    const Bbox<__half>& bbox1,
    const Bbox<__half>& bbox2,
    Bbox<__half>* intersect_bbox)
{
    if (float(bbox2.xmin) > float(bbox1.xmax)
        || float(bbox2.xmax) < float(bbox1.xmin)
        || float(bbox2.ymin) > float(bbox1.ymax)
        || float(bbox2.ymax) < float(bbox1.ymin))
    {
        // Return [0, 0, 0, 0] if there is no intersection.
        intersect_bbox->xmin = __half(0);
        intersect_bbox->ymin = __half(0);
        intersect_bbox->xmax = __half(0);
        intersect_bbox->ymax = __half(0);
    }
    else
    {
        intersect_bbox->xmin = max(float(bbox1.xmin), float(bbox2.xmin));
        intersect_bbox->ymin = max(float(bbox1.ymin), float(bbox2.ymin));
        intersect_bbox->xmax = min(float(bbox1.xmax), float(bbox2.xmax));
        intersect_bbox->ymax = min(float(bbox1.ymax), float(bbox2.ymax));
    }
}


template <typename T_BBOX>
__device__ Bbox<T_BBOX> getDiagonalMinMaxSortedBox(const Bbox<T_BBOX>& bbox1)
{
    Bbox<T_BBOX> result;
    result.xmin = min(bbox1.xmin, bbox1.xmax);
    result.xmax = max(bbox1.xmin, bbox1.xmax);

    result.ymin = min(bbox1.ymin, bbox1.ymax);
    result.ymax = max(bbox1.ymin, bbox1.ymax);
    return result;
}

template <>
__device__ Bbox<__half> getDiagonalMinMaxSortedBox(const Bbox<__half>& bbox1)
{
    Bbox<__half> result;
    result.xmin = min(float(bbox1.xmin), float(bbox1.xmax));
    result.xmax = max(float(bbox1.xmin), float(bbox1.xmax));

    result.ymin = min(float(bbox1.ymin), float(bbox1.ymax));
    result.ymax = max(float(bbox1.ymin), float(bbox1.ymax));
    return result;
}

template <typename T_BBOX>
__device__ float jaccardOverlap(
    const Bbox<T_BBOX>& bbox1,
    const Bbox<T_BBOX>& bbox2,
    const bool normalized)
{
    Bbox<T_BBOX> intersect_bbox;

    Bbox<T_BBOX> localbbox1 = getDiagonalMinMaxSortedBox(bbox1);
    Bbox<T_BBOX> localbbox2 = getDiagonalMinMaxSortedBox(bbox2);

    intersectBbox(localbbox1, localbbox2, &intersect_bbox);
    float intersect_width, intersect_height;
    if (normalized)
    {
        intersect_width = float(intersect_bbox.xmax) - float(intersect_bbox.xmin);
        intersect_height = float(intersect_bbox.ymax) - float(intersect_bbox.ymin);
    }
    else
    {
        intersect_width = float(intersect_bbox.xmax) - float(intersect_bbox.xmin) + float(T_BBOX(1));
        intersect_height = float(intersect_bbox.ymax) - float(intersect_bbox.ymin) + float(T_BBOX(1));
    }
    if (intersect_width > 0 && intersect_height > 0)
    {
        float intersect_size = intersect_width * intersect_height;
        float bbox1_size = bboxSize(localbbox1, normalized);
        float bbox2_size = bboxSize(localbbox2, normalized);
        return intersect_size / (bbox1_size + bbox2_size - intersect_size);
    }
    else
    {
        return 0.;
    }
}

template <typename T_BBOX>
__device__ void emptyBboxInfo(
    BboxInfo<T_BBOX>* bbox_info)
{
    bbox_info->conf_score = T_BBOX(0);
    bbox_info->label = -2; // -1 is used for all labels when shared_location is ture
    bbox_info->bbox_idx = -1;
    bbox_info->kept = false;
}
/********** new NMS for only score and index array **********/

template <typename T_SCORE, typename T_BBOX, int TSIZE>
__global__ void allClassNMS_kernel(
    const int num,
    const int num_classes,
    const int num_preds_per_class,
    const int top_k,
    const float nms_threshold,
    const bool share_location,
    const bool isNormalized,
    T_BBOX* bbox_data, // bbox_data should be float to preserve location information
    T_SCORE* beforeNMS_scores,
    int* beforeNMS_index_array,
    T_SCORE* afterNMS_scores,
    int* afterNMS_index_array,
    bool flipXY,
    const float score_shift)
{
    //__shared__ bool kept_bboxinfo_flag[CAFFE_CUDA_NUM_THREADS * TSIZE];
    extern __shared__ bool kept_bboxinfo_flag[];

    for (int i = 0; i < num; i++)
    {
        const int offset = i * num_classes * num_preds_per_class + blockIdx.x * num_preds_per_class;
        const int max_idx = offset + top_k; // put top_k bboxes into NMS calculation
        const int bbox_idx_offset = share_location ? (i * num_preds_per_class) : (i * num_classes * num_preds_per_class);

        // local thread data
        int loc_bboxIndex[TSIZE];
        Bbox<T_BBOX> loc_bbox[TSIZE];

// initialize Bbox, Bboxinfo, kept_bboxinfo_flag
        // Eliminate shared memory RAW hazard
        __syncthreads();
#pragma unroll
        for (int t = 0; t < TSIZE; t++)
        {
            const int cur_idx = threadIdx.x + blockDim.x * t;
            const int item_idx = offset + cur_idx;

            if (item_idx < max_idx)
            {
                loc_bboxIndex[t] = beforeNMS_index_array[item_idx];

                if (loc_bboxIndex[t] != -1)
                {
                    const int bbox_data_idx = share_location ? (loc_bboxIndex[t] % num_preds_per_class + bbox_idx_offset) : loc_bboxIndex[t];

                    loc_bbox[t].xmin = flipXY ? bbox_data[bbox_data_idx * 4 + 1] : bbox_data[bbox_data_idx * 4 + 0];
                    loc_bbox[t].ymin = flipXY ? bbox_data[bbox_data_idx * 4 + 0] : bbox_data[bbox_data_idx * 4 + 1];
                    loc_bbox[t].xmax = flipXY ? bbox_data[bbox_data_idx * 4 + 3] : bbox_data[bbox_data_idx * 4 + 2];
                    loc_bbox[t].ymax = flipXY ? bbox_data[bbox_data_idx * 4 + 2] : bbox_data[bbox_data_idx * 4 + 3];
                    kept_bboxinfo_flag[cur_idx] = true;
                }
                else
                {
                    kept_bboxinfo_flag[cur_idx] = false;
                }
            }
            else
            {
                kept_bboxinfo_flag[cur_idx] = false;
            }
        }

        // filter out overlapped boxes with lower scores
        int ref_item_idx = offset;
        int ref_bbox_idx = share_location ? (beforeNMS_index_array[ref_item_idx] % num_preds_per_class + bbox_idx_offset) : beforeNMS_index_array[ref_item_idx];

        while ((ref_bbox_idx != -1) && ref_item_idx < max_idx)
        {
            Bbox<T_BBOX> ref_bbox;
            ref_bbox.xmin = flipXY ? bbox_data[ref_bbox_idx * 4 + 1] : bbox_data[ref_bbox_idx * 4 + 0];
            ref_bbox.ymin = flipXY ? bbox_data[ref_bbox_idx * 4 + 0] : bbox_data[ref_bbox_idx * 4 + 1];
            ref_bbox.xmax = flipXY ? bbox_data[ref_bbox_idx * 4 + 3] : bbox_data[ref_bbox_idx * 4 + 2];
            ref_bbox.ymax = flipXY ? bbox_data[ref_bbox_idx * 4 + 2] : bbox_data[ref_bbox_idx * 4 + 3];

            // Eliminate shared memory RAW hazard
            __syncthreads();

            for (int t = 0; t < TSIZE; t++)
            {
                const int cur_idx = threadIdx.x + blockDim.x * t;
                const int item_idx = offset + cur_idx;

                if ((kept_bboxinfo_flag[cur_idx]) && (item_idx > ref_item_idx))
                {
                    // TODO: may need to add bool normalized as argument, HERE true means normalized
                    if (jaccardOverlap(ref_bbox, loc_bbox[t], isNormalized) > nms_threshold)
                    {
                        kept_bboxinfo_flag[cur_idx] = false;
                    }
                }
            }
            __syncthreads();

            do
            {
                ref_item_idx++;
            } while (ref_item_idx < max_idx && !kept_bboxinfo_flag[ref_item_idx - offset]);

            ref_bbox_idx = share_location ? (beforeNMS_index_array[ref_item_idx] % num_preds_per_class + bbox_idx_offset) : beforeNMS_index_array[ref_item_idx];
        }

        // store data
        for (int t = 0; t < TSIZE; t++)
        {
            const int cur_idx = threadIdx.x + blockDim.x * t;
            const int read_item_idx = offset + cur_idx;
            const int write_item_idx = (i * num_classes * top_k + blockIdx.x * top_k) + cur_idx;
            /*
             * If not not keeping the bbox
             * Set the score to 0
             * Set the bounding box index to -1
             */
            if (read_item_idx < max_idx)
            {
                afterNMS_scores[write_item_idx] = kept_bboxinfo_flag[cur_idx] ? T_SCORE(beforeNMS_scores[read_item_idx]) : T_SCORE(score_shift);
                afterNMS_index_array[write_item_idx] = kept_bboxinfo_flag[cur_idx] ? loc_bboxIndex[t] : -1;
            }
        }
    }
}

template <typename T_SCORE, typename T_BBOX>
pluginStatus_t allClassNMS_gpu(
    hipStream_t stream,
    const int num,
    const int num_classes,
    const int num_preds_per_class,
    const int top_k,
    const float nms_threshold,
    const bool share_location,
    const bool isNormalized,
    void* bbox_data,
    void* beforeNMS_scores,
    void* beforeNMS_index_array,
    void* afterNMS_scores,
    void* afterNMS_index_array,
    bool flipXY,
    const float score_shift)
{
#define P(tsize) allClassNMS_kernel<T_SCORE, T_BBOX, (tsize)>

    void (*kernel[8])(const int, const int, const int, const int, const float,
                      const bool, const bool, T_BBOX*, T_SCORE*, int*, T_SCORE*,
                      int*, bool, const float)
        = {
            P(1), P(2), P(3), P(4), P(5), P(6), P(7), P(8),
        };

    const int BS = 512;
    const int GS = num_classes;
    const int t_size = (top_k + BS - 1) / BS;

    kernel[t_size - 1]<<<GS, BS, BS * t_size * sizeof(bool), stream>>>(num, num_classes, num_preds_per_class,
                                                                       top_k, nms_threshold, share_location, isNormalized,
                                                                       (T_BBOX*) bbox_data,
                                                                       (T_SCORE*) beforeNMS_scores,
                                                                       (int*) beforeNMS_index_array,
                                                                       (T_SCORE*) afterNMS_scores,
                                                                       (int*) afterNMS_index_array,
                                                                       flipXY,
                                                                       score_shift);

    CSC(hipGetLastError(), STATUS_FAILURE);
    return STATUS_SUCCESS;
}

// allClassNMS LAUNCH CONFIG
typedef pluginStatus_t (*nmsFunc)(hipStream_t,
                               const int,
                               const int,
                               const int,
                               const int,
                               const float,
                               const bool,
                               const bool,
                               void*,
                               void*,
                               void*,
                               void*,
                               void*,
                               bool,
                               const float);

struct nmsLaunchConfigSSD
{
    DataType t_score;
    DataType t_bbox;
    nmsFunc function;

    nmsLaunchConfigSSD(DataType t_score, DataType t_bbox)
        : t_score(t_score)
        , t_bbox(t_bbox)
    {
    }
    nmsLaunchConfigSSD(DataType t_score, DataType t_bbox, nmsFunc function)
        : t_score(t_score)
        , t_bbox(t_bbox)
        , function(function)
    {
    }
    bool operator==(const nmsLaunchConfigSSD& other)
    {
        return t_score == other.t_score && t_bbox == other.t_bbox;
    }
};

static std::array<nmsLaunchConfigSSD, 2> nmsSsdLCOptions = {
    nmsLaunchConfigSSD(DataType::kFLOAT, DataType::kFLOAT, allClassNMS_gpu<float, float>),
    nmsLaunchConfigSSD(DataType::kHALF, DataType::kHALF, allClassNMS_gpu<__half, __half>)
};

pluginStatus_t allClassNMS(hipStream_t stream,
                        const int num,
                        const int num_classes,
                        const int num_preds_per_class,
                        const int top_k,
                        const float nms_threshold,
                        const bool share_location,
                        const bool isNormalized,
                        const DataType DT_SCORE,
                        const DataType DT_BBOX,
                        void* bbox_data,
                        void* beforeNMS_scores,
                        void* beforeNMS_index_array,
                        void* afterNMS_scores,
                        void* afterNMS_index_array,
                        bool flipXY,
                        const float score_shift)
{
    nmsLaunchConfigSSD lc = nmsLaunchConfigSSD(DT_SCORE, DT_BBOX);
    for (unsigned i = 0; i < nmsSsdLCOptions.size(); ++i)
    {
        if (lc == nmsSsdLCOptions[i])
        {
            DEBUG_PRINTF("all class nms kernel %d\n", i);
            return nmsSsdLCOptions[i].function(stream,
                                          num,
                                          num_classes,
                                          num_preds_per_class,
                                          top_k,
                                          nms_threshold,
                                          share_location,
                                          isNormalized,
                                          bbox_data,
                                          beforeNMS_scores,
                                          beforeNMS_index_array,
                                          afterNMS_scores,
                                          afterNMS_index_array,
                                          flipXY,
                                          score_shift);
        }
    }
    return STATUS_BAD_PARAM;
}
