#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include "maskRCNNKernels.h"
#include "plugin.h"
#include <NvInfer.h>
#include <assert.h>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <stdio.h>
#include <thrust/device_ptr.h>
#include <thrust/fill.h>

#define DUBUG_KERNEL 0
#define DUBUG_BATCH 0
#define DEBUG_T 1

#define dMIN(a, b) ((a) < (b) ? (a) : (b))
#define dMAX(a, b) ((a) > (b) ? (a) : (b))
#define dCLAMP(x, xMin, xMax) ((x) > (xMin) ? ((x) < (xMax) ? (x) : (xMax)) : (xMin))

template <typename BoxType>
struct BBoxT
{
    BoxType y1, x1, y2, x2;
};

template <typename DType>
__global__ void argMaxReset_kernel(
    int samples, int NClass, const DType* in_scores, const int* maxIdx, DType* out_scores)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int max_idx = samples * NClass;
    if (idx >= max_idx)
        return;

    int sampleIdx = idx / NClass;
    int classIdx = idx % NClass;
    if (classIdx != maxIdx[sampleIdx])
        out_scores[idx] = 0;
    else
        out_scores[idx] = in_scores[idx];
}

template <typename DType>
struct ScanItem
{
    DType data;
    int idx;
};

template <typename DType>
struct GreaterItem
{
    __host__ __device__ __forceinline__ ScanItem<DType> operator()(
        const ScanItem<DType>& a, const ScanItem<DType>& b) const
    {
        return (a.data > b.data ? a : b);
    }
};

template <typename DType>
__global__ void resetMemValue_kernel(void* outPtr, int samples, float val)
{
    DType* out = static_cast<DType*>(outPtr);
    int loop = gridDim.x * blockDim.x;
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < samples; idx += loop)
    {
        out[idx] = (DType) val;
    }
}

// blockDim.x : NClass
// GroupDim.x : sample count
// GroupDim.y : batch N
// outScore : DType[ N * sample * 1 ]
// outLabel : int[ N * sample * 1 ]
// outBbox : int[ N * sample * 4 ]
template <typename DType, typename BoxType, int Threads = 32>
__global__ void argMaxGroup_kernel(int samples, int start_class_id, int NClass, const void* inScorePtr, const void* inBboxPtr,
    const void* validSampleCountPtr, void* outScorePtr, void* outLabelPtr, void* outBboxPtr)
{
    const DType* inScore = static_cast<const DType*>(inScorePtr);
    const BoxType* inBbox = static_cast<const BoxType*>(inBboxPtr);
    const int* validSampleCount = static_cast<const int*>(validSampleCountPtr);
    DType* outScore = static_cast<DType*>(outScorePtr);
    BoxType* outLabel = static_cast<BoxType*>(outLabelPtr);
    BoxType* outBbox = static_cast<BoxType*>(outBboxPtr);

    const int N = blockIdx.y;
    const int validSamples = validSampleCount[N];

    typedef ScanItem<DType> ScanItemD;
    typedef hipcub::BlockReduce<ScanItemD, Threads> BlockReduce;

    __shared__ typename BlockReduce::TempStorage temp_storage;

    for (int iSample = blockIdx.x; iSample < validSamples; iSample += gridDim.x)
    {
        int classOffset = (N * samples + iSample) * NClass; // start from [batch, count, class0]
        // total IPerThread * blockDim
        ScanItemD maxItem = {0.0f, -1};
        for (int i = start_class_id; i < NClass; i += Threads)
        {
            int curIdx = i + threadIdx.x;
            ScanItemD item = {0.0f, -1};
            if (curIdx < NClass)
            {
                item.data = inScore[classOffset + curIdx];
                item.idx = curIdx;
            }
            const int validNum = (NClass - i > Threads ? Threads : NClass - i);
            ScanItemD aggregate = BlockReduce(temp_storage).Reduce(item, GreaterItem<DType>(), validNum);
            __syncthreads();
            if (aggregate.data > maxItem.data)
            {
                maxItem = aggregate;
            }
#if DUBUG_KERNEL
            if (N == DUBUG_BATCH && threadIdx.x == 0 && iSample < 15 /*&& maxItem.idx >= 32*/)
            {
                printf("argMaxGroup N:%d, iSample:%d, maxItem(score:%.3f, idx:%d)validReduceNum:%d\n", N, iSample,
                    (float) maxItem.data, maxItem.idx, validNum);
            }
#endif
        }

        const int dstOffset = N * samples + iSample;
        if (threadIdx.x == 0)
        {
            outScore[dstOffset] = maxItem.data;
            outLabel[dstOffset] = (BoxType) maxItem.idx;
            outBbox[dstOffset * 4] = inBbox[(classOffset + maxItem.idx) * 4];
            outBbox[dstOffset * 4 + 1] = inBbox[(classOffset + maxItem.idx) * 4 + 1];
            outBbox[dstOffset * 4 + 2] = inBbox[(classOffset + maxItem.idx) * 4 + 2];
            outBbox[dstOffset * 4 + 3] = inBbox[(classOffset + maxItem.idx) * 4 + 3];
        }
    }
}

struct BlockClassSumPrefix
{
    int total;
    // Constructor
    __device__ BlockClassSumPrefix()
        : total(0)
    {
    }
    // Callback operator to be entered by the first warp of threads in the block.
    // Thread-0 is responsible for returning a value for seeding the block-wide scan.
    __device__ int operator()(int aggregate)
    {
        int old = total;
        total += aggregate;
        return old;
    }
};

#define LabelShift (DType)(2.5f)
#define MinValidScore (DType)(0.01f)

template <typename DType>
__device__ __forceinline__ DType getKey(DType score, int lable, int NClass)
{
    return (lable < 0 ? (DType) 0 : ((DType)(NClass - lable - 1) * LabelShift + score + MinValidScore));
}

template <typename DType, typename BoxType>
__device__ __forceinline__ void getScoreLable(DType key, int NClass, DType& score, BoxType& lable)
{
    int i = key / LabelShift;
    score = (key <= MinValidScore ? (DType) 0 : key - (DType) i * LabelShift - MinValidScore);
    score = dCLAMP(score, (DType) 0, (DType) 1.0);
    lable = (BoxType)(key <= MinValidScore ? -1 : (NClass - i - 1));
}

// blockDim.x : threads
// gridDim.x : batch N
// validSampleCount INPUT : int [N]
// classStartPos OUTPUT: int [N * (Class + 1)], need memset to zero before this kernel
// outScore OUTPUT : DType [N * samples]
// outLabel OUTPUT : int [N * samples]
// outSampleIdx OUTPUT : int [N * samples]
// outValidSampleCount : int [N]
// IPerThread * Threads >= sample-count
#define MaxClassNum 255
template <typename DType, typename BoxType, int Threads = 256, int IPerThread = 4>
__global__ void sortPerClass_kernel(
    // int N,
    int samples, int NClass, int background, float scoreThreshold, const void* validSampleCountPtr,
    const void* inScorePtr, const void* inLabelPtr, const void* inBboxPtr, void* classStartPosPtr, void* outScorePtr,
    void* outLabelPtr, void* outSampleIdxPtr, void* outValidSampleCountPtr)
{
    typedef hipcub::BlockExchange<DType, Threads, IPerThread> BlockExchangeKey;
    typedef hipcub::BlockExchange<int, Threads, IPerThread> BlockExchangeI;
    typedef hipcub::BlockRadixSort<DType, Threads, IPerThread, int> BlockRadixSort;
    typedef hipcub::BlockScan<int, Threads> BlockScanClass;
    __shared__ union
    {
        typename BlockExchangeKey::TempStorage storageKey;
        typename BlockExchangeI::TempStorage storageI;
        typename BlockRadixSort::TempStorage storageSort;
        typename BlockScanClass::TempStorage storageScan;
    } temp_storage;
    __shared__ int smemClassCount[MaxClassNum];
    assert(NClass < MaxClassNum);
    assert(IPerThread * Threads >= samples);

    const int* validSampleCount = static_cast<const int*>(validSampleCountPtr);
    const DType* inScore = static_cast<const DType*>(inScorePtr);
    const BoxType* inLabel = static_cast<const BoxType*>(inLabelPtr);
    int* classStartPos = static_cast<int*>(classStartPosPtr);
    DType* outScore = static_cast<DType*>(outScorePtr);
    BoxType* outLabel = static_cast<BoxType*>(outLabelPtr);
    int* outSampleIdx = static_cast<int*>(outSampleIdxPtr);
    int* outValidSampleCount = static_cast<int*>(outValidSampleCountPtr);

    for (int s = threadIdx.x; s < NClass + 1; s += blockDim.x)
    {
        smemClassCount[s] = 0;
    }

    int N = blockIdx.x;
    int blockOffset = N * samples;
    int validSamples = validSampleCount[N];
    DType key[IPerThread];
    int iSample[IPerThread];
    for (int i = 0; i < IPerThread; ++i)
    {
        iSample[i] = -1;
        key[i] = -1.0f;
        int curIdx = i * Threads + threadIdx.x;
        if (curIdx < validSamples)
        {
            int label = (int) (inLabel[blockOffset + curIdx]);
            DType score = inScore[blockOffset + curIdx];
            if (label != background && label != -1 && score >= (DType) scoreThreshold)
            {
                key[i] = getKey(score, label, NClass);
                iSample[i] = curIdx;
            }
        }
    }

    BlockExchangeKey(temp_storage.storageKey).StripedToBlocked(key);
    __syncthreads();
    BlockExchangeI(temp_storage.storageI).StripedToBlocked(iSample);
    __syncthreads();
    BlockRadixSort(temp_storage.storageSort).SortDescendingBlockedToStriped(key, iSample);
    __syncthreads();

    // store Idx
    hipcub::StoreDirectStriped<Threads>(threadIdx.x, outSampleIdx + blockOffset, iSample, validSamples);
    BoxType lable[IPerThread];
    DType score[IPerThread];

#pragma unroll
    for (int i = 0; i < IPerThread; ++i)
    {
        getScoreLable(key[i], NClass, score[i], lable[i]);
    }
    hipcub::StoreDirectStriped<Threads>(threadIdx.x, outScore + blockOffset, score, validSamples);
    hipcub::StoreDirectStriped<Threads>(threadIdx.x, outLabel + blockOffset, lable, validSamples);

    // final
    for (int i = 0; i < IPerThread; ++i)
    {
        if (lable[i] >= (BoxType) 0)
        {
            atomicAdd(&smemClassCount[(int) lable[i]], 1);
        }
    }
    __syncthreads();

    int classBlockOffset = N * (NClass + 1); // Exclusive-sum, 1st is 0, last is final sum

#if DUBUG_KERNEL
    if (N == DUBUG_BATCH && threadIdx.x == 0)
    {
        printf("sortPerClass(N:%d) final count of each label, valid samples:%d\n", N, validSamples);
        for (int k = 0; k < NClass; ++k)
        {
            if (smemClassCount[k] > 0)
                printf("Batch:%d, L:%d, count:%d, \n", N, k, smemClassCount[k]);
        }
    }
    __syncthreads();
#endif

    BlockClassSumPrefix sumPrefix;
    for (int s = 0; s < NClass; s += blockDim.x)
    { // s start from block
        int iClassSamples = 0;
        int iClass = s + threadIdx.x;
        if (iClass < NClass)
        {
            iClassSamples = smemClassCount[iClass];
        }
        BlockScanClass(temp_storage.storageScan).ExclusiveSum(iClassSamples, iClassSamples, sumPrefix);
        __syncthreads();
        if (iClass < NClass)
        {
            classStartPos[classBlockOffset + iClass] = iClassSamples;
        }
    }
    if (threadIdx.x == 0)
    {
        classStartPos[classBlockOffset + NClass] = sumPrefix.total;
        assert(sumPrefix.total <= validSamples); // background data removed.
        outValidSampleCount[N] = sumPrefix.total;
#if DUBUG_KERNEL
        if (N == DUBUG_BATCH)
            printf("After sortPerClass, batch:%d valid samples total:%d\n", N, sumPrefix.total);
#endif
    }
}

template <typename DType>
__device__ __forceinline__ BBoxT<DType> readBbox(const BBoxT<DType>* inBbox, int idx)
{
    BBoxT<DType> ret = ((BBoxT<DType>*) (inBbox))[idx];
    return ret;
}

template <typename DType>
__device__ __forceinline__ DType boxIoU(const BBoxT<DType>& a, const BBoxT<DType>& b)
{
    BBoxT<DType> overlap = {
        dMAX(a.y1, b.y1), dMAX(a.x1, b.x1), dMIN(a.y2, b.y2), dMIN(a.x2, b.x2),
    };
    DType oW = overlap.x2 - overlap.x1;
    DType oH = overlap.y2 - overlap.y1;
    if (oW < (DType) 0 || oH < (DType) 0)
        return (DType) 0;
    DType oA = oW * oH;
    return (oA / ((a.y2 - a.y1) * (a.x2 - a.x1) + (b.y2 - b.y1) * (b.x2 - b.x1) - oA));
}

// PerClassNMS
// gridDim.x : batch-N
// blockDim.x : Threads
// ItemsPerThreads : = divUp(samples, Threads)
// outFlagSamples OUT: int [N * samples]
template <typename DType, typename BoxType, int Threads = 256, int ItemsPerThreads = 4>
__global__ void PerClassNMS_kernel(
    // int N,
    int samples, int NClass, const float nmsThreshold, const void* validSampleCountPtr,
    // const void *inScorePtr,
    const void* inLabelPtr, const void* inBboxPtr, const void* inBboxRefIdxPtr, const void* classStartsPtr,
    void* outFlagSamplesPtr)
{
    typedef BBoxT<BoxType> BBox;
    __shared__ struct
    {
        BBox refBox[MaxClassNum];
        int endIdx[MaxClassNum];
        int refIdx[MaxClassNum + 1];
        bool markSamples[Threads * ItemsPerThreads];
        int done;
    } smemClasses;
    assert(NClass + 1 < MaxClassNum);
    assert(samples <= Threads * ItemsPerThreads);

    const int* validSampleCount = static_cast<const int*>(validSampleCountPtr);
    // const DType *inScore = static_cast<const DType *>(inScorePtr);
    const BoxType* inLabel = static_cast<const BoxType*>(inLabelPtr);
    const BBox* inBbox = static_cast<const BBox*>(inBboxPtr);
    const int* inBboxRefIdx = static_cast<const int*>(inBboxRefIdxPtr);
    const int* classStarts = static_cast<const int*>(classStartsPtr);
    int* outFlagSamples = static_cast<int*>(outFlagSamplesPtr);

    int N = blockIdx.x;
    int blockOffset = N * samples;
    int validSamples = validSampleCount[N];

    if (threadIdx.x == 0)
    {
        smemClasses.done = 0;
    }

    BBox curBox[ItemsPerThreads];
    int label[ItemsPerThreads];
#pragma unroll
    for (int ite = 0; ite * blockDim.x < validSamples; ++ite)
    {
        int curIdx = ite * blockDim.x + threadIdx.x;
        if (curIdx < validSamples)
        {
            label[ite] = (int) inLabel[blockOffset + curIdx];
            curBox[ite] = readBbox(inBbox, blockOffset + inBboxRefIdx[blockOffset + curIdx]);
        }
        else
        {
            label[ite] = -1;
        }
        smemClasses.markSamples[curIdx] = (label[ite] < 0 ? false : true);
    }

    int classBlockOffset = N * (NClass + 1);
    for (int i = threadIdx.x; i < NClass + 1; i += blockDim.x)
    {
        int refIdx = classStarts[classBlockOffset + i];
        smemClasses.refIdx[i] = refIdx;
        smemClasses.refBox[i] = readBbox(inBbox, blockOffset + inBboxRefIdx[blockOffset + refIdx]);
    }
    __syncthreads();

    for (int i = threadIdx.x; i < NClass; i += blockDim.x)
    {
        int endIdx = smemClasses.refIdx[i + 1];
        smemClasses.endIdx[i] = endIdx;
        if (endIdx == smemClasses.refIdx[i])
        {
            atomicAdd(&smemClasses.done, 1);
        }
    }
    __syncthreads();

#if DUBUG_KERNEL
    // print info
    if (N == DUBUG_BATCH && threadIdx.x == 0)
    {
        printf("batch:%d, before starting NMS, done count:%d\n", N, smemClasses.done);
        printf("batch:%d, Total num:%d, startPos:\n", N, validSamples);
        for (int k = 0; k < NClass; ++k)
        {
            if (smemClasses.refIdx[k] != smemClasses.endIdx[k])
            {
                printf("Batch:%d, label:%d [%d : %d], check ref-label:%d\n", N, k, smemClasses.refIdx[k],
                    smemClasses.endIdx[k], (int) inLabel[blockOffset + smemClasses.refIdx[k]]);
            }
        }
        printf("\n");
    }
    __syncthreads();
#endif

    // class done to check stop point
    while (smemClasses.done < NClass)
    {

        for (int ite = 0; ite * blockDim.x < validSamples; ++ite)
        {
            int curIdx = ite * blockDim.x + threadIdx.x;
            int refIdx = -1;
            int endIdx = -1;
            if (curIdx < validSamples && smemClasses.markSamples[curIdx])
            {
                if (label[ite] >= 0)
                {
                    refIdx = smemClasses.refIdx[label[ite]];
                    endIdx = smemClasses.endIdx[label[ite]];
                    if (curIdx > refIdx && curIdx < endIdx)
                    {
                        BBox refBox = smemClasses.refBox[label[ite]];
                        if (boxIoU(refBox, curBox[ite]) > (DType) nmsThreshold)
                        {
                            smemClasses.markSamples[curIdx] = false;
                        }
                    }
                }
            }
        }
        __syncthreads();

        // push refIdx/refBox forward to next mark
        // only the refIdx thread to push itself. other threads idle
        for (int i = threadIdx.x; i < NClass; i += blockDim.x)
        {
            int refIdx = smemClasses.refIdx[i];
            int endIdx = smemClasses.endIdx[i];
            if (refIdx < endIdx)
            {
                do
                {
                    ++refIdx;
                } while (refIdx < endIdx && smemClasses.markSamples[refIdx] == false);
                smemClasses.refIdx[i] = refIdx;
                if (refIdx < endIdx)
                {
                    smemClasses.refBox[i] = readBbox(inBbox, blockOffset + inBboxRefIdx[blockOffset + refIdx]);
                }
                else
                {
                    atomicAdd(&smemClasses.done, 1);
                }
            }
        }
        __syncthreads();
    }

    // no need to write all data out
    for (int segment = 0; segment < validSamples; segment += blockDim.x)
    {
        int curIdx = segment + threadIdx.x;
        if (curIdx < validSamples)
        {
            outFlagSamples[blockOffset + curIdx] = (smemClasses.markSamples[curIdx] ? 1 : 0);
        }
    }
}

// TopKGather
// gridDim.x : batch-N
// blockDim.x : Threads
// ItemsPerThreads : = divUp(samples, Threads)
// outDetectionCount : int [N], must be set 0 before kernel
#define MaxItemsPerThreads 8
template <typename DType, typename BoxType, int Threads = 256>
__global__ void TopKGatherProposal_kernel(int samples, int keepTopK, const void* validSampleCountPtr,
    const void* inScorePtr, const void* inLabelPtr, const void* inBboxPtr, const void* inBboxRefIdxPtr,
    const void* inFlagSamplesPtr, void* outBboxPtr)
{
    typedef BBoxT<BoxType> BBox;
    typedef hipcub::BlockRadixSort<DType, Threads, 1, int> BlockRadixSort1;
    typedef hipcub::BlockRadixSort<DType, Threads, 2, int> BlockRadixSort2;
    typedef hipcub::BlockRadixSort<DType, Threads, 3, int> BlockRadixSort3;
    typedef hipcub::BlockRadixSort<DType, Threads, 4, int> BlockRadixSort4;
    typedef hipcub::BlockRadixSort<DType, Threads, 5, int> BlockRadixSort5;
    typedef hipcub::BlockRadixSort<DType, Threads, 6, int> BlockRadixSort6;
    typedef hipcub::BlockRadixSort<DType, Threads, 7, int> BlockRadixSort7;
    typedef hipcub::BlockRadixSort<DType, Threads, 8, int> BlockRadixSort8;
    __shared__ union
    {
        typename BlockRadixSort8::TempStorage sort8;
        typename BlockRadixSort7::TempStorage sort7;
        typename BlockRadixSort6::TempStorage sort6;
        typename BlockRadixSort5::TempStorage sort5;
        typename BlockRadixSort4::TempStorage sort4;
        typename BlockRadixSort3::TempStorage sort3;
        typename BlockRadixSort2::TempStorage sort2;
        typename BlockRadixSort1::TempStorage sort1;
    } temp_storage;
    assert(MaxItemsPerThreads * Threads >= samples);

    const int* validSampleCount = static_cast<const int*>(validSampleCountPtr);
    const DType* inScore = static_cast<const DType*>(inScorePtr);
    const BBox* inBbox = static_cast<const BBox*>(inBboxPtr);
    const int* inBboxRefIdx = static_cast<const int*>(inBboxRefIdxPtr);
    const int* inFlagSamples = static_cast<const int*>(inFlagSamplesPtr);
    BBox* outBbox = static_cast<BBox*>(outBboxPtr);

    int N = blockIdx.x;
    int blockOffset = N * samples;
    int validSamples = validSampleCount[N];
    int finalTopK = dMIN(keepTopK, validSamples);

    int idx[MaxItemsPerThreads];
    DType score[MaxItemsPerThreads];
    int totalItems = (validSamples + (blockDim.x - 1)) / blockDim.x;

    for (int ite = 0; ite < totalItems; ++ite)
    {
        int curIdx = ite * blockDim.x + threadIdx.x;
        if (curIdx < validSamples && inFlagSamples[blockOffset + curIdx])
        {
            idx[ite] = curIdx;
            score[ite] = inScore[blockOffset + curIdx];
        }
        else
        {
            idx[ite] = -1;
            score[ite] = 0.0f;
        }
    }

    switch (totalItems)
    {
    case 0: break;
    case 1:
        BlockRadixSort1(temp_storage.sort1).SortDescendingBlockedToStriped((DType(&)[1]) score, (int(&)[1]) idx);
        break;
    case 2:
        BlockRadixSort2(temp_storage.sort2).SortDescendingBlockedToStriped((DType(&)[2]) score, (int(&)[2]) idx);
        break;
    case 3:
        BlockRadixSort3(temp_storage.sort3).SortDescendingBlockedToStriped((DType(&)[3]) score, (int(&)[3]) idx);
        break;
    case 4:
        BlockRadixSort4(temp_storage.sort4).SortDescendingBlockedToStriped((DType(&)[4]) score, (int(&)[4]) idx);
        break;
    case 5:
        BlockRadixSort5(temp_storage.sort5).SortDescendingBlockedToStriped((DType(&)[5]) score, (int(&)[5]) idx);
        break;
    case 6:
        BlockRadixSort6(temp_storage.sort6).SortDescendingBlockedToStriped((DType(&)[6]) score, (int(&)[6]) idx);
        break;
    case 7:
        BlockRadixSort7(temp_storage.sort7).SortDescendingBlockedToStriped((DType(&)[7]) score, (int(&)[7]) idx);
        break;
    case 8:
        BlockRadixSort8(temp_storage.sort8).SortDescendingBlockedToStriped((DType(&)[8]) score, (int(&)[8]) idx);
        break;
    default: assert(false);
    }
    __syncthreads();

    int outBlockOffset = N * keepTopK;
    int topkItems = (keepTopK + (Threads - 1)) / Threads;
    for (int i = 0; i < topkItems; ++i)
    {
        int curI = i * blockDim.x + threadIdx.x;
        if (curI < keepTopK)
        {
            BBox oB = {(BoxType) 0.0f, (BoxType) 0.0f, (BoxType) 0.0f, (BoxType) 0.0f};
            if (curI < finalTopK && idx[i] >= 0 && score[i] > MinValidScore)
            {
                oB = ((BBox*) inBbox)[blockOffset + inBboxRefIdx[blockOffset + idx[i]]];
            }
            ((BBox*) outBbox)[outBlockOffset + curI] = oB;
        }
    }
}

#define MaxItemsPerThreads 8
template <typename DType, typename BoxType, int Threads = 256>
__global__ void TopKGather_kernel(int samples, int keepTopK, const void* validSampleCountPtr, const void* inScorePtr,
    const void* inLabelPtr, const void* inBboxPtr, const void* inBboxRefIdxPtr, const void* inFlagSamplesPtr,
    void* outDetectionPtr)
{
    typedef BBoxT<BoxType> BBox;
    typedef hipcub::BlockRadixSort<DType, Threads, 1, int> BlockRadixSort1;
    typedef hipcub::BlockRadixSort<DType, Threads, 2, int> BlockRadixSort2;
    typedef hipcub::BlockRadixSort<DType, Threads, 3, int> BlockRadixSort3;
    typedef hipcub::BlockRadixSort<DType, Threads, 4, int> BlockRadixSort4;
    typedef hipcub::BlockRadixSort<DType, Threads, 5, int> BlockRadixSort5;
    typedef hipcub::BlockRadixSort<DType, Threads, 6, int> BlockRadixSort6;
    typedef hipcub::BlockRadixSort<DType, Threads, 7, int> BlockRadixSort7;
    typedef hipcub::BlockRadixSort<DType, Threads, 8, int> BlockRadixSort8;
    __shared__ union
    {
        typename BlockRadixSort8::TempStorage sort8;
        typename BlockRadixSort7::TempStorage sort7;
        typename BlockRadixSort6::TempStorage sort6;
        typename BlockRadixSort5::TempStorage sort5;
        typename BlockRadixSort4::TempStorage sort4;
        typename BlockRadixSort3::TempStorage sort3;
        typename BlockRadixSort2::TempStorage sort2;
        typename BlockRadixSort1::TempStorage sort1;
    } temp_storage;
    assert(MaxItemsPerThreads * Threads >= samples);

    const int* validSampleCount = static_cast<const int*>(validSampleCountPtr);
    const DType* inScore = static_cast<const DType*>(inScorePtr);
    const BoxType* inLabel = static_cast<const BoxType*>(inLabelPtr); // InLabel keeps INT32
    const BBox* inBbox = static_cast<const BBox*>(inBboxPtr);
    const int* inBboxRefIdx = static_cast<const int*>(inBboxRefIdxPtr);
    const int* inFlagSamples = static_cast<const int*>(inFlagSamplesPtr);
    DType* outDetections = static_cast<DType*>(outDetectionPtr);

    int N = blockIdx.x;
    int blockOffset = N * samples;
    int validSamples = validSampleCount[N];
    int finalTopK = dMIN(keepTopK, validSamples);

    int idx[MaxItemsPerThreads];
    DType score[MaxItemsPerThreads];
    int totalItems = (validSamples + (blockDim.x - 1)) / blockDim.x;

    for (int ite = 0; ite < totalItems; ++ite)
    {
        int curIdx = ite * blockDim.x + threadIdx.x;
        if (curIdx < validSamples && inFlagSamples[blockOffset + curIdx])
        {
            idx[ite] = curIdx;
            score[ite] = inScore[blockOffset + curIdx];
        }
        else
        {
            idx[ite] = -1;
            score[ite] = 0.0f;
        }
    }

    switch (totalItems)
    {
    case 0: break;
    case 1:
        BlockRadixSort1(temp_storage.sort1).SortDescendingBlockedToStriped((DType(&)[1]) score, (int(&)[1]) idx);
        break;
    case 2:
        BlockRadixSort2(temp_storage.sort2).SortDescendingBlockedToStriped((DType(&)[2]) score, (int(&)[2]) idx);
        break;
    case 3:
        BlockRadixSort3(temp_storage.sort3).SortDescendingBlockedToStriped((DType(&)[3]) score, (int(&)[3]) idx);
        break;
    case 4:
        BlockRadixSort4(temp_storage.sort4).SortDescendingBlockedToStriped((DType(&)[4]) score, (int(&)[4]) idx);
        break;
    case 5:
        BlockRadixSort5(temp_storage.sort5).SortDescendingBlockedToStriped((DType(&)[5]) score, (int(&)[5]) idx);
        break;
    case 6:
        BlockRadixSort6(temp_storage.sort6).SortDescendingBlockedToStriped((DType(&)[6]) score, (int(&)[6]) idx);
        break;
    case 7:
        BlockRadixSort7(temp_storage.sort7).SortDescendingBlockedToStriped((DType(&)[7]) score, (int(&)[7]) idx);
        break;
    case 8:
        BlockRadixSort8(temp_storage.sort8).SortDescendingBlockedToStriped((DType(&)[8]) score, (int(&)[8]) idx);
        break;
    default: assert(false);
    }
    __syncthreads();

    int outBlockOffset = N * keepTopK;
    int topkItems = (keepTopK + (Threads - 1)) / Threads;
    for (int i = 0; i < topkItems; ++i)
    {
        int curI = i * blockDim.x + threadIdx.x;
        if (curI < keepTopK)
        {
            BBox oB = {(BoxType) 0.0f, (BoxType) 0.0f, (BoxType) 0.0f, (BoxType) 0.0f};
            DType oS = 0.0f;
            BoxType oL = -1;
            if (curI < finalTopK && idx[i] >= 0 && score[i] > MinValidScore)
            {
                oB = ((BBox*) inBbox)[blockOffset + inBboxRefIdx[blockOffset + idx[i]]];
                oS = score[i];
                oL = (BoxType) inLabel[blockOffset + idx[i]];
            }
            outDetections[(outBlockOffset + curI) * 6] = oB.y1;
            outDetections[(outBlockOffset + curI) * 6 + 1] = oB.x1;
            outDetections[(outBlockOffset + curI) * 6 + 2] = oB.y2;
            outDetections[(outBlockOffset + curI) * 6 + 3] = oB.x2;
            outDetections[(outBlockOffset + curI) * 6 + 4] = oL;
            outDetections[(outBlockOffset + curI) * 6 + 5] = oS;
        }
    }
}

RefineDetectionWorkSpace::RefineDetectionWorkSpace(
    const int batchSize, const int sampleCount, const RefineNMSParameters& param, const nvinfer1::DataType inType)
    : argMaxScoreDims(sampleCount, 1)
    , argMaxBboxDims(sampleCount, 4)
    , argMaxLabelDims(sampleCount, 1)
    , sortClassScoreDims(sampleCount, 1)
    , sortClassLabelDims(sampleCount, 1)
    , sortClassSampleIdxDims(sampleCount + 1, 1)
    , sortClassPosDims(param.numClasses + 1, 1)
    , sortNMSMarkDims(sampleCount, 1)
{
    size_t sumSize = 0;

    const nvinfer1::DataType type = nvinfer1::DataType::kFLOAT;

    // resource
    // arMaxScore : [N, samples] : m_Type
    argMaxScoreOffset = sumSize;
    sumSize += AlignMem(dimVolume(argMaxScoreDims) * typeSize(type) * batchSize);

    argMaxBboxOffset = sumSize;
    // argMaxBbox : [N, samples, 4] : m_Type
    sumSize += AlignMem(dimVolume(argMaxBboxDims) * typeSize(type) * batchSize);

    argMaxLabelOffset = sumSize;
    // argMaxLabel : [N, samples] : kINT32
    sumSize += AlignMem(dimVolume(argMaxLabelDims) * typeSize(nvinfer1::DataType::kINT32) * batchSize);

    sortClassScoreOffset = sumSize;
    // sortClassScore : [N, samples] : m_Type
    sumSize += AlignMem(dimVolume(sortClassScoreDims) * typeSize(type) * batchSize);

    sortClassLabelOffset = sumSize;
    // sortClassLabel : [N, samples] : m_Type
    sumSize += AlignMem(dimVolume(sortClassLabelDims) * typeSize(type) * batchSize);

    sortClassSampleIdxOffset = sumSize;
    // sortClassSampleIdx : [N, samples] : kINT32
    sumSize += AlignMem(dimVolume(sortClassSampleIdxDims) * typeSize(nvinfer1::DataType::kINT32) * batchSize);

    sortClassValidCountOffset = sumSize;
    // sortClassValidCount : [N, 1] : kINT32
    sumSize += AlignMem(dimVolume(sortClassValidCountDims) * typeSize(nvinfer1::DataType::kINT32) * batchSize);

    sortClassPosOffset = sumSize;
    // sortClassPos : [N, numClasses+1] : kINT32
    sumSize += AlignMem(dimVolume(sortClassPosDims) * typeSize(nvinfer1::DataType::kINT32) * batchSize);

    sortNMSMarkOffset = sumSize;
    // sortNMSMark : [N, samples] : kINT32
    sumSize += AlignMem(dimVolume(sortNMSMarkDims) * typeSize(nvinfer1::DataType::kINT32) * batchSize);

    totalSize = sumSize;
}

ProposalWorkSpace::ProposalWorkSpace(const int batchSize, const int inputCnt, const int sampleCount,
    const RefineNMSParameters& param, const nvinfer1::DataType inType)
    : preRefineScoreDims(inputCnt, 1)
    , preRefineSortedScoreDims(inputCnt, 1)
    , preRefineBboxDims(inputCnt, 4)
    , argMaxScoreDims(sampleCount, 1)
    , argMaxBboxDims(sampleCount, 4)
    , argMaxLabelDims(sampleCount, 1)
    , sortClassScoreDims(sampleCount, 1)
    , sortClassLabelDims(sampleCount, 1)
    , sortClassSampleIdxDims(sampleCount, 1)
    , sortClassPosDims(param.numClasses + 1, 1)
    , sortNMSMarkDims(sampleCount, 1)
{
    size_t sumSize = 0;

    const nvinfer1::DataType type = nvinfer1::DataType::kFLOAT;

    // resource
    // temp storage size for sorting scores
    tempStorageOffset = sumSize;
    sumSize += (1 << 23) * batchSize;

    // preRefineScore : [N, inputcnt, 1] // extracted foreground score from inputs[0]
    preRefineScoreOffset = sumSize;
    sumSize += AlignMem(dimVolume(preRefineScoreDims) * typeSize(type) * batchSize);

    // preRefineSortedScore: [N, inputcnt, 1]
    preRefineSortedScoreOffset = sumSize;
    sumSize += AlignMem(dimVolume(preRefineSortedScoreDims) * typeSize(type) * batchSize);

    // preRefineBbox: [N, inputcnt, 4] // sorted bbox
    preRefineBboxOffset = sumSize;
    sumSize += AlignMem(dimVolume(preRefineBboxDims) * typeSize(type) * batchSize);

    // arMaxScore : [N, samples] : m_Type
    argMaxScoreOffset = sumSize;
    sumSize += AlignMem(dimVolume(argMaxScoreDims) * typeSize(type) * batchSize);

    argMaxBboxOffset = sumSize;
    // argMaxBbox : [N, samples, 4] : m_Type
    sumSize += AlignMem(dimVolume(argMaxBboxDims) * typeSize(type) * batchSize);

    argMaxLabelOffset = sumSize;
    // argMaxLabel : [N, samples] : kINT32
    sumSize += AlignMem(dimVolume(argMaxLabelDims) * typeSize(nvinfer1::DataType::kINT32) * batchSize);

    sortClassScoreOffset = sumSize;
    // sortClassScore : [N, samples] : m_Type
    sumSize += AlignMem(dimVolume(sortClassScoreDims) * typeSize(type) * batchSize);

    sortClassLabelOffset = sumSize;
    // sortClassLabel : [N, samples] : m_Type
    sumSize += AlignMem(dimVolume(sortClassLabelDims) * typeSize(type) * batchSize);

    sortClassSampleIdxOffset = sumSize;
    // sortClassSampleIdx : [N, samples] : kINT32
    sumSize += AlignMem(dimVolume(sortClassSampleIdxDims) * typeSize(nvinfer1::DataType::kINT32) * batchSize);

    sortClassValidCountOffset = sumSize;
    // sortClassValidCount : [N, 1] : kINT32
    sumSize += AlignMem(dimVolume(sortClassValidCountDims) * typeSize(nvinfer1::DataType::kINT32) * batchSize);

    sortClassPosOffset = sumSize;
    // sortClassPos : [N, numClasses+1] : kINT32
    sumSize += AlignMem(dimVolume(sortClassPosDims) * typeSize(nvinfer1::DataType::kINT32) * batchSize);

    sortNMSMarkOffset = sumSize;
    // sortNMSMark : [N, samples] : kINT32
    sumSize += AlignMem(dimVolume(sortNMSMarkDims) * typeSize(nvinfer1::DataType::kINT32) * batchSize);

    totalSize = sumSize;
}

MultilevelProposeROIWorkSpace::MultilevelProposeROIWorkSpace(const int batchSize, const int inputCnt, const int sampleCount,
    const RefineNMSParameters& param, const nvinfer1::DataType inType)
    : preRefineSortedScoreDims(inputCnt, 1)
    , preRefineBboxDims(inputCnt, 4)
    , argMaxScoreDims(sampleCount, 1)
    , argMaxBboxDims(sampleCount, 4)
    , argMaxLabelDims(sampleCount, 1)
    , sortClassScoreDims(sampleCount, 1)
    , sortClassLabelDims(sampleCount, 1)
    , sortClassSampleIdxDims(sampleCount+1, 1)
    , sortClassPosDims(param.numClasses + 1, 1)
    , sortNMSMarkDims(sampleCount, 1)
{
    size_t sumSize = 0;

    const nvinfer1::DataType type = nvinfer1::DataType::kFLOAT;

    // resource
    // temp storage size for sorting scores
    tempStorageOffset = sumSize;
    sumSize += (1 << 23) * batchSize;

    // preRefineSortedScore: [N, inputcnt, 1]
    preRefineSortedScoreOffset = sumSize;
    sumSize += AlignMem(dimVolume(preRefineSortedScoreDims) * typeSize(type) * batchSize);

    // preRefineBbox: [N, inputcnt, 4] // sorted bbox
    preRefineBboxOffset = sumSize;
    sumSize += AlignMem(dimVolume(preRefineBboxDims) * typeSize(type) * batchSize);

    // argMaxScore : [N, samples] : m_Type
    argMaxScoreOffset = sumSize;
    sumSize += AlignMem(dimVolume(argMaxScoreDims) * typeSize(type) * batchSize);

    argMaxBboxOffset = sumSize;
    // argMaxBbox : [N, samples, 4] : m_Type
    sumSize += AlignMem(dimVolume(argMaxBboxDims) * typeSize(type) * batchSize);

    argMaxLabelOffset = sumSize;
    // argMaxLabel : [N, samples] : m_Type
    sumSize += AlignMem(dimVolume(argMaxLabelDims) * typeSize(type) * batchSize);

    sortClassScoreOffset = sumSize;
    // sortClassScore : [N, samples] : m_Type
    sumSize += AlignMem(dimVolume(sortClassScoreDims) * typeSize(type) * batchSize);

    sortClassLabelOffset = sumSize;
    // sortClassLabel : [N, samples] : m_Type
    sumSize += AlignMem(dimVolume(sortClassLabelDims) * typeSize(type) * batchSize);

    sortClassSampleIdxOffset = sumSize;
    // sortClassSampleIdx : [N, samples] : kINT32
    sumSize += AlignMem(dimVolume(sortClassSampleIdxDims) * typeSize(nvinfer1::DataType::kINT32) * batchSize);

    sortClassValidCountOffset = sumSize;
    // sortClassValidCount : [N, 1] : kINT32
    sumSize += AlignMem(dimVolume(sortClassValidCountDims) * typeSize(nvinfer1::DataType::kINT32) * batchSize);

    sortClassPosOffset = sumSize;
    // sortClassPos : [N, numClasses+1] : kINT32
    sumSize += AlignMem(dimVolume(sortClassPosDims) * typeSize(nvinfer1::DataType::kINT32) * batchSize);

    sortNMSMarkOffset = sumSize;
    // sortNMSMark : [N, samples] : kINT32
    sumSize += AlignMem(dimVolume(sortNMSMarkDims) * typeSize(nvinfer1::DataType::kINT32) * batchSize);

    totalSize = sumSize;
}

ConcatTopKWorkSpace::ConcatTopKWorkSpace(const int batchSize, const int concatCnt, const int topK,const nvinfer1::DataType inType)
    : concatedScoreDims(concatCnt*topK, 1)
    , concatedBBoxDims(concatCnt*topK, 4)
    , sortedScoreDims(concatCnt*topK, 1)
    , sortedBBoxDims(concatCnt*topK, 4) 
{
    size_t sumSize = 0;

    const nvinfer1::DataType type = nvinfer1::DataType::kFLOAT;

    // resource
    // temp storage size for sorting scores
    tempStorageOffset = sumSize;
    sumSize += (1 << 23) * batchSize;

    // concatedScoreOffset: [N, concatCnt*topK, 1] 
    concatedScoreOffset = sumSize;
    sumSize += AlignMem(dimVolume(concatedScoreDims) * typeSize(type) * batchSize);

    // concatedBBoxOffset: [N, concatCnt*topK, 4]
    concatedBBoxOffset = sumSize;
    sumSize += AlignMem(dimVolume(concatedBBoxDims) * typeSize(type) * batchSize);

    // sortedScoreOffset: [N, concatCnt * topK, 1]
    sortedScoreOffset = sumSize;
    sumSize += AlignMem(dimVolume(sortedScoreDims) * typeSize(type) * batchSize);

    // sortedBBoxOffset: [N, concatCnt * topK, 4]
    sortedBBoxOffset = sumSize;
    sumSize += AlignMem(dimVolume(sortedBBoxDims) * typeSize(type) * batchSize);

    totalSize = sumSize;
}

template <int Threads>
hipError_t argMaxGroup(hipStream_t stream, int N, nvinfer1::DataType dtype, int samples, int NClass,
    const void* inScore, const void* inBbox, const void* validSamples, void* outScore, void* outLabel, void* outBbox)
{
    int maxGridX = dMIN(samples, 512 / N);
    dim3 gridDim = {(unsigned int) nAlignDown(maxGridX, 32), (unsigned int) N, 1};
    dim3 threads = {Threads, 1, 1};
    switch (dtype)
    {
    case nvinfer1::DataType::kFLOAT:
        argMaxGroup_kernel<float, float, Threads><<<gridDim, threads, 0, stream>>>(
            samples, 0, NClass, inScore, inBbox, validSamples, outScore, outLabel, outBbox);
        break;
    case nvinfer1::DataType::kHALF: break;
    default: assert(false);
    }

    return hipGetLastError();
}

template <int Threads>
hipError_t argMaxWOBackground(hipStream_t stream, int N, nvinfer1::DataType dtype, int samples, int NClass,
    const void* inScore, const void* inBbox, const void* validSamples, void* outScore, void* outLabel, void* outBbox)
{
    int maxGridX = dMIN(samples, 512 / N);
    dim3 gridDim = {(unsigned int) nAlignDown(maxGridX, 32), (unsigned int) N, 1};
    dim3 threads = {Threads, 1, 1};
    switch (dtype)
    {
    case nvinfer1::DataType::kFLOAT:
        argMaxGroup_kernel<float, float, Threads><<<gridDim, threads, 0, stream>>>(
            samples, 1, NClass, inScore, inBbox, validSamples, outScore, outLabel, outBbox);
        break;
    case nvinfer1::DataType::kHALF: break;
    default: assert(false);
    }

    return hipGetLastError();
}

template <int Threads, int ItermPerThreads>
hipError_t sortPerClass(hipStream_t stream, int N, nvinfer1::DataType dtype, int samples, int NClass, int background,
    float scoreThreshold, const void* inSampleValidCount, const void* inScorePtr, const void* inLabelPtr,
    const void* inBboxPtr, void* outclassStartPosPtr, void* outScorePtr, void* outLabelPtr, void* outSampleIdxPtr,
    void* outValidSampleCountPtr)
{
    int blocks = N;
    int threads = Threads;

    switch (dtype)
    {
    case nvinfer1::DataType::kFLOAT:
        sortPerClass_kernel<float, float, Threads, ItermPerThreads><<<blocks, threads, 0, stream>>>(samples, NClass,
            background, scoreThreshold, inSampleValidCount, inScorePtr, inLabelPtr, inBboxPtr, outclassStartPosPtr,
            outScorePtr, outLabelPtr, outSampleIdxPtr, outValidSampleCountPtr);
        break;
    case nvinfer1::DataType::kHALF: break;
    default: assert(false);
    }

    return hipGetLastError();
};

template <int Threads>
hipError_t PerClassNMS(hipStream_t stream, int N, nvinfer1::DataType dtype, int samples, int NClass,
    const float nmsThreshold, const void* validSampleCount,
    // const void *inScore,
    const void* inLabel, const void* inBbox, const void* inBboxRefIdx, const void* classStarts, void* outFlagSamples)
{
    int blocks = N;
    int threads = Threads;

    switch (dtype)
    {
    case nvinfer1::DataType::kFLOAT:
        PerClassNMS_kernel<float, float, Threads><<<blocks, threads, 0, stream>>>(samples, NClass, nmsThreshold,
            validSampleCount, inLabel, inBbox, inBboxRefIdx, classStarts, outFlagSamples);
        break;
    case nvinfer1::DataType::kHALF: break;
    default: assert(false);
    }

    return hipGetLastError();
}

template <int Threads>
hipError_t KeepTopKGather(hipStream_t stream, int N, nvinfer1::DataType dtype, int samples, int keepTopK,
    const void* validSampleCountPtr, const void* inScorePtr, const void* inLabelPtr, const void* inBboxPtr,
    const void* inBboxRefIdxPtr, const void* inFlagSamplesPtr, void* outDetections, int proposal)
{
    int blocks = N;
    int threads = Threads;

    switch (dtype)
    {
    case nvinfer1::DataType::kFLOAT:
        if (proposal)
        {
            TopKGatherProposal_kernel<float, float, Threads><<<blocks, threads, 0, stream>>>(samples, keepTopK,
                validSampleCountPtr, inScorePtr, inLabelPtr, inBboxPtr, inBboxRefIdxPtr, inFlagSamplesPtr,
                outDetections);
        }
        else
        {
            TopKGather_kernel<float, float, Threads><<<blocks, threads, 0, stream>>>(samples, keepTopK,
                validSampleCountPtr, inScorePtr, inLabelPtr, inBboxPtr, inBboxRefIdxPtr, inFlagSamplesPtr,
                outDetections);
        }
        break;
    case nvinfer1::DataType::kHALF: break;
    default: assert(false);
    }

    return hipGetLastError();
}

// TopKGather For TLT RPN Proposal
// gridDim.x : batch-N
// blockDim.x : Threads
// ItemsPerThreads : = divUp(samples, Threads)
// outDetectionCount : int [N], must be set 0 before kernel
#define MaxItemsPerThreads 8
template <typename DType, typename BoxType, int Threads = 256>
__global__ void TopKGatherBoxScore_kernel(int samples, int keepTopK, const void* validSampleCountPtr,
    const void* inScorePtr, const void* inLabelPtr, const void* inBboxPtr, const void* inBboxRefIdxPtr,
    const void* inFlagSamplesPtr, void* outScorePtr, void* outBboxPtr)
{
    typedef hipcub::BlockRadixSort<DType, Threads, 1, int> BlockRadixSort1;
    typedef hipcub::BlockRadixSort<DType, Threads, 2, int> BlockRadixSort2;
    typedef hipcub::BlockRadixSort<DType, Threads, 3, int> BlockRadixSort3;
    typedef hipcub::BlockRadixSort<DType, Threads, 4, int> BlockRadixSort4;
    typedef hipcub::BlockRadixSort<DType, Threads, 5, int> BlockRadixSort5;
    typedef hipcub::BlockRadixSort<DType, Threads, 6, int> BlockRadixSort6;
    typedef hipcub::BlockRadixSort<DType, Threads, 7, int> BlockRadixSort7;
    typedef hipcub::BlockRadixSort<DType, Threads, 8, int> BlockRadixSort8;
    __shared__ union
    {
        typename BlockRadixSort8::TempStorage sort8;
        typename BlockRadixSort7::TempStorage sort7;
        typename BlockRadixSort6::TempStorage sort6;
        typename BlockRadixSort5::TempStorage sort5;
        typename BlockRadixSort4::TempStorage sort4;
        typename BlockRadixSort3::TempStorage sort3;
        typename BlockRadixSort2::TempStorage sort2;
        typename BlockRadixSort1::TempStorage sort1;
    } temp_storage;
    assert(MaxItemsPerThreads * Threads >= samples);

    typedef BBoxT<BoxType> BBox;
    const int* validSampleCount = static_cast<const int*>(validSampleCountPtr);
    const DType* inScore = static_cast<const DType*>(inScorePtr);
    const BBox* inBbox = static_cast<const BBox*>(inBboxPtr);
    const int* inBboxRefIdx = static_cast<const int*>(inBboxRefIdxPtr);
    const int* inFlagSamples = static_cast<const int*>(inFlagSamplesPtr);
    BBox* outBbox = static_cast<BBox*>(outBboxPtr);
    DType* outScore = static_cast<DType*>(outScorePtr);

    int N = blockIdx.x;
    int blockOffset = N * samples;
    int validSamples = validSampleCount[N];
    int finalTopK = dMIN(keepTopK, validSamples);

    int idx[MaxItemsPerThreads];
    DType score[MaxItemsPerThreads];
    int totalItems = (validSamples + (blockDim.x - 1)) / blockDim.x;
    
    for (int ite = 0; ite < totalItems; ++ite)
    {
        int curIdx = ite * blockDim.x + threadIdx.x;
        if (curIdx < validSamples && inFlagSamples[blockOffset + curIdx])
        {
            idx[ite] = curIdx;
            score[ite] = inScore[blockOffset + curIdx];
        }
        else
        {
            idx[ite] = -1;
            score[ite] = 0.0f;
        }
    }


    switch (totalItems)
    {
    case 0: break;
    case 1:
        BlockRadixSort1(temp_storage.sort1).SortDescendingBlockedToStriped((DType(&)[1]) score, (int(&)[1]) idx);
        break;
    case 2:
        BlockRadixSort2(temp_storage.sort2).SortDescendingBlockedToStriped((DType(&)[2]) score, (int(&)[2]) idx);
        break;
    case 3:
        BlockRadixSort3(temp_storage.sort3).SortDescendingBlockedToStriped((DType(&)[3]) score, (int(&)[3]) idx);
        break;
    case 4:
        BlockRadixSort4(temp_storage.sort4).SortDescendingBlockedToStriped((DType(&)[4]) score, (int(&)[4]) idx);
        break;
    case 5:
        BlockRadixSort5(temp_storage.sort5).SortDescendingBlockedToStriped((DType(&)[5]) score, (int(&)[5]) idx);
        break;
    case 6:
        BlockRadixSort6(temp_storage.sort6).SortDescendingBlockedToStriped((DType(&)[6]) score, (int(&)[6]) idx);
        break;
    case 7:
        BlockRadixSort7(temp_storage.sort7).SortDescendingBlockedToStriped((DType(&)[7]) score, (int(&)[7]) idx);
        break;
    case 8:
        BlockRadixSort8(temp_storage.sort8).SortDescendingBlockedToStriped((DType(&)[8]) score, (int(&)[8]) idx);
        break;
    default: assert(false);
    }
    __syncthreads();

    int outBlockOffset = N * keepTopK;
    int topkItems = (keepTopK + (Threads - 1)) / Threads;
    for (int i = 0; i < topkItems; ++i)
    {
        int curI = i * blockDim.x + threadIdx.x;
        if (curI < keepTopK)
        {
            BBox oB = {(BoxType) 0.0f, (BoxType) 0.0f, (BoxType) 0.0f, (BoxType) 0.0f};
            DType oS = 0.0f;
            if (curI < finalTopK && idx[i] >= 0)
            {
                oB = ((BBox*) inBbox)[blockOffset + inBboxRefIdx[blockOffset + idx[i]]];
                oS = score[i];
            }
            ((BBox*) outBbox)[outBlockOffset + curI] = oB;
            outScore[outBlockOffset + curI] = oS;
        }
    }
}

template <int Threads>
hipError_t KeepTopKGatherBoxScore(hipStream_t stream, int N, nvinfer1::DataType dtype, int samples, int keepTopK,
    const void* validSampleCountPtr, const void* inScorePtr, const void* inLabelPtr, const void* inBboxPtr,
    const void* inBboxRefIdxPtr, const void* inFlagSamplesPtr, void* outScores, void* outDetections, int proposal)
{
    int blocks = N;
    int threads = Threads;

    switch (dtype)
    {
    case nvinfer1::DataType::kFLOAT:
        if (proposal)
        {
            TopKGatherBoxScore_kernel<float, float, Threads><<<blocks, threads, 0, stream>>>(samples, keepTopK,
                validSampleCountPtr, inScorePtr, inLabelPtr, inBboxPtr, inBboxRefIdxPtr, inFlagSamplesPtr,
                outScores, outDetections);
        }
        else
        {
            TopKGather_kernel<float, float, Threads><<<blocks, threads, 0, stream>>>(samples, keepTopK,
                validSampleCountPtr, inScorePtr, inLabelPtr, inBboxPtr, inBboxRefIdxPtr, inFlagSamplesPtr,
                outDetections);
        }
        break;
    case nvinfer1::DataType::kHALF: break;
    default: assert(false);
    }

    return hipGetLastError();
}

hipError_t RefineBatchClassNMS(hipStream_t stream, int N, int samples, nvinfer1::DataType dtype,
    const RefineNMSParameters& param, const RefineDetectionWorkSpace& refineOffset, void* workspace,
    const void* inScores, const void* inDelta, const void* inCountValid, const void* inROI, void* outDetections)
{
    int NClass = param.numClasses;
    int8_t* wsPtr = static_cast<int8_t*>(workspace);
    void* argMaxScorePtr = wsPtr + refineOffset.argMaxScoreOffset;
    void* argMaxLabelPtr = wsPtr + refineOffset.argMaxLabelOffset;
    void* argMaxBBoxPtr = wsPtr + refineOffset.argMaxBboxOffset;

    void* sortClassScorePtr = wsPtr + refineOffset.sortClassScoreOffset;
    void* sortClassLabelPtr = wsPtr + refineOffset.sortClassLabelOffset;
    void* sortClassSampleIdxPtr = wsPtr + refineOffset.sortClassSampleIdxOffset;
    void* sortClassValidCountPtr = wsPtr + refineOffset.sortClassValidCountOffset;
    void* sortClassPosPtr = wsPtr + refineOffset.sortClassPosOffset;
    void* sortNMSMarkPtr = wsPtr + refineOffset.sortNMSMarkOffset;

    hipError_t status = hipSuccess;
    CUASSERT(hipMemsetAsync(sortClassValidCountPtr, 0, N * sizeof(int), stream));

    if (NClass > 1)
    { // multiple classes
        status = argMaxGroup<32>(stream, N, dtype, samples, NClass, inScores, inDelta, inCountValid, argMaxScorePtr,
            argMaxLabelPtr, argMaxBBoxPtr); // argMaxBBoxPtr means delta of bboxes
        assert(status == hipSuccess);
        CUASSERT(status);
    }
    else
    { // Only one class
        argMaxScorePtr = const_cast<void*>(inScores);
        argMaxBBoxPtr = const_cast<void*>(inDelta);
        int threads = 512;
        int blocks = (N * samples + threads - 1) / threads;
        blocks = dMIN(blocks, 8);
        switch (dtype)
        {
        case nvinfer1::DataType::kFLOAT:
        {
            resetMemValue_kernel<float><<<blocks, threads, 0, stream>>>(argMaxLabelPtr, N * samples, 0);
            break;
        }
        case nvinfer1::DataType::kHALF: { break;
        }
        default: assert(false);
        }
    }

    status = ApplyDelta2Bboxes(stream, N, samples, inROI, argMaxBBoxPtr, argMaxBBoxPtr);
    assert(status == hipSuccess);

    if (samples <= 1024)
    {
        status = sortPerClass<256, 4>(stream, N, dtype, samples, NClass, param.backgroundLabelId, param.scoreThreshold,
            inCountValid, argMaxScorePtr, argMaxLabelPtr, argMaxBBoxPtr, sortClassPosPtr, sortClassScorePtr,
            sortClassLabelPtr, sortClassSampleIdxPtr, sortClassValidCountPtr);
    }
    else if (samples <= 2048)
    {
        status = sortPerClass<256, 8>(stream, N, dtype, samples, NClass, param.backgroundLabelId, param.scoreThreshold,
            inCountValid, argMaxScorePtr, argMaxLabelPtr, argMaxBBoxPtr, sortClassPosPtr, sortClassScorePtr,
            sortClassLabelPtr, sortClassSampleIdxPtr, sortClassValidCountPtr);
    }
    else if (samples <= 4096)
    {
        status = sortPerClass<256, 16>(stream, N, dtype, samples, NClass, param.backgroundLabelId, param.scoreThreshold,
            inCountValid, argMaxScorePtr, argMaxLabelPtr, argMaxBBoxPtr, sortClassPosPtr, sortClassScorePtr,
            sortClassLabelPtr, sortClassSampleIdxPtr, sortClassValidCountPtr);
    }
    else
    {
        assert(false && "unsupported sortPerClass");
        return hipErrorLaunchFailure;
    }
    assert(status == hipSuccess);
    CUASSERT(status);

    status = PerClassNMS<256>(stream, N, dtype, samples, NClass, param.iouThreshold, sortClassValidCountPtr,
        // sortClassScorePtr,
        sortClassLabelPtr, argMaxBBoxPtr, sortClassSampleIdxPtr, sortClassPosPtr, sortNMSMarkPtr);
    assert(status == hipSuccess);
    CUASSERT(status);

    status = KeepTopKGather<256>(stream, N, dtype, samples, param.keepTopK, sortClassValidCountPtr, sortClassScorePtr,
        sortClassLabelPtr, argMaxBBoxPtr, sortClassSampleIdxPtr, sortNMSMarkPtr, outDetections, 0);
    assert(status == hipSuccess);
    CUASSERT(status);
    return status;
}

hipError_t DetectionPostProcess(hipStream_t stream, int N, int samples, const float* regWeight, 
    const float inputHeight, const float inputWidth, nvinfer1::DataType dtype,
    const RefineNMSParameters& param, const RefineDetectionWorkSpace& refineOffset, void* workspace,
    const void* inScores, const void* inDelta, const void* inCountValid, const void* inROI, void* outDetections)
{
    int NClass = param.numClasses;
    int8_t* wsPtr = static_cast<int8_t*>(workspace);
    void* argMaxScorePtr = wsPtr + refineOffset.argMaxScoreOffset;
    void* argMaxLabelPtr = wsPtr + refineOffset.argMaxLabelOffset;
    void* argMaxBBoxPtr = wsPtr + refineOffset.argMaxBboxOffset;

    void* sortClassScorePtr = wsPtr + refineOffset.sortClassScoreOffset;
    void* sortClassLabelPtr = wsPtr + refineOffset.sortClassLabelOffset;
    void* sortClassSampleIdxPtr = wsPtr + refineOffset.sortClassSampleIdxOffset;
    void* sortClassValidCountPtr = wsPtr + refineOffset.sortClassValidCountOffset;
    void* sortClassPosPtr = wsPtr + refineOffset.sortClassPosOffset;
    void* sortNMSMarkPtr = wsPtr + refineOffset.sortNMSMarkOffset;

    hipError_t status = hipSuccess;
    CUASSERT(hipMemsetAsync(argMaxScorePtr, 0, N * samples * sizeof(float), stream));
    CUASSERT(hipMemsetAsync(argMaxBBoxPtr, 0, N * samples * 4* sizeof(float), stream));
    CUASSERT(hipMemsetAsync(sortClassValidCountPtr, 0, N * sizeof(int), stream));
    CUASSERT(hipMemsetAsync(sortClassPosPtr, 0, N * (NClass+1) * sizeof(int), stream));
    CUASSERT(hipMemsetAsync(sortClassSampleIdxPtr, 0, N * (samples + 1) * sizeof(int), stream));

    if (NClass > 1)
    { // multiple classes
        status = argMaxWOBackground<32>(stream, N, dtype, samples, NClass, inScores, inDelta, inCountValid, argMaxScorePtr,
            argMaxLabelPtr, argMaxBBoxPtr); // argMaxBBoxPtr means delta of bboxes
        assert(status == hipSuccess);
        CUASSERT(status);
    }
    else
    { // Only one class
        argMaxScorePtr = const_cast<void*>(inScores);
        argMaxBBoxPtr = const_cast<void*>(inDelta);
        int threads = 512;
        int blocks = (N * samples + threads - 1) / threads;
        blocks = dMIN(blocks, 8);
        switch (dtype)
        {
        case nvinfer1::DataType::kFLOAT:
        {
            resetMemValue_kernel<float><<<blocks, threads, 0, stream>>>(argMaxLabelPtr, N * samples, 0);
            break;
        }
        case nvinfer1::DataType::kHALF: { break;
        }
        default: assert(false);
        }
    }

    status = DecodeBBoxes(stream, N, samples, regWeight, inputHeight, inputWidth, 
        inROI, argMaxBBoxPtr, argMaxBBoxPtr);
    assert(status == hipSuccess);
    
    if (samples <= 1024)
    {
        status = sortPerClass<256, 4>(stream, N, dtype, samples, NClass, param.backgroundLabelId, param.scoreThreshold,
            inCountValid, argMaxScorePtr, argMaxLabelPtr, argMaxBBoxPtr, sortClassPosPtr, sortClassScorePtr,
            sortClassLabelPtr, sortClassSampleIdxPtr, sortClassValidCountPtr);
    }
    else if (samples <= 2048)
    {
        status = sortPerClass<256, 8>(stream, N, dtype, samples, NClass, param.backgroundLabelId, param.scoreThreshold,
            inCountValid, argMaxScorePtr, argMaxLabelPtr, argMaxBBoxPtr, sortClassPosPtr, sortClassScorePtr,
            sortClassLabelPtr, sortClassSampleIdxPtr, sortClassValidCountPtr);
    }
    else if (samples <= 4096)
    {
        status = sortPerClass<256, 16>(stream, N, dtype, samples, NClass, param.backgroundLabelId, param.scoreThreshold,
            inCountValid, argMaxScorePtr, argMaxLabelPtr, argMaxBBoxPtr, sortClassPosPtr, sortClassScorePtr,
            sortClassLabelPtr, sortClassSampleIdxPtr, sortClassValidCountPtr);
    }
    else
    {
        assert(false && "unsupported sortPerClass");
        return hipErrorLaunchFailure;
    }
    assert(status == hipSuccess);
    CUASSERT(status);

    status = PerClassNMS<256>(stream, N, dtype, samples, NClass, param.iouThreshold, sortClassValidCountPtr,
        // sortClassScorePtr,
        sortClassLabelPtr, argMaxBBoxPtr, sortClassSampleIdxPtr, sortClassPosPtr, sortNMSMarkPtr);
    CUASSERT(status);

    status = KeepTopKGather<256>(stream, N, dtype, samples, param.keepTopK, sortClassValidCountPtr, sortClassScorePtr,
        sortClassLabelPtr, argMaxBBoxPtr, sortClassSampleIdxPtr, sortNMSMarkPtr, outDetections, 0);
    CUASSERT(status);

    return status;
}

struct BF_SCORE
{
    float bg, fg;
};
// in_scores : [N, samples, 2]
// output_score : [N, samples, 1]
__global__ void extract_fg_kernel(int samples, const void* in_scores, void* output_score)
{
    const BF_SCORE* in = static_cast<const BF_SCORE*>(in_scores);
    float* out = static_cast<float*>(output_score);

    int N = blockIdx.x;
    int blockOffset = N * samples;
    int totalItems = (samples + (blockDim.x - 1)) / blockDim.x;

    for (int i = 0; i < totalItems; i++)
    {
        int cur_id = i * blockDim.x + threadIdx.x;
        out[blockOffset + cur_id] = in[blockOffset + cur_id].fg;
    }
}
__global__ void set_offset_kernel(int stride, int size, int* output)
{
    // One block, because batch size shouldn't be too large.
    for (int i = threadIdx.x; i < size; i += blockDim.x)
    {
        output[i] = i * stride;
    }
}

__global__ void resample_kernel(int orig_size, int sample_size, const void* orig_score_ptr, const void* orig_bbox_ptr,
    void* sampled_score_ptr, void* sampled_bbox_ptr)
{
    const float* in_score = static_cast<const float*>(orig_score_ptr);
    const BBoxT<float>* in_bbox = static_cast<const BBoxT<float>*>(orig_bbox_ptr);
    float* out_score = static_cast<float*>(sampled_score_ptr);
    BBoxT<float>* out_bbox = static_cast<BBoxT<float>*>(sampled_bbox_ptr);

    int N = blockIdx.x;
    int blockOffset_in = N * orig_size;
    int blockOffset_out = N * sample_size;
    int realSampleCnt = dMIN(sample_size, orig_size);
    int totalItems = (realSampleCnt + (blockDim.x - 1)) / blockDim.x;

    for (int i = 0; i < totalItems; i++)
    {
        int cur_id = i * blockDim.x + threadIdx.x;
        if (cur_id < realSampleCnt)
        {
            out_score[blockOffset_out + cur_id] = in_score[blockOffset_in + cur_id];
            out_bbox[blockOffset_out + cur_id] = in_bbox[blockOffset_in + cur_id];
        }
    }
}

hipError_t proposalRefineBatchClassNMS(hipStream_t stream, int N, int inputCnt, int samples, nvinfer1::DataType dtype,
    const RefineNMSParameters& param, const ProposalWorkSpace& proposalOffset, void* workspace,
    const void* inScores, //[N, inputcnt, 2]
    const void* inDelta,  //[N, inputcnt, 4]
    const void* inCountValid,
    const void* inAnchors, //[N, inputcnt, 4]
    void* outProposals)
{
    int8_t* wsPtr = static_cast<int8_t*>(workspace);
    void* tempStoragePtr = wsPtr + proposalOffset.tempStorageOffset;
    void* preRefineScorePtr = wsPtr + proposalOffset.preRefineScoreOffset;
    void* preRefineSortedScorePtr = wsPtr + proposalOffset.preRefineSortedScoreOffset;
    void* preRefineBboxPtr = wsPtr + proposalOffset.preRefineBboxOffset;

    void* argMaxScorePtr = wsPtr + proposalOffset.argMaxScoreOffset;
    void* argMaxLabelPtr = wsPtr + proposalOffset.argMaxLabelOffset;
    void* argMaxBBoxPtr = wsPtr + proposalOffset.argMaxBboxOffset;

    void* sortClassScorePtr = wsPtr + proposalOffset.sortClassScoreOffset;
    void* sortClassLabelPtr = wsPtr + proposalOffset.sortClassLabelOffset;
    void* sortClassSampleIdxPtr = wsPtr + proposalOffset.sortClassSampleIdxOffset;
    void* sortClassValidCountPtr = wsPtr + proposalOffset.sortClassValidCountOffset;
    void* sortClassPosPtr = wsPtr + proposalOffset.sortClassPosOffset;
    void* sortNMSMarkPtr = wsPtr + proposalOffset.sortNMSMarkOffset;

    hipError_t status = hipSuccess;
    CUASSERT(hipMemsetAsync(sortClassValidCountPtr, 0, N * sizeof(int), stream));

    // extract foreground score
    extract_fg_kernel<<<N, dMIN(inputCnt, 1024), 0, stream>>>(inputCnt, inScores, preRefineScorePtr);
    CUASSERT(hipGetLastError());

    // Here, inDelta are converted to normalize coordinates based on anchors
    status = ApplyDelta2Bboxes(stream, N, inputCnt, inAnchors, inDelta, const_cast<void*>(inDelta));
    CUASSERT(status);

    // sort the score
    // d_key_in: preRefineScorePtr [N, inputCnt, 1]
    // d_key_out: preRefineSortedScorePtr
    // d_values_in: inDelta [N, inputCnt, 4]
    // d_values_out: preRefineBboxPtr
    // num_items: inputCnt*N
    // num_segments: N
    // offsets: [0, inputCnt, inputCnt*2, ..., ]
    int* offsets = static_cast<int*>(tempStoragePtr);
    set_offset_kernel<<<1, 1024, 0, stream>>>(inputCnt, N + 1, offsets);
    assert(hipGetLastError() == hipSuccess);
    tempStoragePtr = static_cast<void*>(static_cast<int*>(tempStoragePtr) + (N + 1));

    size_t temp_storage_bytes = 0;
    hipcub::DeviceSegmentedRadixSort::SortPairsDescending(NULL, temp_storage_bytes, (float*) preRefineScorePtr,
        (float*) preRefineSortedScorePtr, (BBoxT<float>*) inDelta, (BBoxT<float>*) preRefineBboxPtr, N * inputCnt, N,
        offsets, offsets + 1, 0, 8 * sizeof(float), stream);

    assert((1 << 23) * (size_t)N > temp_storage_bytes);

    hipcub::DeviceSegmentedRadixSort::SortPairsDescending(tempStoragePtr, temp_storage_bytes, (float*) preRefineScorePtr,
        (float*) preRefineSortedScorePtr, (BBoxT<float>*) inDelta, (BBoxT<float>*) preRefineBboxPtr, N * inputCnt, N,
        offsets, offsets + 1, 0, 8 * sizeof(float), stream);

    int NClass = param.numClasses;
    assert(NClass == 1);
    if (NClass == 1)
    { // Only one class
        resample_kernel<<<N, dMIN(samples, 1024), 0, stream>>>(
            inputCnt, samples, preRefineSortedScorePtr, preRefineBboxPtr, argMaxScorePtr, argMaxBBoxPtr);

        int threads = 512;
        int blocks = (N * samples + threads - 1) / threads;
        blocks = dMIN(blocks, 8);
        switch (dtype)
        {
        case nvinfer1::DataType::kFLOAT:
        {
            resetMemValue_kernel<float><<<blocks, threads, 0, stream>>>(argMaxLabelPtr, N * samples, 0);
            break;
        }
        case nvinfer1::DataType::kHALF: { 
            break;
        }
        default: assert(false);
        }
    }

    if (samples <= 1024)
    {
        status = sortPerClass<256, 4>(stream, N, dtype, samples, NClass, param.backgroundLabelId, param.scoreThreshold,
            inCountValid, argMaxScorePtr, argMaxLabelPtr, argMaxBBoxPtr, sortClassPosPtr, sortClassScorePtr,
            sortClassLabelPtr, sortClassSampleIdxPtr, sortClassValidCountPtr);
    }
    else if (samples <= 2048)
    {
        status = sortPerClass<256, 8>(stream, N, dtype, samples, NClass, param.backgroundLabelId, param.scoreThreshold,
            inCountValid, argMaxScorePtr, argMaxLabelPtr, argMaxBBoxPtr, sortClassPosPtr, sortClassScorePtr,
            sortClassLabelPtr, sortClassSampleIdxPtr, sortClassValidCountPtr);
    }
    else if (samples <= 4096)
    {
        status = sortPerClass<256, 16>(stream, N, dtype, samples, NClass, param.backgroundLabelId, param.scoreThreshold,
            inCountValid, argMaxScorePtr, argMaxLabelPtr, argMaxBBoxPtr, sortClassPosPtr, sortClassScorePtr,
            sortClassLabelPtr, sortClassSampleIdxPtr, sortClassValidCountPtr);
    }
    else
    {
        assert(false && "unsupported sortPerClass");
        return hipErrorLaunchFailure;
    }
    CUASSERT(status);

    status = PerClassNMS<256>(stream, N, dtype, samples, NClass, param.iouThreshold, sortClassValidCountPtr,
        // sortClassScorePtr,
        sortClassLabelPtr, argMaxBBoxPtr, sortClassSampleIdxPtr, sortClassPosPtr, sortNMSMarkPtr);
    CUASSERT(status);

    status = KeepTopKGather<256>(stream, N, dtype, samples, param.keepTopK, sortClassValidCountPtr, sortClassScorePtr,
        sortClassLabelPtr, argMaxBBoxPtr, sortClassSampleIdxPtr, sortNMSMarkPtr, outProposals, 1);
    CUASSERT(status);

    return status;
}

hipError_t MultilevelPropose(hipStream_t stream, int N, int inputCnt, int samples, const float* regWeight, 
    const float inputHeight, const float inputWidth, nvinfer1::DataType dtype, const RefineNMSParameters& param, 
    const MultilevelProposeROIWorkSpace& proposalOffset, void* workspace,
    const void* inScore, //[N, inputcnt, 1]
    const void* inDelta,  //[N, inputcnt, 4]
    void* inCountValid,
    const void* inAnchors, //[N, inputcnt, 4]
    void* outScore, 
    void* outBbox)
{
    int8_t* wsPtr = static_cast<int8_t*>(workspace);
    void* tempStoragePtr = wsPtr + proposalOffset.tempStorageOffset;
    void* preRefineSortedScorePtr = wsPtr + proposalOffset.preRefineSortedScoreOffset;
    void* preRefineBboxPtr = wsPtr + proposalOffset.preRefineBboxOffset;

    void* argMaxScorePtr = wsPtr + proposalOffset.argMaxScoreOffset;
    void* argMaxLabelPtr = wsPtr + proposalOffset.argMaxLabelOffset;
    void* argMaxBBoxPtr = wsPtr + proposalOffset.argMaxBboxOffset;

    void* sortClassScorePtr = wsPtr + proposalOffset.sortClassScoreOffset;
    void* sortClassLabelPtr = wsPtr + proposalOffset.sortClassLabelOffset;
    void* sortClassSampleIdxPtr = wsPtr + proposalOffset.sortClassSampleIdxOffset;
    void* sortClassValidCountPtr = wsPtr + proposalOffset.sortClassValidCountOffset;
    void* sortClassPosPtr = wsPtr + proposalOffset.sortClassPosOffset;
    void* sortNMSMarkPtr = wsPtr + proposalOffset.sortNMSMarkOffset;

    hipError_t status = hipSuccess;
    int NClass = param.numClasses;
    assert(NClass == 1);
    CUASSERT(hipMemsetAsync(argMaxScorePtr, 0, N * samples * sizeof(float), stream));
    CUASSERT(hipMemsetAsync(argMaxBBoxPtr, 0, N * samples * 4* sizeof(float), stream));
    CUASSERT(hipMemsetAsync(sortClassValidCountPtr, 0, N * sizeof(int), stream));
    CUASSERT(hipMemsetAsync(sortClassPosPtr, 0, N * (NClass+1) * sizeof(int), stream));
    CUASSERT(hipMemsetAsync(sortClassSampleIdxPtr, 0, N * (samples + 1) * sizeof(int), stream));

    CUASSERT(hipGetLastError());

    // Here, inDelta are converted to normalize coordinates based on anchors
    status = DecodeBBoxes(stream, N, inputCnt, regWeight, inputHeight, inputWidth, 
                                  inAnchors, inDelta, const_cast<void*>(inDelta));
    CUASSERT(hipGetLastError());

    // sort the score
    // d_key_in: preRefineScorePtr [N, inputCnt, 1]
    // d_key_out: preRefineSortedScorePtr
    // d_values_in: inDelta [N, inputCnt, 4]
    // d_values_out: preRefineBboxPtr
    // num_items: inputCnt*N
    // num_segments: N
    // offsets: [0, inputCnt, inputCnt*2, ..., ]
    
    int* offsets = static_cast<int*>(tempStoragePtr);
    set_offset_kernel<<<1, 1024, 0, stream>>>(inputCnt, N + 1, offsets);
    CUASSERT(hipGetLastError());
    tempStoragePtr = static_cast<void*>(static_cast<int*>(tempStoragePtr) + (N + 1));

    size_t temp_storage_bytes = 0;
    hipcub::DeviceSegmentedRadixSort::SortPairsDescending(NULL, temp_storage_bytes, (float*) inScore,
        (float*) preRefineSortedScorePtr, (BBoxT<float>*) inDelta, (BBoxT<float>*) preRefineBboxPtr, N * inputCnt, N,
        offsets, offsets + 1, 0, 8 * sizeof(float), stream);
    CUASSERT(hipGetLastError());

    assert((1 << 23) * N > temp_storage_bytes);

    hipcub::DeviceSegmentedRadixSort::SortPairsDescending(tempStoragePtr, temp_storage_bytes, (float*) inScore,
        (float*) preRefineSortedScorePtr, (BBoxT<float>*) inDelta, (BBoxT<float>*) preRefineBboxPtr, N * inputCnt, N,
        offsets, offsets + 1, 0, 8 * sizeof(float), stream);
    CUASSERT(hipGetLastError());

    if (NClass == 1)
    { // Only one class
        resample_kernel<<<N, dMIN(samples, 1024), 0, stream>>>(
            inputCnt, samples, preRefineSortedScorePtr, preRefineBboxPtr, argMaxScorePtr, argMaxBBoxPtr);

        CUASSERT(hipGetLastError());

        int threads = 512;
        int blocks = (N * samples + threads - 1) / threads;
        blocks = dMIN(blocks, 8);
       
        switch (dtype)
        {
        case nvinfer1::DataType::kFLOAT:
        {
            resetMemValue_kernel<float><<<blocks, threads, 0, stream>>>(argMaxLabelPtr, N * samples, 0);
            CUASSERT(hipGetLastError());
            break;
        }
        case nvinfer1::DataType::kHALF: { 
            break;
        }
        default: assert(false);
        }
    }
    

    if (samples <= 1024)
    {
        status = sortPerClass<256, 4>(stream, N, dtype, samples, NClass, param.backgroundLabelId, param.scoreThreshold,
            inCountValid, argMaxScorePtr, argMaxLabelPtr, argMaxBBoxPtr, sortClassPosPtr, sortClassScorePtr,
            sortClassLabelPtr, sortClassSampleIdxPtr, sortClassValidCountPtr);
    }
    else if (samples <= 2048)
    {
        status = sortPerClass<256, 8>(stream, N, dtype, samples, NClass, param.backgroundLabelId, param.scoreThreshold,
            inCountValid, argMaxScorePtr, argMaxLabelPtr, argMaxBBoxPtr, sortClassPosPtr, sortClassScorePtr,
            sortClassLabelPtr, sortClassSampleIdxPtr, sortClassValidCountPtr);
    }
    else if (samples <= 4096)
    {
        status = sortPerClass<256, 16>(stream, N, dtype, samples, NClass, param.backgroundLabelId, param.scoreThreshold,
            inCountValid, argMaxScorePtr, argMaxLabelPtr, argMaxBBoxPtr, sortClassPosPtr, sortClassScorePtr,
            sortClassLabelPtr, sortClassSampleIdxPtr, sortClassValidCountPtr);
    }
    else
    {
        assert(false && "unsupported sortPerClass");
        return hipErrorLaunchFailure;
    }
    CUASSERT(hipGetLastError());

    status = PerClassNMS<1024>(stream, N, dtype, samples, NClass, param.iouThreshold, sortClassValidCountPtr,
        // sortClassScorePtr,
        sortClassLabelPtr, argMaxBBoxPtr, sortClassSampleIdxPtr, sortClassPosPtr, sortNMSMarkPtr);

    CUASSERT(hipGetLastError());

    status = KeepTopKGatherBoxScore<1024>(stream, N, dtype, samples, param.keepTopK, sortClassValidCountPtr, sortClassScorePtr,
        sortClassLabelPtr, argMaxBBoxPtr, sortClassSampleIdxPtr, sortNMSMarkPtr, outScore, outBbox, 1);

    CUASSERT(hipGetLastError());

    return status;
}

struct BBOX
{
    float y1, x1, y2, x2;
};

struct DELTA
{
    float dy, dx, logdh, logdw;
};

__global__ void decode_bboxes_kernel(int samples, const void* anchors, const void* delta, 
                const float* regWeight, const float inputHeight, const float inputWidth, 
                void* outputBbox, float bboxClipThresh)
{

    const BBOX* anchors_in = static_cast<const BBOX*>(anchors);
    const DELTA* delta_in = static_cast<const DELTA*>(delta);
    BBOX* bbox_out = static_cast<BBOX*>(outputBbox);

    int N = blockIdx.x;
    int blockOffset = N * samples;
    int totalItems = (samples + (blockDim.x - 1)) / blockDim.x;

    for (int i = 0; i < totalItems; i++)
    {
        int cur_id = i * blockDim.x + threadIdx.x;

        if (cur_id < samples)
        {
          BBOX cur_anchor_yxyx = anchors_in[blockOffset + cur_id];
          // convert yxyx -> cyxhw
          // cy, cx, h, w
          /*BBOX cur_anchor_cyxhw;*/

          float cur_anchor_h = (cur_anchor_yxyx.y2 - cur_anchor_yxyx.y1 + 1.0);
          float cur_anchor_w = (cur_anchor_yxyx.x2 - cur_anchor_yxyx.x1 + 1.0); //w
          float cur_anchor_yc = cur_anchor_yxyx.y1 + cur_anchor_h * 0.5; //cy
          float cur_anchor_xc = cur_anchor_yxyx.x1 + cur_anchor_w * 0.5; //cx

          DELTA cur_delta = delta_in[blockOffset + cur_id];

          // divided by regWeight
          cur_delta.dy /= regWeight[0];
          cur_delta.dx /= regWeight[1];
          cur_delta.logdh /= regWeight[2];
          cur_delta.logdw /= regWeight[3];

          cur_delta.logdh = dMIN(cur_delta.logdh, bboxClipThresh);
          cur_delta.logdw = dMIN(cur_delta.logdw, bboxClipThresh);

          // apply delta
          float decoded_box_yc = cur_anchor_yc + cur_delta.dy * cur_anchor_h;
          float decoded_box_xc = cur_anchor_xc + cur_delta.dx * cur_anchor_w; 
          float decoded_box_h = expf(cur_delta.logdh) * cur_anchor_h;
          float decoded_box_w = expf(cur_delta.logdw) * cur_anchor_w;

          float decoded_box_ymin = decoded_box_yc - 0.5 * decoded_box_h;
          float decoded_box_xmin = decoded_box_xc - 0.5 * decoded_box_w;
          float decoded_box_ymax = decoded_box_ymin + decoded_box_h - 1.0;
          float decoded_box_xmax = decoded_box_xmin + decoded_box_w - 1.0;

          // clip bbox: a more precision clip method based on real window could be implemented
          decoded_box_ymin = dMAX(dMIN(decoded_box_ymin, inputHeight - 1.0), 0.0);
          decoded_box_xmin = dMAX(dMIN(decoded_box_xmin, inputWidth - 1.0), 0.0);
          decoded_box_ymax = dMAX(dMIN(decoded_box_ymax, inputHeight - 1.0), 0.0);
          decoded_box_xmax = dMAX(dMIN(decoded_box_xmax, inputWidth - 1.0), 0.0);

          bbox_out[blockOffset + cur_id].y1 = decoded_box_ymin;
          bbox_out[blockOffset + cur_id].x1 = decoded_box_xmin;
          bbox_out[blockOffset + cur_id].y2 = decoded_box_ymax;
          bbox_out[blockOffset + cur_id].x2 = decoded_box_xmax;
        }
    }
}

hipError_t DecodeBBoxes(hipStream_t stream, int N,
    int samples,         // number of anchors per image
    const float* regWeight, 
    const float inputHeight,
    const float inputWidth,
    const void* anchors, // [N, anchors, (y1, x1, y2, x2)]
    const void* delta,   //[N, anchors, (dy, dx, log(dh), log(dw)])
    void* outputBbox     //[N, anchors, (y1, x1, y2, x2)]
    )
{

    int blocks = N;
    int threads = dMIN(samples, 1024);

    // delta multiply bbox_std
    // apply delta steps:
    //  cy = anchor_cy + dy*height
    //  cx = anchor_cx + dx*weight
    //  h = exp(dh)*anchor_h
    //  w = exp(dw)*anchor_w
    // clip the bbox in absolute coordinates
    float bboxClipThresh = log(1000.0f/16.0f);

    decode_bboxes_kernel<<<blocks, threads, 0, stream>>>(samples, anchors, 
          delta, regWeight, inputHeight, inputWidth,  outputBbox, bboxClipThresh);

    return hipGetLastError();
}

__global__ void apply_delta_kernel(int samples, const void* anchors, const void* delta, void* outputBbox)
{

    const BBOX* anchors_in = static_cast<const BBOX*>(anchors);
    const DELTA* delta_in = static_cast<const DELTA*>(delta);
    BBOX* bbox_out = static_cast<BBOX*>(outputBbox);

    int N = blockIdx.x;
    int blockOffset = N * samples;
    int totalItems = (samples + (blockDim.x - 1)) / blockDim.x;

    for (int i = 0; i < totalItems; i++)
    {
        int cur_id = i * blockDim.x + threadIdx.x;

        BBOX cur_anchor_yxyx = anchors_in[blockOffset + cur_id];
        // convert yxyx -> cyxhw
        // cy, cx, h, w
        BBOX cur_anchor_cyxhw;

        cur_anchor_cyxhw.y1 = (cur_anchor_yxyx.y1 + cur_anchor_yxyx.y2) / 2;
        cur_anchor_cyxhw.x1 = (cur_anchor_yxyx.x1 + cur_anchor_yxyx.x2) / 2;
        cur_anchor_cyxhw.y2 = (cur_anchor_yxyx.y2 - cur_anchor_yxyx.y1);
        cur_anchor_cyxhw.x2 = (cur_anchor_yxyx.x2 - cur_anchor_yxyx.x1);

        DELTA cur_delta = delta_in[blockOffset + cur_id];

        // multiply std_dev
        cur_delta.dy *= 0.1;
        cur_delta.dx *= 0.1;
        cur_delta.logdh *= 0.2;
        cur_delta.logdw *= 0.2;

        // apply delta
        cur_anchor_cyxhw.y1 += cur_delta.dy * cur_anchor_cyxhw.y2;
        cur_anchor_cyxhw.x1 += cur_delta.dx * cur_anchor_cyxhw.x2;
        cur_anchor_cyxhw.y2 *= expf(cur_delta.logdh);
        cur_anchor_cyxhw.x2 *= expf(cur_delta.logdw);

        cur_anchor_yxyx.y1 = cur_anchor_cyxhw.y1 - 0.5 * cur_anchor_cyxhw.y2;
        cur_anchor_yxyx.x1 = cur_anchor_cyxhw.x1 - 0.5 * cur_anchor_cyxhw.x2;
        cur_anchor_yxyx.y2 = cur_anchor_yxyx.y1 + cur_anchor_cyxhw.y2;
        cur_anchor_yxyx.x2 = cur_anchor_yxyx.x1 + cur_anchor_cyxhw.x2;

        // clip bbox: a more precision clip method based on real window could be implemented
        cur_anchor_yxyx.y1 = dMAX(dMIN(cur_anchor_yxyx.y1, 1.0), 0.0);
        cur_anchor_yxyx.x1 = dMAX(dMIN(cur_anchor_yxyx.x1, 1.0), 0.0);
        cur_anchor_yxyx.y2 = dMAX(dMIN(cur_anchor_yxyx.y2, 1.0), 0.0);
        cur_anchor_yxyx.x2 = dMAX(dMIN(cur_anchor_yxyx.x2, 1.0), 0.0);

        bbox_out[blockOffset + cur_id].y1 = cur_anchor_yxyx.y1;
        bbox_out[blockOffset + cur_id].x1 = cur_anchor_yxyx.x1;
        bbox_out[blockOffset + cur_id].y2 = cur_anchor_yxyx.y2;
        bbox_out[blockOffset + cur_id].x2 = cur_anchor_yxyx.x2;
    }
}

hipError_t ApplyDelta2Bboxes(hipStream_t stream, int N,
    int samples,         // number of anchors per image
    const void* anchors, // [N, anchors, (y1, x1, y2, x2)]
    const void* delta,   //[N, anchors, (dy, dx, log(dh), log(dw)])
    void* outputBbox     //[N, anchors, (y1, x1, y2, x2)]
    )
{

    int blocks = N;
    int threads = dMIN(samples, 1024);

    // delta multiply bbox_std
    // apply delta steps:
    //  cy = anchor_cy + dy*height
    //  cx = anchor_cx + dx*weight
    //  h = exp(dh)*anchor_h
    //  w = exp(dw)*anchor_w
    // clip the bbox

    apply_delta_kernel<<<blocks, threads, 0, stream>>>(samples, anchors, delta, outputBbox);

    return hipGetLastError();
}

template <typename Tfeat>
__device__ inline Tfeat interpolateBilinear(const Tfeat* src, xy_t srcDims, float y, float x)
{
    const int y0 = static_cast<int>(y);
    const float yAlpha = y - static_cast<float>(y0);
    const int x0 = static_cast<int>(x);
    const float xAlpha = x - static_cast<float>(x0);

    assert(y0 < srcDims.y);
    assert(x0 < srcDims.x);

    const int y1 = (yAlpha == 0) ? y0 : y0 + 1; // ceil
    const int x1 = (xAlpha == 0) ? x0 : x0 + 1; // ceil

    assert(y1 < srcDims.y);
    assert(x1 < srcDims.x);

    const float src00 = src[(y0) *srcDims.x + (x0)];
    const float src01 = src[(y0) *srcDims.x + (x1)];
    const float src10 = src[(y1) *srcDims.x + (x0)];
    const float src11 = src[(y1) *srcDims.x + (x1)];

    const float src0 = src00 * (1 - xAlpha) + src01 * xAlpha;
    const float src1 = src10 * (1 - xAlpha) + src11 * xAlpha;

    return src0 * (1 - yAlpha) + src1 * yAlpha;
}

template <typename Trois, typename Tfeat>
__global__ void roiAlign_kernel(int featureCount, int roiCount,

    float threshold, const Trois* rois,

    const Tfeat* P2, const xy_t P2dims, const Tfeat* P3, const xy_t P3dims, const Tfeat* P4, const xy_t P4dims,
    const Tfeat* P5, const xy_t P5dims,

    Tfeat* pooled, const xy_t poolDims)
{
    const int batch = blockIdx.x;
    const int feature = blockIdx.y;

    for (int roiIdx = threadIdx.x; roiIdx < roiCount; roiIdx += blockDim.x)
    {
        const Trois* roi = rois + 4 * (batch * roiCount + roiIdx);

        const float y1 = roi[0];
        const float x1 = roi[1];
        const float y2 = roi[2];
        const float x2 = roi[3];

        if (!(0 <= y1 && y1 <= 1 && 0 <= x1 && x1 <= 1 && 0 <= y2 && y2 <= 1 && 0 <= x2 && x2 <= 1 && y1 < y2
                && x1 < x2))
        {

            continue;
        }
        else
        {
        }

        const float hw = (y2 - y1) * (x2 - x1);

        const Tfeat* src = P2;
        xy_t srcDims = P2dims;
        int iP = 2;

        if (hw > threshold)
        {
            src = P3;
            srcDims = P3dims;
            ++iP;
        }
        threshold *= 4;

        if (hw > threshold)
        {
            src = P4;
            srcDims = P4dims;
            ++iP;
        }
        threshold *= 4;

        if (hw > threshold)
        {
            src = P5;
            srcDims = P5dims;
            ++iP;
        }

        src += srcDims.x * srcDims.y * (batch * featureCount + feature);

        Tfeat* dst
            = pooled + poolDims.x * poolDims.y * (batch * roiCount * featureCount + roiIdx * featureCount + feature);

        const float yStart = y1 * (srcDims.y - 1);
        const float xStart = x1 * (srcDims.x - 1);

        const float yEnd = y2 * (srcDims.y - 1);
        const float xEnd = x2 * (srcDims.x - 1);

        const float yDelta = (yEnd - yStart) / (poolDims.y - 1);
        const float xDelta = (xEnd - xStart) / (poolDims.x - 1);

        for (int yy = 0; yy < poolDims.y; ++yy)
        {
            const float ySample = min(yStart + yDelta * yy, yEnd);

            for (int xx = 0; xx < poolDims.x; ++xx)
            {
                const float xSample = min(xStart + xDelta * xx, xEnd);

                float result = interpolateBilinear(src, srcDims, ySample, xSample);

                *dst = result;
                dst++;
            }
        }
    }
}

hipError_t roiAlign(hipStream_t stream, int batchSize, int featureCount, int roiCount, float firstThreshold,

    const void* rois, const void* const layers[], const xy_t* layerDims,

    void* pooled, const xy_t poolDims)
{
    const dim3 blocks(batchSize, featureCount);
    const int threads(256);

    roiAlign_kernel<<<blocks, threads, 0, stream>>>(featureCount, roiCount, firstThreshold,
        static_cast<const float*>(rois),

        static_cast<const float*>(layers[0]), layerDims[0], static_cast<const float*>(layers[1]), layerDims[1],
        static_cast<const float*>(layers[2]), layerDims[2], static_cast<const float*>(layers[3]), layerDims[3],

        static_cast<float*>(pooled), poolDims);
    return hipGetLastError();
}


template <typename Trois, typename Tfeat>
__global__ void roiAlignHalfCenter_kernel(int featureCount, int roiCount,

    float threshold, int inputHeight, int inputWidth, const Trois* rois,

    const Tfeat* P2, const xy_t P2dims, const Tfeat* P3, const xy_t P3dims, const Tfeat* P4, const xy_t P4dims,
    const Tfeat* P5, const xy_t P5dims, const Tfeat* P6, const xy_t P6dims, 

    Tfeat* pooled, const xy_t poolDims)
{
    const int batch = blockIdx.x;
    const int feature = blockIdx.y;

    for (int roiIdx = threadIdx.x; roiIdx < roiCount; roiIdx += blockDim.x)
    {
        const Trois* roi = rois + 4 * (batch * roiCount + roiIdx);

        const float y1 = roi[0];
        const float x1 = roi[1];
        const float y2 = roi[2];
        const float x2 = roi[3];

        if (!(0 <= y1 && y1 <= inputHeight && 0 <= x1 && x1 <= inputWidth && 0 <= y2 && y2 <= inputHeight && 0 <= x2 && x2 <= inputWidth && y1 < y2
                && x1 < x2))
        {

            continue;
        }
        else
        {
        }

        const float hw = (y2 - y1) * (x2 - x1);

        const Tfeat* src = P2;
        xy_t srcDims = P2dims;
        int iP = 2;

        if (hw > threshold)
        {
            src = P3;
            srcDims = P3dims;
            ++iP;
        }
        threshold *= 4;

        if (hw > threshold)
        {
            src = P4;
            srcDims = P4dims;
            ++iP;
        }
        threshold *= 4;

        if (hw > threshold)
        {
            src = P5;
            srcDims = P5dims;
            ++iP;
        }
        threshold *= 4;

        if (hw > threshold)
        {
            src = P6;
            srcDims = P6dims;
            ++iP; 
        }
    

        src += srcDims.x * srcDims.y * (batch * featureCount + feature);

        Tfeat* dst
            = pooled + poolDims.x * poolDims.y * (batch * roiCount * featureCount + roiIdx * featureCount + feature);

        float scale_to_level = 1.0f;
        for(int i = 0; i < iP; i++)
        {
          scale_to_level *= 2.0f;
        }

        const float yStart = y1 / scale_to_level;
        const float xStart = x1 / scale_to_level;

        const float yEnd = y2 / scale_to_level;
        const float xEnd = x2 / scale_to_level;

        const float yDelta = (yEnd - yStart) / (poolDims.y);
        const float xDelta = (xEnd - xStart) / (poolDims.x);

        for (int yy = 0; yy < poolDims.y; ++yy)
        {
            const float ySample = dMIN(dMAX(yStart + yDelta * (yy + 0.5), 0.0f), srcDims.y - 1.0f);

            for (int xx = 0; xx < poolDims.x; ++xx)
            {
                const float xSample = dMIN(dMAX(xStart + xDelta * (xx + 0.5), 0.0f), srcDims.x - 1.0f);

                float result = interpolateBilinear(src, srcDims, ySample, xSample);

                *dst = result;
                dst++;
            }
        }
    }
}

hipError_t roiAlignHalfCenter(hipStream_t stream, int batchSize, int featureCount, int roiCount, float firstThreshold,

    int inputHeight, int inputWidth, const void* rois, const void* const layers[], const xy_t* layerDims,

    void* pooled, const xy_t poolDims)
{
    const dim3 blocks(batchSize, featureCount);
    const int threads(256);

    roiAlignHalfCenter_kernel<<<blocks, threads, 0, stream>>>(featureCount, roiCount, firstThreshold, inputHeight, inputWidth,
        static_cast<const float*>(rois),
        static_cast<const float*>(layers[0]), layerDims[0], static_cast<const float*>(layers[1]), layerDims[1],
        static_cast<const float*>(layers[2]), layerDims[2], static_cast<const float*>(layers[3]), layerDims[3],
        static_cast<const float*>(layers[4]), layerDims[4], 
        static_cast<float*>(pooled), poolDims);

    return hipGetLastError();
}

__global__ void resize_nearest_kernel_2d(int nbatch, float scale, int2 osize, float const* idata, int istride,
    int ibatchstride, float* odata, int ostride, int obatchstride)
{

    int x0 = threadIdx.x + blockIdx.x * blockDim.x;
    int y0 = threadIdx.y + blockIdx.y * blockDim.y;
    int z0 = blockIdx.z;
    for (int batch = z0; batch < nbatch; batch += gridDim.z)
    {
        for (int oy = y0; oy < osize.y; oy += blockDim.y * gridDim.y)
        {
            for (int ox = x0; ox < osize.x; ox += blockDim.x * gridDim.x)
            {
                int ix = int(ox / scale);
                int iy = int(oy / scale);
                odata[batch * obatchstride + oy * ostride + ox] = idata[batch * ibatchstride + iy * istride + ix];
            }
        }
    }
}

void resizeNearest(dim3 grid, dim3 block, hipStream_t stream, int nbatch, float scale, int2 osize, float const* idata,
    int istride, int ibatchstride, float* odata, int ostride, int obatchstride)
{

    resize_nearest_kernel_2d<<<grid, block, 0, stream>>>(
        nbatch, scale, osize, idata, istride, ibatchstride, odata, ostride, obatchstride);
}

struct BOX
{
    float y1, x1, y2, x2;
};

struct DETECTION
{
    float y1, x1, y2, x2, class_id, score;
};

__global__ void specialslice_kernel(int samples, const void* idata, void* odata)
{

    int N = blockIdx.x;
    int blockOffset = N * samples;
    int totalItems = (samples + (blockDim.x - 1)) / blockDim.x;
    const DETECTION* in_detections = static_cast<const DETECTION*>(idata);
    BOX* out_bboxes = static_cast<BOX*>(odata);

    for (int i = 0; i < totalItems; i++)
    {
        int cur_id = i * blockDim.x + threadIdx.x;

        out_bboxes[blockOffset + cur_id].y1 = in_detections[blockOffset + cur_id].y1;
        out_bboxes[blockOffset + cur_id].x1 = in_detections[blockOffset + cur_id].x1;
        out_bboxes[blockOffset + cur_id].y2 = in_detections[blockOffset + cur_id].y2;
        out_bboxes[blockOffset + cur_id].x2 = in_detections[blockOffset + cur_id].x2;
    }
}

void specialSlice(hipStream_t stream, int batch_size, int boxes_cnt, const void* idata, void* odata)
{
    int blocks = batch_size;
    int threads = dMIN(boxes_cnt, 2048);

    specialslice_kernel<<<blocks, threads, 0, stream>>>(boxes_cnt, idata, odata);
}

template <typename Dtype>
__global__ void concatenate(int featureCnt, int sampleCnt, const void* const* inScores, const void* const* inBBox, 
                            void* outScore, void* outBBox)
{
    int N = blockIdx.x;
    int outBlockOffset = N * sampleCnt * featureCnt; 
    int inBlockOffset = N * sampleCnt; 
    int itemsPerThread = (sampleCnt + blockDim.x - 1) / blockDim.x;
    Dtype* outScorePtr = static_cast<Dtype*>(outScore);
    BOX* outBBoxPtr = static_cast<BOX*>(outBBox); 

    for(int fId = 0; fId < featureCnt; fId++)
    {
        const Dtype* fInScorePtr = static_cast<const Dtype*>(inScores[fId]);
        const BOX* fInBBoxPtr = static_cast<const BOX*>(inBBox[fId]);
        int featureOffset = fId * sampleCnt;
        for(int i = 0; i < itemsPerThread; i++)
        {
            int curId = i * blockDim.x + threadIdx.x;
            if (curId < sampleCnt)
            {
              outScorePtr[outBlockOffset + featureOffset + curId] = fInScorePtr[inBlockOffset + curId];
              outBBoxPtr[outBlockOffset + featureOffset + curId] = fInBBoxPtr[inBlockOffset + curId];
            }
        }
    }
}

__global__ void resampleBBox_kernel(int orig_size, int sample_size, const void* orig_bbox_ptr, void* sampled_bbox_ptr)
{
    const BBoxT<float>* in_bbox = static_cast<const BBoxT<float>*>(orig_bbox_ptr);
    BBoxT<float>* out_bbox = static_cast<BBoxT<float>*>(sampled_bbox_ptr);

    int N = blockIdx.x;
    int blockOffset_in = N * orig_size;
    int blockOffset_out = N * sample_size;
    int totalItems = (sample_size + (blockDim.x - 1)) / blockDim.x;

    for (int i = 0; i < totalItems; i++)
    {
        int cur_id = i * blockDim.x + threadIdx.x;
        if (cur_id < sample_size)
        {
          out_bbox[blockOffset_out + cur_id] = in_bbox[blockOffset_in + cur_id];
        }
    }
}

hipError_t ConcatTopK(hipStream_t stream, 
    int N, 
    int featureCnt, 
    int topK, 
    nvinfer1::DataType dtype, 
    void* workspace, 
    const ConcatTopKWorkSpace& spaceOffset,
    void** inScores, 
    void** inBBox,
    void* outProposals)
{
    //Prepare Offset
    int8_t* wsPtr = static_cast<int8_t*>(workspace);
    void* tempStoragePtr = wsPtr + spaceOffset.tempStorageOffset;
    void* concatedScorePtr = wsPtr + spaceOffset.concatedScoreOffset;
    void* concatedBBoxPtr = wsPtr + spaceOffset.concatedBBoxOffset;
    void* sortedScorePtr = wsPtr + spaceOffset.sortedScoreOffset;
    void* sortedBBoxPtr = wsPtr + spaceOffset.sortedBBoxOffset;

    int blocks = N; //batch_size
    int threads = dMIN(topK, 2048);
    //Concat Scores and inBBox 
    switch (dtype)
    {
    case nvinfer1::DataType::kFLOAT:
        concatenate<float><<<blocks, threads, 0, stream>>>(featureCnt, topK, inScores, inBBox,
                                                          concatedScorePtr, concatedBBoxPtr);

        CUASSERT(hipGetLastError());
        break;
    case nvinfer1::DataType::kHALF: assert(false);
    default: assert(false);
    }


    //Sort and sample topK
    int itemCnt = topK * featureCnt;
    int* offsets = static_cast<int*>(tempStoragePtr);
    set_offset_kernel<<<1, 1024, 0, stream>>>(itemCnt, N + 1, offsets);
    assert(hipGetLastError() == hipSuccess);
    tempStoragePtr = static_cast<void*>(static_cast<int*>(tempStoragePtr) + (N + 1));

    //Sort
    size_t temp_storage_bytes = 0;
    hipcub::DeviceSegmentedRadixSort::SortPairsDescending(NULL, temp_storage_bytes, (float*) concatedScorePtr,
        (float*) sortedScorePtr, (BBoxT<float>*) concatedBBoxPtr, (BBoxT<float>*) sortedBBoxPtr, N * itemCnt, N,
        offsets, offsets + 1, 0, 8 * sizeof(float), stream);

    assert((1 << 23) * N > temp_storage_bytes);

    hipcub::DeviceSegmentedRadixSort::SortPairsDescending(tempStoragePtr, temp_storage_bytes, (float*) concatedScorePtr,
        (float*) sortedScorePtr, (BBoxT<float>*) concatedBBoxPtr, (BBoxT<float>*) sortedBBoxPtr, N * itemCnt, N,
        offsets, offsets + 1, 0, 8 * sizeof(float), stream);

    assert(hipGetLastError() == hipSuccess);

    //Sample
    resampleBBox_kernel<<<N, dMIN(topK, 1024), 0, stream>>>(itemCnt, topK, sortedBBoxPtr, outProposals);   
    
    assert(hipGetLastError() == hipSuccess);
    return hipGetLastError();
} 
