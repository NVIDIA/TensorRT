#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "kernel.h"
#include "bboxUtils.h"

#define CUBLAS_CHECK(condition)                                                                 \
    do                                                                                          \
    {                                                                                           \
        hipblasStatus_t status = condition;                                                      \
        if (status != HIPBLAS_STATUS_SUCCESS)                                                    \
        {                                                                                       \
            printf("%s %d CUBLAS FAIL %s\n", __FILE__, __LINE__, cublasGetErrorString(status)); \
        }                                                                                       \
    } while (0)
namespace nvinfer1
{
namespace plugin
{
size_t normalizePluginWorkspaceSize(bool acrossSpatial, int C, int H, int W)
{
    if (acrossSpatial)
        return sizeof(float) * C * H * W;
    else
        return (size_t) 0;
}
} // namespace plugin
} // namespace nvinfer1

size_t normalizePluginWorkspaceSize(bool acrossSpatial, int C, int H, int W)
{
    if (acrossSpatial)
        return sizeof(float) * C * H * W;
    else
        return (size_t) 0;
}

template <unsigned nthds_per_cta>
__launch_bounds__(nthds_per_cta)
    __global__ void normalizeNotAcrossSpatialKernel(
        const bool channelShared,
        const int N,
        const int C,
        const int H,
        const int W,
        const float eps,
        const float* scale,
        float* inputData,
        float* outputData)
{
    const int dim = C * H * W;
    const int spatialDim = H * W;
    const int tile = 32;
    const int numTile = (spatialDim + tile - 1) / tile;
    for (int n = blockIdx.x; n < N * numTile; n += gridDim.x)
    {
        float* input = inputData + (n / numTile) * dim;
        float* output = outputData + (n / numTile) * dim;
        __shared__ float sum[tile];
        float localsum = 0.0F;
        for (int i = threadIdx.x; i < tile; i += nthds_per_cta)
        {
            sum[i] = 0.0F;
        }
        __syncthreads();
        for (int i = threadIdx.x; i < C * tile; i += nthds_per_cta)
        {
            int row = i / tile;
            int col = (n % numTile) * tile + i % tile;
            float data = 0.0F;
            if (col < spatialDim)
                data = input[row * spatialDim + col];
            localsum += data * data;
        }
        atomicAdd(&sum[threadIdx.x & 31], localsum);
        __syncthreads();
        for (int i = threadIdx.x; i < C * tile; i += nthds_per_cta)
        {
            int row = i / tile;
            int col = (n % numTile) * tile + i % tile;
            if (col < spatialDim)
            {
                int offset = row * spatialDim + col;
                output[offset] = input[offset] / sqrt(sum[threadIdx.x & 31] + eps);
            }
        }
        if (channelShared)
        {
            for (int i = threadIdx.x; i < C * tile; i += nthds_per_cta)
            {
                int row = i / tile;
                int col = (n % numTile) * tile + i % tile;
                if (col < spatialDim)
                    output[row * spatialDim + col] *= scale[0];
            }
        }
        else
        {
            for (int i = threadIdx.x; i < C * tile; i += nthds_per_cta)
            {
                int row = i / tile;
                int col = (n % numTile) * tile + i % tile;
                if (col < spatialDim)
                    output[row * spatialDim + col] *= scale[row];
            }
        }
    }
}

pluginStatus_t normalizeNotAcrossSpatialGpu(
    hipStream_t stream,
    const bool channelShared,
    const int N,
    const int C,
    const int H,
    const int W,
    const float eps,
    const void* scale,
    const void* inputData,
    void* outputData)
{
    const int BS = 128;
    const int GS = 256;
    // assumes warp size == 32
    ASSERT(BS % 32 == 0);
    normalizeNotAcrossSpatialKernel<BS><<<GS, BS, 0, stream>>>(channelShared, N, C, H, W, eps,
                                                               (const float*) scale,
                                                               (float*) inputData,
                                                               (float*) outputData);
    CSC(hipGetLastError(), STATUS_FAILURE);
    return STATUS_SUCCESS;
}

__global__ void squareKernel(
    const int n,
    const float* x,
    float* y)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n; i += gridDim.x * blockDim.x)
    {
        y[i] = x[i] * x[i];
    }
}

__global__ void scalChannelKernel(
    const int n,
    const int spatialDim,
    const float* inputData,
    const float* scale,
    float* outputData)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n; i += gridDim.x * blockDim.x)
    {
        // scale factors are indepedent across different channels
        // scale[i / spatialDim]: find the right scale factor for specific channels
        outputData[i] = inputData[i] / scale[i / spatialDim];
    }
}
namespace nvinfer1
{
namespace plugin
{
pluginStatus_t normalizeInference(
    hipStream_t stream,
    hipblasHandle_t handle,
    const bool acrossSpatial,
    const bool channelShared,
    const int N,
    const int C,
    const int H,
    const int W,
    const float eps,
    const void* scale,
    const void* inputData,
    void* outputData,
    void* workspace)
{
    const int dim = C * H * W;
    // Normalization is conducted for each sample from the batch indepdently
    if (acrossSpatial)
    {
        float* input = (float*) const_cast<void*>(inputData);
        float* output = (float*) outputData;
        float* buffer = (float*) workspace;
        for (int n = 0; n < N; ++n)
        {
            // Take the square of each element in the input
            squareKernel<<<(dim + 511) / 512, 512, 0, stream>>>(dim, input, buffer);
            float normsqr = 0.0F;
            // Sum up all the squared elements
            CUBLAS_CHECK(hipblasSasum(handle, dim, buffer, 1, &normsqr));
            // Make a copy of the input to the output
            CUBLAS_CHECK(hipblasScopy(handle, dim, input, 1, output, 1));
            // Calculate the inverse of the square root of the sum
            // Use eps to prevent being divided by zero
            normsqr = 1 / sqrt(normsqr + eps);
            // Scale all the outputs by normsqr
            CUBLAS_CHECK(hipblasSscal(handle, dim, &normsqr, output, 1));
            // If channel shared is true, scale all the outputs
            if (channelShared)
            {
                CUBLAS_CHECK(hipblasSscal(handle, dim, (float*) scale, output, 1));
            }
            // Use different scale factors for different channels
            else
            {
                // scale the output according to channels
                scalChannelKernel<<<(dim + 511) / 512, 512, 0, stream>>>(dim, H * W, output, (float*) scale, output);
            }
            // Move cursors
            input += dim;
            output += dim;
        }
        return STATUS_SUCCESS;
    }
    // Normalization ignoring the batch
    else
    {
        return normalizeNotAcrossSpatialGpu(stream, channelShared, N, C, H, W, eps, scale, inputData, outputData);
    }
}
} // namespace plugin
} // namespace nvinfer1

pluginStatus_t normalizeInference(
    hipStream_t stream,
    hipblasHandle_t handle,
    const bool acrossSpatial,
    const bool channelShared,
    const int N,
    const int C,
    const int H,
    const int W,
    const float eps,
    const void* scale,
    const void* inputData,
    void* outputData,
    void* workspace)
{
    const int dim = C * H * W;
    // Normalization is conducted for each sample from the batch indepdently
    if (acrossSpatial)
    {
        float* input = (float*) const_cast<void*>(inputData);
        float* output = (float*) outputData;
        float* buffer = (float*) workspace;
        for (int n = 0; n < N; ++n)
        {
            // Take the square of each element in the input
            squareKernel<<<(dim + 511) / 512, 512, 0, stream>>>(dim, input, buffer);
            float normsqr = 0.0F;
            // Sum up all the squared elements
            CUBLAS_CHECK(hipblasSasum(handle, dim, buffer, 1, &normsqr));
            // Make a copy of the input to the output
            CUBLAS_CHECK(hipblasScopy(handle, dim, input, 1, output, 1));
            // Calculate the inverse of the square root of the sum
            // Use eps to prevent being divided by zero
            normsqr = 1 / sqrt(normsqr + eps);
            // Scale all the outputs by normsqr
            CUBLAS_CHECK(hipblasSscal(handle, dim, &normsqr, output, 1));
            // If channel shared is true, scale all the outputs
            if (channelShared)
            {
                CUBLAS_CHECK(hipblasSscal(handle, dim, (float*) scale, output, 1));
            }
            // Use different scale factors for different channels
            else
            {
                // scale the output according to channels
                scalChannelKernel<<<(dim + 511) / 512, 512, 0, stream>>>(dim, H * W, output, (float*) scale, output);
            }
            // Move cursors
            input += dim;
            output += dim;
        }
        return STATUS_SUCCESS;
    }
    // Normalization ignoring the batch
    else
    {
        return normalizeNotAcrossSpatialGpu(stream, channelShared, N, C, H, W, eps, scale, inputData, outputData);
    }
}
