/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "kernel.h"
#include "bboxUtils.h"

pluginStatus_t detectionInference(
    hipStream_t stream,
    const int N,
    const int C1,
    const int C2,
    const bool shareLocation,
    const bool varianceEncodedInTarget,
    const int backgroundLabelId,
    const int numPredsPerClass,
    const int numClasses,
    const int topK,
    const int keepTopK,
    const float confidenceThreshold,
    const float nmsThreshold,
    const CodeTypeSSD codeType,
    const DataType DT_BBOX,
    const void* locData,
    const void* priorData,
    const DataType DT_SCORE,
    const void* confData,
    void* keepCount,
    void* topDetections,
    void* workspace,
    bool isNormalized,
    bool confSigmoid)
{
    // Batch size * number bbox per sample * 4 = total number of bounding boxes * 4
    const int locCount = N * C1;
    // Do not clip the bounding box that goes outside the image
    const bool clipBBox = false;
    /*
     * shareLocation
     * Bounding box are shared among all classes, i.e., a bounding box could be classified as any candidate class.
     * Otherwise
     * Bounding box are designed for specific classes, i.e., a bounding box could be classified as one certain class or not (binary classification).
     */
    const int numLocClasses = shareLocation ? 1 : numClasses;

    size_t bboxDataSize = detectionForwardBBoxDataSize(N, C1, DataType::kFLOAT);
    void* bboxDataRaw = workspace;

    pluginStatus_t status = decodeBBoxes(stream,
                                      locCount,
                                      codeType,
                                      varianceEncodedInTarget,
                                      numPredsPerClass,
                                      shareLocation,
                                      numLocClasses,
                                      backgroundLabelId,
                                      clipBBox,
                                      DataType::kFLOAT,
                                      locData,
                                      priorData,
                                      bboxDataRaw);

    ASSERT_FAILURE(status == STATUS_SUCCESS);

    /*
     * bboxDataRaw format:
     * [batch size, numPriors (per sample), numLocClasses, 4]
     */
    // float for now
    void* bboxData;
    size_t bboxPermuteSize = detectionForwardBBoxPermuteSize(shareLocation, N, C1, DataType::kFLOAT);
    void* bboxPermute = nextWorkspacePtr((int8_t*) bboxDataRaw, bboxDataSize);

    /*
     * After permutation, bboxData format:
     * [batch_size, numLocClasses, numPriors (per sample) (numPredsPerClass), 4]
     * This is equivalent to swapping axis
     */
    if (!shareLocation)
    {
        status = permuteData(stream,
                             locCount,
                             numLocClasses,
                             numPredsPerClass,
                             4,
                             DataType::kFLOAT,
                             false,
                             bboxDataRaw,
                             bboxPermute);
        ASSERT_FAILURE(status == STATUS_SUCCESS);
        bboxData = bboxPermute;
    }
    /*
     * If shareLocation, numLocClasses = 1
     * No need to permute data on linear memory
     */
    else
    {
        bboxData = bboxDataRaw;
    }
    /*
     * Conf data format
     * [batch size, numPriors * param.numClasses, 1, 1]
     */
    const int numScores = N * C2;
    size_t scoresSize = detectionForwardPreNMSSize(N, C2);
    void* scores = nextWorkspacePtr((int8_t*) bboxPermute, bboxPermuteSize);
    // need a conf_scores
    /*
     * After permutation, bboxData format:
     * [batch_size, numClasses, numPredsPerClass, 1]
     */
    status = permuteData(stream,
                         numScores,
                         numClasses,
                         numPredsPerClass,
                         1,
                         DataType::kFLOAT,
                         confSigmoid,
                         confData,
                         scores);
    ASSERT_FAILURE(status == STATUS_SUCCESS);

    size_t indicesSize = detectionForwardPreNMSSize(N, C2);
    void* indices = nextWorkspacePtr((int8_t*) scores, scoresSize);

    size_t postNMSScoresSize = detectionForwardPostNMSSize(N, numClasses, topK);
    size_t postNMSIndicesSize = detectionForwardPostNMSSize(N, numClasses, topK);
    void* postNMSScores = nextWorkspacePtr((int8_t*) indices, indicesSize);
    void* postNMSIndices = nextWorkspacePtr((int8_t*) postNMSScores, postNMSScoresSize);

    //size_t sortingWorkspaceSize = sortScoresPerClassWorkspaceSize(N, numClasses, numPredsPerClass, FLOAT32);
    void* sortingWorkspace = nextWorkspacePtr((int8_t*) postNMSIndices, postNMSIndicesSize);
    // Sort the scores so that the following NMS could be applied.
    status = sortScoresPerClass(stream,
                                N,
                                numClasses,
                                numPredsPerClass,
                                backgroundLabelId,
                                confidenceThreshold,
                                DataType::kFLOAT,
                                scores,
                                indices,
                                sortingWorkspace);
    ASSERT_FAILURE(status == STATUS_SUCCESS);

    // NMS
    status = allClassNMS(stream,
                         N,
                         numClasses,
                         numPredsPerClass,
                         topK,
                         nmsThreshold,
                         shareLocation,
                         isNormalized,
                         DataType::kFLOAT,
                         DataType::kFLOAT,
                         bboxData,
                         scores,
                         indices,
                         postNMSScores,
                         postNMSIndices,
                         false);
    ASSERT_FAILURE(status == STATUS_SUCCESS);

    // Sort the bounding boxes after NMS using scores
    status = sortScoresPerImage(stream,
                                N,
                                numClasses * topK,
                                DataType::kFLOAT,
                                postNMSScores,
                                postNMSIndices,
                                scores,
                                indices,
                                sortingWorkspace);
    ASSERT_FAILURE(status == STATUS_SUCCESS);

    // Gather data from the sorted bounding boxes after NMS
    status = gatherTopDetections(stream,
                                 shareLocation,
                                 N,
                                 numPredsPerClass,
                                 numClasses,
                                 topK,
                                 keepTopK,
                                 DataType::kFLOAT,
                                 DataType::kFLOAT,
                                 indices,
                                 scores,
                                 bboxData,
                                 keepCount,
                                 topDetections);
    ASSERT_FAILURE(status == STATUS_SUCCESS);

    return STATUS_SUCCESS;
}

namespace nvinfer1
{
namespace plugin
{
    pluginStatus_t detectionInference(
    hipStream_t stream,
    const int N,
    const int C1,
    const int C2,
    const bool shareLocation,
    const bool varianceEncodedInTarget,
    const int backgroundLabelId,
    const int numPredsPerClass,
    const int numClasses,
    const int topK,
    const int keepTopK,
    const float confidenceThreshold,
    const float nmsThreshold,
    const CodeTypeSSD codeType,
    const DataType DT_BBOX,
    const void* locData,
    const void* priorData,
    const DataType DT_SCORE,
    const void* confData,
    void* keepCount,
    void* topDetections,
    void* workspace,
    bool isNormalized,
    bool confSigmoid)
{
    // Batch size * number bbox per sample * 4 = total number of bounding boxes * 4
    const int locCount = N * C1;
    // Do not clip the bounding box that goes outside the image
    const bool clipBBox = false;
    /*
     * shareLocation
     * Bounding box are shared among all classes, i.e., a bounding box could be classified as any candidate class.
     * Otherwise
     * Bounding box are designed for specific classes, i.e., a bounding box could be classified as one certain class or not (binary classification).
     */
    const int numLocClasses = shareLocation ? 1 : numClasses;

    size_t bboxDataSize = detectionForwardBBoxDataSize(N, C1, DataType::kFLOAT);
    void* bboxDataRaw = workspace;

    pluginStatus_t status = decodeBBoxes(stream,
                                      locCount,
                                      codeType,
                                      varianceEncodedInTarget,
                                      numPredsPerClass,
                                      shareLocation,
                                      numLocClasses,
                                      backgroundLabelId,
                                      clipBBox,
                                      DataType::kFLOAT,
                                      locData,
                                      priorData,
                                      bboxDataRaw);

    ASSERT_FAILURE(status == STATUS_SUCCESS);

    /*
     * bboxDataRaw format:
     * [batch size, numPriors (per sample), numLocClasses, 4]
     */
    // float for now
    void* bboxData;
    size_t bboxPermuteSize = detectionForwardBBoxPermuteSize(shareLocation, N, C1, DataType::kFLOAT);
    void* bboxPermute = nextWorkspacePtr((int8_t*) bboxDataRaw, bboxDataSize);

    /*
     * After permutation, bboxData format:
     * [batch_size, numLocClasses, numPriors (per sample) (numPredsPerClass), 4]
     * This is equivalent to swapping axis
     */
    if (!shareLocation)
    {
        status = permuteData(stream,
                             locCount,
                             numLocClasses,
                             numPredsPerClass,
                             4,
                             DataType::kFLOAT,
                             false,
                             bboxDataRaw,
                             bboxPermute);
        ASSERT_FAILURE(status == STATUS_SUCCESS);
        bboxData = bboxPermute;
    }
    /*
     * If shareLocation, numLocClasses = 1
     * No need to permute data on linear memory
     */
    else
    {
        bboxData = bboxDataRaw;
    }
    /*
     * Conf data format
     * [batch size, numPriors * param.numClasses, 1, 1]
     */
    const int numScores = N * C2;
    size_t scoresSize = detectionForwardPreNMSSize(N, C2);
    void* scores = nextWorkspacePtr((int8_t*) bboxPermute, bboxPermuteSize);
    // need a conf_scores
    /*
     * After permutation, bboxData format:
     * [batch_size, numClasses, numPredsPerClass, 1]
     */
    status = permuteData(stream,
                         numScores,
                         numClasses,
                         numPredsPerClass,
                         1,
                         DataType::kFLOAT,
                         confSigmoid,
                         confData,
                         scores);
    ASSERT_FAILURE(status == STATUS_SUCCESS);

    size_t indicesSize = detectionForwardPreNMSSize(N, C2);
    void* indices = nextWorkspacePtr((int8_t*) scores, scoresSize);

    size_t postNMSScoresSize = detectionForwardPostNMSSize(N, numClasses, topK);
    size_t postNMSIndicesSize = detectionForwardPostNMSSize(N, numClasses, topK);
    void* postNMSScores = nextWorkspacePtr((int8_t*) indices, indicesSize);
    void* postNMSIndices = nextWorkspacePtr((int8_t*) postNMSScores, postNMSScoresSize);

    //size_t sortingWorkspaceSize = sortScoresPerClassWorkspaceSize(N, numClasses, numPredsPerClass, FLOAT32);
    void* sortingWorkspace = nextWorkspacePtr((int8_t*) postNMSIndices, postNMSIndicesSize);
    // Sort the scores so that the following NMS could be applied.
    status = sortScoresPerClass(stream,
                                N,
                                numClasses,
                                numPredsPerClass,
                                backgroundLabelId,
                                confidenceThreshold,
                                DataType::kFLOAT,
                                scores,
                                indices,
                                sortingWorkspace);
    ASSERT_FAILURE(status == STATUS_SUCCESS);

    // NMS
    status = allClassNMS(stream,
                         N,
                         numClasses,
                         numPredsPerClass,
                         topK,
                         nmsThreshold,
                         shareLocation,
                         isNormalized,
                         DataType::kFLOAT,
                         DataType::kFLOAT,
                         bboxData,
                         scores,
                         indices,
                         postNMSScores,
                         postNMSIndices,
                         false);
    ASSERT_FAILURE(status == STATUS_SUCCESS);

    // Sort the bounding boxes after NMS using scores
    status = sortScoresPerImage(stream,
                                N,
                                numClasses * topK,
                                DataType::kFLOAT,
                                postNMSScores,
                                postNMSIndices,
                                scores,
                                indices,
                                sortingWorkspace);
    ASSERT_FAILURE(status == STATUS_SUCCESS);

    // Gather data from the sorted bounding boxes after NMS
    status = gatherTopDetections(stream,
                                 shareLocation,
                                 N,
                                 numPredsPerClass,
                                 numClasses,
                                 topK,
                                 keepTopK,
                                 DataType::kFLOAT,
                                 DataType::kFLOAT,
                                 indices,
                                 scores,
                                 bboxData,
                                 keepCount,
                                 topDetections);
    ASSERT_FAILURE(status == STATUS_SUCCESS);

    return STATUS_SUCCESS;
}

} // namespace plugin
} // namespace nvinfer1
