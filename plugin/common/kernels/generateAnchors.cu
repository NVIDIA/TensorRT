#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 1993-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "common/kernel.h"
#include <cstdio>

pluginStatus_t generateAnchors_cpu(
    int numRatios, float* ratios, int numScales, float* scales, int baseSize, float* anchors)
{
#ifdef DEBUG
    DEBUG_PRINTF("Generating Anchors with:\n");
    DEBUG_PRINTF("Scales:");
    for (int s = 0; s < numScales; ++s)
    {
        DEBUG_PRINTF("%f\t", scales[s]);
    }
    DEBUG_PRINTF("\n");
    DEBUG_PRINTF("Ratios:");
    for (int r = 0; r < numRatios; ++r)
    {
        DEBUG_PRINTF("%f\t", ratios[r]);
    }
    DEBUG_PRINTF("\n");
#endif

    if ((numScales <= 0) || (numRatios <= 0) || (baseSize <= 0))
    {
        return STATUS_BAD_PARAM;
    }

    // Generate parameters for numRatios * numScales general anchor boxes
    for (int r = 0; r < numRatios; ++r)
    {
        for (int s = 0; s < numScales; ++s)
        {
            int id = r * numScales + s;
            float scale = scales[s];
            float ratio = ratios[r];
            float bs = baseSize;
            float ws = round(sqrt((float) (bs * bs) / ratio));
            float hs = round(ws * ratio);
            // Width: bs / sqrt(ratio) * scale
            // Height: bs * sqrt(ratio) * scale
            ws *= scale;
            hs *= scale;

            // x_anchor_ctr
            /*
             * This value should not useful in this implementation of generating numRatios * numScales general anchor boxes.
             * Because the center of anchor box in the original input raw image scale will not be dependent on this.
             */
            anchors[id * 4] = (bs - 1) / 2;
            // y_anchor_ctr
            /*
             * This value should not useful in this implementation of generating numRatios * numScales general anchor boxes.
             * Because the center of anchor box in the original input raw image scale will not be dependent on this.
             */
            anchors[id * 4 + 1] = (bs - 1) / 2;
            // w_anchor
            anchors[id * 4 + 2] = ws;
            // h_anchor
            anchors[id * 4 + 3] = hs;
        }
    }
    return STATUS_SUCCESS;
}

pluginStatus_t generateAnchors(hipStream_t stream,
                              int numRatios,
                              float* ratios,
                              int numScales,
                              float* scales,
                              int baseSize,
                              float* anchors)
{
    // Each anchor box has 4 parameters
    int ac = numRatios * numScales * 4;
    float* anchors_cpu;
    CSC(hipHostMalloc((void**) &anchors_cpu, sizeof(float) * ac), STATUS_FAILURE);
    pluginStatus_t status = generateAnchors_cpu(numRatios, ratios, numScales, scales, baseSize, anchors_cpu);
    CSC(hipMemcpyAsync(anchors, anchors_cpu, sizeof(float) * ac, hipMemcpyHostToDevice, stream), STATUS_FAILURE);
    CSC(hipHostFree(anchors_cpu), STATUS_FAILURE);
    return status;
}
