#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "hipcub/hipcub.hpp"
#include <array>
#include "kernel.h"
#include "bboxUtils.h"
#include "cub_helper.h"

template <typename T_SCORE>
pluginStatus_t sortScoresPerImage_gpu(
    hipStream_t stream,
    const int num_images,
    const int num_items_per_image,
    void* unsorted_scores,
    void* unsorted_bbox_indices,
    void* sorted_scores,
    void* sorted_bbox_indices,
    void* workspace,
    int score_bits
)
{
    void* d_offsets = workspace;
    void* cubWorkspace = nextWorkspacePtr((int8_t*) d_offsets, (num_images + 1) * sizeof(int));

    setUniformOffsets(stream, num_images, num_items_per_image, (int*) d_offsets);

    const int arrayLen = num_images * num_items_per_image;
    size_t temp_storage_bytes = cubSortPairsWorkspaceSize<T_SCORE, int>(arrayLen, num_images);
    size_t begin_bit = 0;
    size_t end_bit = sizeof(T_SCORE) * 8;
    if (sizeof(T_SCORE) == 2 && score_bits > 0 && score_bits <= 10)
    {
        end_bit = 10;
        begin_bit = end_bit - score_bits;
    }
    hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
        cubWorkspace, temp_storage_bytes,
        (const T_SCORE*) (unsorted_scores), (T_SCORE*) (sorted_scores),
        (const int*) (unsorted_bbox_indices), (int*) (sorted_bbox_indices),
        arrayLen, num_images,
        (const int*) d_offsets, (const int*) d_offsets + 1,
        begin_bit, end_bit,
        stream);
    CSC(hipGetLastError(), STATUS_FAILURE);
    return STATUS_SUCCESS;
}

// sortScoresPerImage LAUNCH CONFIG
typedef pluginStatus_t (*sspiFunc)(hipStream_t,
                                const int,
                                const int,
                                void*,
                                void*,
                                void*,
                                void*,
                                void*,
                                int);
struct sspiLaunchConfig
{
    DataType t_score;
    sspiFunc function;

    sspiLaunchConfig(DataType t_score)
        : t_score(t_score)
    {
    }
    sspiLaunchConfig(DataType t_score, sspiFunc function)
        : t_score(t_score)
        , function(function)
    {
    }
    bool operator==(const sspiLaunchConfig& other)
    {
        return t_score == other.t_score;
    }
};

static std::array<sspiLaunchConfig, 2> sspiLCOptions = {
    sspiLaunchConfig(DataType::kFLOAT, sortScoresPerImage_gpu<float>),
    sspiLaunchConfig(DataType::kHALF, sortScoresPerImage_gpu<__half>),
};

pluginStatus_t sortScoresPerImage(
    hipStream_t stream,
    const int num_images,
    const int num_items_per_image,
    const DataType DT_SCORE,
    void* unsorted_scores,
    void* unsorted_bbox_indices,
    void* sorted_scores,
    void* sorted_bbox_indices,
    void* workspace,
    int score_bits
)
{
    sspiLaunchConfig lc = sspiLaunchConfig(DT_SCORE);
    for (unsigned i = 0; i < sspiLCOptions.size(); ++i)
    {
        if (lc == sspiLCOptions[i])
        {
            DEBUG_PRINTF("sortScoresPerImage kernel %d\n", i);
            return sspiLCOptions[i].function(stream,
                                           num_images,
                                           num_items_per_image,
                                           unsorted_scores,
                                           unsorted_bbox_indices,
                                           sorted_scores,
                                           sorted_bbox_indices,
                                           workspace,
                                           score_bits);
        }
    }
    return STATUS_BAD_PARAM;
}

size_t sortScoresPerImageWorkspaceSize(
    const int num_images,
    const int num_items_per_image,
    const DataType DT_SCORE)
{
    const int arrayLen = num_images * num_items_per_image;
    size_t wss[2];
    wss[0] = (num_images + 1) * sizeof(int); // offsets
    if (DT_SCORE == DataType::kFLOAT)
    {
        wss[1] = cubSortPairsWorkspaceSize<float, int>(arrayLen, num_images); // cub workspace
    }
    else if (DT_SCORE == DataType::kHALF)
    {
        wss[1] = cubSortPairsWorkspaceSize<__half, int>(arrayLen, num_images); // cub workspace
    }
    else
    {
        printf("SCORE type not supported.\n");
        return (size_t) -1;
    }

    return calculateTotalWorkspaceSize(wss, 2);
}
