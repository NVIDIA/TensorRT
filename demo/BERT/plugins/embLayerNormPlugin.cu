#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "NvInfer.h"
#include "embLayerNormPlugin.h"
#include "logger.h"
#include "pluginKernels.h"
#include "pluginUtil.h"

#include <cassert>
#include <cstring>
#include <vector>

using bert::operator+;

namespace bert
{

template <typename T, unsigned TPB>
__global__ void embLayerNormKernel(int ld, const int* inputIds, const int* tokenIds, const float* beta,
    const float* gamma, const float* wordEmb, const float* posEmb, const float* tokEmb, T* output)
{

    hipcub::Sum pairSum;
    // 1. lookup word and token of the block
    // blockIdx.x = position in the sequence
    // blockIdx.y = batch
    // gridDim.x = S
    // gridDim.y = B
    __shared__ int wordId;
    __shared__ int tokenId;

    const T rld = T(1.f) / T(ld);
    const int seqPos = blockIdx.y * gridDim.x + blockIdx.x;
    if (threadIdx.x == 0)
    {
        wordId = inputIds[seqPos];
        tokenId = tokenIds[seqPos];
    }
    __syncthreads();

    // 2. load pos/tok/word embeddings and add them toghether
    // offset into embeddings is given by wordId * hidden_size
    const int poffset = blockIdx.x * ld;
    const int woffset = wordId * ld;
    const int toffset = tokenId * ld;
    // the output offset is given by b * (S*hidden_size) + s * hidden_size
    const int outOffset = seqPos * ld;

    kvp<T> threadData(0, 0);

    for (int it = threadIdx.x; it < ld; it += TPB)
    {
        const T w(wordEmb[woffset + it]);
        const T t(tokEmb[toffset + it]);
        const T p(posEmb[poffset + it]);
        const T val = w + t + p;

        output[outOffset + it] = val;
        const T rldval = rld * val;
        threadData = pairSum(threadData, kvp<T>(rldval, rldval * val));
    }

    // 3. layer norm on the sum
    layerNorm<T, TPB>(threadData, ld, outOffset, beta, gamma, output);
}

template <typename T>
int embSkipLayerNorm(hipStream_t stream, int ld, int B, int S, const int* inputIds, const int* token_ids,
    const float* beta, const float* gamma, const float* wordEmb, const float* posEmb, const float* tokEmb, T* output)
{

    constexpr int tpb = 256;
    const dim3 grid(S, B, 1);
    const dim3 block(tpb, 1, 1);

    embLayerNormKernel<T, tpb>
        <<<grid, block, 0, stream>>>(ld, inputIds, token_ids, beta, gamma, wordEmb, posEmb, tokEmb, output);
    CHECK(hipPeekAtLastError());

    return 0;
}

using namespace nvinfer1;

// Clip plugin specific constants
namespace
{
static const char* EMB_LAYER_NORM_VERSION{"1"};
static const char* EMB_LAYER_NORM_NAME{"CustomEmbLayerNormPlugin"};
} // namespace

// Static class fields initialization
PluginFieldCollection EmbLayerNormPluginCreator::mFC{};
std::vector<PluginField> EmbLayerNormPluginCreator::mPluginAttributes;

REGISTER_TENSORRT_PLUGIN(EmbLayerNormPluginCreator);

EmbLayerNormPlugin::EmbLayerNormPlugin(const std::string& name, const bool outputFp16, const Weights& beta,
    const Weights& gamma, const Weights& wordEmb, const Weights& posEmb, const Weights& tokEmb)
    : mLayerName(name)
    , mLd(beta.count)
    , mGamma(gamma)
    , mBeta(beta)
    , mWordEmb(wordEmb)
    , mPosEmb(posEmb)
    , mTokEmb(tokEmb)
    , mGammaDev(nullptr)
    , mBetaDev(nullptr)
    , mWordEmbDev(nullptr)
    , mTokEmbDev(nullptr)
    , mPosEmbDev(nullptr)
{
    // Assuming Weights.count is the number of elements and not bytes
    assert(beta.count == gamma.count);
    assert(wordEmb.count % mLd == 0);
    assert(posEmb.count % mLd == 0);
    assert(tokEmb.count % mLd == 0);
    mWordVocabSize = wordEmb.count / mLd;
    mPosVocabSize = posEmb.count / mLd;
    mTokVocabSize = tokEmb.count / mLd;
    // We set mS in configure
    mType = outputFp16 ? DataType::kHALF : DataType::kFLOAT;
}

EmbLayerNormPlugin::EmbLayerNormPlugin(const std::string& name, const void* data, size_t length)
    : mLayerName(name)
{
    gLogVerbose << "EMB LN Deser start\n";
    // Deserialize in the same order as serialization
    const char* d = static_cast<const char*>(data);
    const char* a = d;
    DESER(d, mType);
    DESER(d, mLd);
    DESER(d, mS);
    DESER(d, mWordVocabSize);
    DESER(d, mPosVocabSize);
    DESER(d, mTokVocabSize);
    mBetaDev = deserToDev<float>(d, mLd);
    mGammaDev = deserToDev<float>(d, mLd);

    mWordEmbDev = deserToDev<float>(d, mLd * mWordVocabSize);
    mPosEmbDev = deserToDev<float>(d, mLd * mPosVocabSize);
    mTokEmbDev = deserToDev<float>(d, mLd * mTokVocabSize);
    assert(d == (a + length));
    // this signals init not to allocate/copy
    mGamma.count = -1;
    mBeta.count = -1;
    mWordEmb.count = -1;
    mTokEmb.count = -1;
    mPosEmb.count = -1;
    mGamma.values = nullptr;
    mBeta.values = nullptr;
    mWordEmb.values = nullptr;
    mTokEmb.values = nullptr;
    mPosEmb.values = nullptr;

    gLogVerbose << "EMB LN Deser done\n";
}

const char* EmbLayerNormPlugin::getPluginType() const
{
    return EMB_LAYER_NORM_NAME;
}

const char* EmbLayerNormPlugin::getPluginVersion() const
{
    return EMB_LAYER_NORM_VERSION;
}

int EmbLayerNormPlugin::getNbOutputs() const
{
    return 2;
}

DataType EmbLayerNormPlugin::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const
{
    assert(index == 0 || index == 1);
    if (index == 0)
    {
        assert(mType == DataType::kHALF || mType == DataType::kFLOAT);
        return mType;
    }
    return DataType::kINT32;
}

Dims EmbLayerNormPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims)
{
    // Input should be input ids and token ids and the input mask
    // Output should be the embeddings tensor and mask indices
    assert(nbInputDims == 3);
    assert(inputs[0].nbDims == 1); // S
    assert(inputs[0].nbDims == inputs[1].nbDims);
    const int S = inputs[0].d[0];
    assert(inputs[1].d[0] == S);
    assert(inputs[2].d[0] == S);

    assert(index == 0 || index == 1);

    if (index == 0)
    {
        const int hidden_size = mLd;
        return Dims4{S, hidden_size, 1, 1};
    }
    return Dims{1, 1};
}

int EmbLayerNormPlugin::initialize()
{
    if (mGamma.values)
    {
        CHECK(hipMalloc(&mGammaDev, sizeof(float) * mGamma.count));
        CHECK(hipMemcpy(mGammaDev, mGamma.values, sizeof(float) * mGamma.count, hipMemcpyHostToDevice));
    }
    if (mBeta.values)
    {
        CHECK(hipMalloc(&mBetaDev, sizeof(float) * mBeta.count));
        CHECK(hipMemcpy(mBetaDev, mBeta.values, sizeof(float) * mBeta.count, hipMemcpyHostToDevice));
    }

    if (mWordEmb.values)
    {
        CHECK(hipMalloc(&mWordEmbDev, sizeof(float) * mWordEmb.count));
        CHECK(hipMemcpy(mWordEmbDev, mWordEmb.values, sizeof(float) * mWordEmb.count, hipMemcpyHostToDevice));
    }
    if (mTokEmb.values)
    {
        CHECK(hipMalloc(&mTokEmbDev, sizeof(float) * mTokEmb.count));
        CHECK(hipMemcpy(mTokEmbDev, mTokEmb.values, sizeof(float) * mTokEmb.count, hipMemcpyHostToDevice));
    }

    if (mPosEmb.values)
    {
        CHECK(hipMalloc(&mPosEmbDev, sizeof(float) * mPosEmb.count));
        CHECK(hipMemcpy(mPosEmbDev, mPosEmb.values, sizeof(float) * mPosEmb.count, hipMemcpyHostToDevice));
    }
    return 0;
}

int EmbLayerNormPlugin::enqueue(int batchSize, const void* const* inputs, void** outputs, void*, hipStream_t stream)
{
    int status = -1;

    // Our plugin outputs only one tensor
    const int* inputIds = static_cast<const int*>(inputs[0]);
    const int* segmentIds = static_cast<const int*>(inputs[1]);
    const int* inputMask = static_cast<const int*>(inputs[2]);

    if (mType == DataType::kFLOAT)
    {
        float* output = static_cast<float*>(outputs[0]);
        embSkipLayerNorm<float>(stream, mLd, batchSize, mS, inputIds, segmentIds, mBetaDev, mGammaDev, mWordEmbDev,
            mPosEmbDev, mTokEmbDev, output);
    }
    else if (mType == DataType::kHALF)
    {
        half* output = static_cast<half*>(outputs[0]);
        embSkipLayerNorm<half>(stream, mLd, batchSize, mS, inputIds, segmentIds, mBetaDev, mGammaDev, mWordEmbDev,
            mPosEmbDev, mTokEmbDev, output);
    }
    else
    {
        assert(false);
    }
    int* maskIdx = static_cast<int*>(outputs[1]);
    computeMaskIdx(stream, mS, batchSize, inputMask, maskIdx);

    return status;
}

size_t EmbLayerNormPlugin::getSerializationSize() const
{
    return 2 * sizeof(float) * mLd             // beta + gamma
        + sizeof(mType) + sizeof(mLd) * 5      //mLd, mS, m*VocabSize
        + sizeof(float) * mLd * mWordVocabSize // word emb
        + sizeof(float) * mLd * mPosVocabSize  // pos emb
        + sizeof(float) * mLd * mTokVocabSize  // tok emb
        ;
}

void EmbLayerNormPlugin::serialize(void* buffer) const
{
    char* d = static_cast<char*>(buffer);
    const char* a = d;
    writeToBuffer(d, mType);
    writeToBuffer(d, mLd);
    writeToBuffer(d, mS);
    writeToBuffer(d, mWordVocabSize);
    writeToBuffer(d, mPosVocabSize);
    writeToBuffer(d, mTokVocabSize);
    serFromDev(d, mBetaDev, mLd);
    serFromDev(d, mGammaDev, mLd);
    serFromDev(d, mWordEmbDev, mLd * mWordVocabSize);
    serFromDev(d, mPosEmbDev, mLd * mPosVocabSize);
    serFromDev(d, mTokEmbDev, mLd * mTokVocabSize);

    assert(d == a + getSerializationSize());
}

void EmbLayerNormPlugin::configurePlugin(const Dims* inputs, int nbInputs, const Dims* outputs, int nbOutputs,
    const DataType* inputTypes, const DataType* outputTypes, const bool* inputIsBroadcast,
    const bool* outputIsBroadcast, PluginFormat format, int maxBatchSize)
{

    // Validate input arguments
    assert(nbOutputs == 2);
    assert(nbInputs == 3);

    assert(inputs[0].nbDims == 1);
    mS = inputs[0].d[0];
    assert(mS == inputs[1].d[0]);
    assert(mS == inputs[2].d[0]);

    assert(outputs[0].nbDims == 4);
    assert(outputs[0].d[0] == mS);
    assert(outputs[0].d[1] == mLd);
    assert(outputs[0].d[2] == 1);
    assert(outputs[0].d[3] == 1);

    assert(outputs[1].nbDims == 1);
    assert(outputs[1].d[0] == 1);

    assert(format == PluginFormat::kNCHW);
    assert(inputTypes[0] == DataType::kINT32);
    assert(inputTypes[1] == DataType::kINT32);
    assert(inputTypes[2] == DataType::kINT32);
    const DataType out_type = outputTypes[0];
    assert(out_type == DataType::kFLOAT || out_type == DataType::kHALF);
    assert(outputTypes[1] == DataType::kINT32);
}

bool EmbLayerNormPlugin::supportsFormat(DataType type, PluginFormat format) const
{
    if (type == DataType::kINT32 || type == DataType::kFLOAT || type == DataType::kHALF)
    {
        return format == PluginFormat::kNCHW;
    }
    else
    {
        return false;
    }
}

void EmbLayerNormPlugin::terminate()
{
    gLogVerbose << "EMBLN terminate start" << std::endl;
    hipFree(mGammaDev);
    hipFree(mBetaDev);
    hipFree(mWordEmbDev);
    hipFree(mTokEmbDev);
    hipFree(mPosEmbDev);
    gLogVerbose << "EMBLN terminate done" << std::endl;
}

void EmbLayerNormPlugin::destroy()
{
    gLogVerbose << "EMBLN destroy start" << std::endl;
    // This gets called when the network containing plugin is destroyed
    delete this;
    gLogVerbose << "EMBLN destroy start" << std::endl;
}

IPluginV2Ext* EmbLayerNormPlugin::clone() const
{
    gLogVerbose << "EMBLN clone start" << std::endl;
    auto ret = new EmbLayerNormPlugin(mLayerName, mType == DataType::kHALF, mBeta, mGamma, mWordEmb, mPosEmb, mTokEmb);
    ret->mS = mS;

    ret->mWordEmbDev = mWordEmbDev;
    ret->mPosEmbDev = mPosEmbDev;
    ret->mTokEmbDev = mTokEmbDev;
    ret->mBetaDev = mBetaDev;
    ret->mGammaDev = mGammaDev;
    gLogVerbose << "EMBLN clone done" << std::endl;
    return ret;
}

void EmbLayerNormPlugin::setPluginNamespace(const char* libNamespace)
{
    mNamespace = libNamespace;
}

const char* EmbLayerNormPlugin::getPluginNamespace() const
{
    return mNamespace.c_str();
}

EmbLayerNormPluginCreator::EmbLayerNormPluginCreator()
{
    mFC.nbFields = mPluginAttributes.size();
    mFC.fields = mPluginAttributes.data();
}

const char* EmbLayerNormPluginCreator::getPluginName() const
{
    return EMB_LAYER_NORM_NAME;
}

const char* EmbLayerNormPluginCreator::getPluginVersion() const
{
    return EMB_LAYER_NORM_VERSION;
}

const PluginFieldCollection* EmbLayerNormPluginCreator::getFieldNames()
{
    return &mFC;
}

IPluginV2* EmbLayerNormPluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc)
{
    gLogVerbose << "Creating EmbLayerNormPlugin...\n";

    bool output_fp16 = true;
    Weights beta;
    Weights gamma;
    Weights word_emb;
    Weights pos_emb;
    Weights tok_emb;
    for(int i=0; i< fc->nbFields; i++)
    {
        std::string field_name(fc->fields[i].name);
        if (field_name.compare("bert_embeddings_layernorm_beta")==0)
        {
            gLogVerbose << "Building bert_embeddings_layernorm_beta...\n";
            beta.values = fc->fields[i].data;
            beta.count = fc->fields[i].length;
            beta.type = static_cast<DataType>(fc->fields[i].type);
        }

        if (field_name.compare("bert_embeddings_layernorm_gamma")==0)
        {
            gLogVerbose << "Building bert_embeddings_layernorm_gamma...\n";
            gamma.values = fc->fields[i].data;
            gamma.count = fc->fields[i].length;
            gamma.type = static_cast<DataType>(fc->fields[i].type);
        }

        if (field_name.compare("bert_embeddings_word_embeddings")==0)
        {
            gLogVerbose << "Building bert_embeddings_word_embeddings...\n";
            word_emb.values = fc->fields[i].data;
            word_emb.count = fc->fields[i].length;
            word_emb.type = static_cast<DataType>(fc->fields[i].type);
        }

        if (field_name.compare("bert_embeddings_token_type_embeddings")==0)
        {
            gLogVerbose << "Building bert_embeddings_token_type_embeddings...\n";
            tok_emb.values = fc->fields[i].data;
            tok_emb.count = fc->fields[i].length;
            tok_emb.type = static_cast<DataType>(fc->fields[i].type);
        }

        if (field_name.compare("bert_embeddings_position_embeddings")==0)
        {
            gLogVerbose << "Building bert_embeddings_position_embeddings...\n";
            pos_emb.values = fc->fields[i].data;
            pos_emb.count = fc->fields[i].length;
            pos_emb.type = static_cast<DataType>(fc->fields[i].type);
        }
    }

    gLogVerbose << "Building the Plugin...\n";
    EmbLayerNormPlugin* p =  new EmbLayerNormPlugin(name, output_fp16, beta, gamma, word_emb, pos_emb, tok_emb);
    return p;
}

IPluginV2* EmbLayerNormPluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength)
{
    // This object will be deleted when the network is destroyed, which will
    // call EmbLayerNormPlugin::destroy()
    return new EmbLayerNormPlugin(name, serialData, serialLength);
}

void EmbLayerNormPluginCreator::setPluginNamespace(const char* libNamespace)
{
    mNamespace = libNamespace;
}

const char* EmbLayerNormPluginCreator::getPluginNamespace() const
{
    return mNamespace.c_str();
}
}
